
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU
#include <cuda.h>
#include <cuda_runtime.h>
#include <unsupported/Eigen/CXX11/Tensor>
#include <TensorBase/benchmarks/BenchmarkPixelsGpu.h>

using namespace TensorBase;
using namespace TensorBaseBenchmarks;

/* Benchmark for toy 4D pixels data where x, y, and z describe the coordinates of the pixel in 3D space (type=int),
	t describes the time of the pixel (type=int), and the value of the pixel (from 0 to 255) describes the intensity of the pixel
*/
int main(int argc, char** argv)
{
	// Parse the user commands
	std::string data_dir = "C:/Users/dmccloskey/Documents/GitHub/mnist/";
	int n_dims = 2;
	int data_size = 1296;
	bool in_memory = true;
	double shard_span_perc = 1;
  int n_engines = 1;
	parseCmdArgs(argc, argv, data_dir, n_dims, data_size, in_memory, shard_span_perc, n_engines);

	// Setup the Benchmarking suite
	Benchmark1TimePointGpu<int, float> benchmark_1_tp;
	//Benchmark1TimePointGpu<int, int> benchmark_1_tp; // 0D only

	// Setup the TensorCollectionGenerator
	TensorCollectionGeneratorGpu<int, float> tensor_collection_generator;
	//TensorCollectionGeneratorGpu<int, int> tensor_collection_generator; // 0D only

	// Setup the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

	// run the application
	runBenchmarkPixels(data_dir, n_dims, data_size, in_memory, shard_span_perc, benchmark_1_tp, tensor_collection_generator, device);

	return 0;
}
#endif