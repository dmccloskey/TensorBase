
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU
#include <cuda.h>
#include <cuda_runtime.h>
#include <unsupported/Eigen/CXX11/Tensor>
#include <TensorBase/benchmarks/BenchmarkPixelsGpu.h>

using namespace TensorBase;
using namespace TensorBaseBenchmarks;

/* Benchmark for toy 4D pixels data where x, y, and z describe the coordinates of the pixel in 3D space (type=int),
	t describes the time of the pixel (type=int), and the value of the pixel (from 0 to 255) describes the intensity of the pixel

Example usage:
	pixels_benchmark [data_dir] [n_dims] [data_size] [in_memory] [shard_size_perc] 
	pixels_benchmark C:/Users/dmccloskey/Documents/GitHub/mnist/ 2 1296 true 100

@param[in] n_dims The number of dimensions (i.e., 1-4) with default of 4
	1 dimension: x, y, z, and t on a single axis with a "values" dimensions on the other axis
	2 dimensions: x, y, z on a single axis, and t on another axis
	3 dimensions: y, z on a single axis, x on an axis, and t on an axis
	4 dimensions: x, y, z, and t on seperate axes
@param[in] data_size Options include small, medium, large, and XL (i.e., 1296, 1048576, 1003875856, and 1e12 pixels, respectively) with default of small
	where x, y, z, and t span 1 to 6, 32, 178, and 1000, respectively
@param[in] in_memory Simulate all data loaded into memory (true) or JIT load into memory from disk (false) with default of true
@param[in] shard_size_perc Different shard span configurations.  Options include 5, 20, and 100 with a default of 100
*/
int main(int argc, char** argv)
{
	// Parse the user commands
	std::string data_dir = "C:/Users/dmccloskey/Documents/GitHub/mnist/";
	int n_dims = 2;
	int data_size = 1296;
	bool in_memory = true;
	double shard_span_perc = 1;
	parseCmdArgs(argc, argv, data_dir, n_dims, data_size, in_memory, shard_span_perc);

	// Setup the Benchmarking suite
	Benchmark1TimePointGpu<int, float> benchmark_1_tp;
	//Benchmark1TimePointGpu<int, int> benchmark_1_tp; // 0D only

	// Setup the TensorCollectionGenerator
	TensorCollectionGeneratorGpu<int, float> tensor_collection_generator;
	//TensorCollectionGeneratorGpu<int, int> tensor_collection_generator; // 0D only

	// Setup the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

	// run the application
	runBenchmarkPixels(data_dir, n_dims, data_size, in_memory, shard_span_perc, benchmark_1_tp, tensor_collection_generator, device);

	return 0;
}
#endif