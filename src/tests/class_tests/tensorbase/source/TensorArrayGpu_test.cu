
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/


#if COMPILE_WITH_CUDA
#include <TensorBase/ml/TensorArrayGpu.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

using namespace TensorBase;
using namespace std;

/* TensorArray8 Tests
*/
void test_constructorTensorArray8Gpu() 
{
	TensorArrayGpu8<float>* ptr = nullptr;
	TensorArrayGpu8<float>* nullPointer = nullptr;
	ptr = new TensorArrayGpu8<float>();
  assert(ptr != nullPointer);
  delete ptr;
}

void test_destructorTensorArray8Gpu()
{
  TensorArrayGpu8<float>* ptr = nullptr;
	ptr = new TensorArrayGpu8<float>();
  delete ptr;
}

void test_gettersAndSettersTensorArray8Gpu()
{
  // Check same and equal length float
  Eigen::Tensor<float, 1> same_equal_float_1(8);
  same_equal_float_1.setValues({ 1,2,3,4,5,6,7,8 });
  TensorArrayGpu8<float> tensorArrayFloat1(same_equal_float_1);
  assert(tensorArrayFloat1.getArraySize() == 8);
  assert(tensorArrayFloat1.getTensorArray()(0) == 1);
  assert(tensorArrayFloat1.getTensorArray()(1) == 2);
  assert(tensorArrayFloat1.getTensorArray()(2) == 3);
  assert(tensorArrayFloat1.getTensorArray()(3) == 4);
  assert(tensorArrayFloat1.getTensorArray()(4) == 5);
  assert(tensorArrayFloat1.getTensorArray()(5) == 6);
  assert(tensorArrayFloat1.getTensorArray()(6) == 7);
  assert(tensorArrayFloat1.getTensorArray()(7) == 8);
  assert(tensorArrayFloat1.at(0) == 1);
  assert(tensorArrayFloat1.at(1) == 2);
  assert(tensorArrayFloat1.at(2) == 3);
  assert(tensorArrayFloat1.at(3) == 4);
  assert(tensorArrayFloat1.at(4) == 5);
  assert(tensorArrayFloat1.at(5) == 6);
  assert(tensorArrayFloat1.at(6) == 7);
  assert(tensorArrayFloat1.at(7) == 8);

  // Check same and equal length char
  TensorArrayGpu8<char> tensorArrayChar1({ '1','2','3','4','5','6','7','8' });
  assert(tensorArrayChar1.getArraySize() == 8);
  assert(tensorArrayChar1.getTensorArray()(0) == '1');
  assert(tensorArrayChar1.getTensorArray()(1) == '2');
  assert(tensorArrayChar1.getTensorArray()(2) == '3');
  assert(tensorArrayChar1.getTensorArray()(3) == '4');
  assert(tensorArrayChar1.getTensorArray()(4) == '5');
  assert(tensorArrayChar1.getTensorArray()(5) == '6');
  assert(tensorArrayChar1.getTensorArray()(6) == '7');
  assert(tensorArrayChar1.getTensorArray()(7) == '8');
  assert(tensorArrayChar1.at(0) == '1');
  assert(tensorArrayChar1.at(1) == '2');
  assert(tensorArrayChar1.at(2) == '3');
  assert(tensorArrayChar1.at(3) == '4');
  assert(tensorArrayChar1.at(4) == '5');
  assert(tensorArrayChar1.at(5) == '6');
  assert(tensorArrayChar1.at(6) == '7');
  assert(tensorArrayChar1.at(7) == '8');

  TensorArrayGpu8<char> tensorArrayChar2({ '1','2','3','4','5','6' });
  assert(tensorArrayChar2.getArraySize() == 8);
  assert(tensorArrayChar2.getTensorArray()(0) == '1');
  assert(tensorArrayChar2.getTensorArray()(1) == '2');
  assert(tensorArrayChar2.getTensorArray()(2) == '3');
  assert(tensorArrayChar2.getTensorArray()(3) == '4');
  assert(tensorArrayChar2.getTensorArray()(4) == '5');
  assert(tensorArrayChar2.getTensorArray()(5) == '6');
  assert(tensorArrayChar2.getTensorArray()(6) == '\0');
  assert(tensorArrayChar2.getTensorArray()(7) == '\0');
  assert(tensorArrayChar2.at(0) == '1');
  assert(tensorArrayChar2.at(1) == '2');
  assert(tensorArrayChar2.at(2) == '3');
  assert(tensorArrayChar2.at(3) == '4');
  assert(tensorArrayChar2.at(4) == '5');
  assert(tensorArrayChar2.at(5) == '6');
  assert(tensorArrayChar2.at(6) == '\0');
  assert(tensorArrayChar2.at(7) == '\0');

  // Check same and equal length char
  TensorArrayGpu8<char> tensorArrayString1("12345678");
  assert(tensorArrayString1.getArraySize() == 8);
  assert(tensorArrayString1.getTensorArray()(0) == '1');
  assert(tensorArrayString1.getTensorArray()(1) == '2');
  assert(tensorArrayString1.getTensorArray()(2) == '3');
  assert(tensorArrayString1.getTensorArray()(3) == '4');
  assert(tensorArrayString1.getTensorArray()(4) == '5');
  assert(tensorArrayString1.getTensorArray()(5) == '6');
  assert(tensorArrayString1.getTensorArray()(6) == '7');
  assert(tensorArrayString1.getTensorArray()(7) == '8');
  assert(tensorArrayString1.at(0) == '1');
  assert(tensorArrayString1.at(1) == '2');
  assert(tensorArrayString1.at(2) == '3');
  assert(tensorArrayString1.at(3) == '4');
  assert(tensorArrayString1.at(4) == '5');
  assert(tensorArrayString1.at(5) == '6');
  assert(tensorArrayString1.at(6) == '7');
  assert(tensorArrayString1.at(7) == '8');
}

void test_getTensorArrayAsStringTensorArray8Gpu()
{
  TensorArrayGpu8<int> tensorArrayInt1({ 1,2,3,4,5,6,7,8 });
  // Check << operator
  std::ostringstream os;
  os << tensorArrayInt1;
  assert(std::string(os.str()) == "12345678");

  // Check getter
  assert(tensorArrayInt1.getTensorArrayAsString() == "12345678");
}

void test_comparisonTensorArray8Gpu()
{
  // Check same and equal length float
  Eigen::Tensor<float, 1> same_equal_float_1(8);
  same_equal_float_1.setValues({ 1,2,3,4,5,6,7,8 });
  TensorArrayGpu8<float> tensorArrayFloat1(same_equal_float_1);
  Eigen::Tensor<float, 1> same_equal_float_2(8);
  same_equal_float_2.setValues({ 1,2,3,4,5,6,7,8 });
  TensorArrayGpu8<float> tensorArrayFloat2(same_equal_float_2);
  assert(tensorArrayFloat1 == tensorArrayFloat2);
  assert(!(tensorArrayFloat1 != tensorArrayFloat2));
  assert(!(tensorArrayFloat1 < tensorArrayFloat2));
  assert(!(tensorArrayFloat1 > tensorArrayFloat2));
  assert(tensorArrayFloat1 <= tensorArrayFloat2);
  assert(tensorArrayFloat1 >= tensorArrayFloat2);

  // Check different and equal length float
  Eigen::Tensor<float, 1> same_equal_float_3(8);
  same_equal_float_3.setValues({ 1,2,0,4,5,6,7,8 });
  TensorArrayGpu8<float> tensorArrayFloat3(same_equal_float_3);
  assert(!(tensorArrayFloat1 == tensorArrayFloat3));
  assert(tensorArrayFloat1 != tensorArrayFloat3);
  assert(!(tensorArrayFloat1 < tensorArrayFloat3));
  assert(tensorArrayFloat1 > tensorArrayFloat3);
  assert(!(tensorArrayFloat1 <= tensorArrayFloat3));
  assert(tensorArrayFloat1 >= tensorArrayFloat3);

  // Check same and equal length char
  Eigen::Tensor<char, 1> same_equal_char_1(8);
  same_equal_char_1.setValues({ 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar1(same_equal_char_1);
  Eigen::Tensor<char, 1> same_equal_char_2(8);
  same_equal_char_2.setValues({ 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar2(same_equal_char_2);
  assert(tensorArrayChar1 == tensorArrayChar2);
  assert(!(tensorArrayChar1 != tensorArrayChar2));
  assert(!(tensorArrayChar1 < tensorArrayChar2));
  assert(!(tensorArrayChar1 > tensorArrayChar2));
  assert(tensorArrayChar1 <= tensorArrayChar2);
  assert(tensorArrayChar1 >= tensorArrayChar2);

  // Check different and equal length char
  Eigen::Tensor<char, 1> same_equal_char_3(8);
  same_equal_char_3.setValues({ 'a', 'b', 'a', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar3(same_equal_char_3);
  assert(!(tensorArrayChar1 == tensorArrayChar3));
  assert(tensorArrayChar1 != tensorArrayChar3);
  assert(!(tensorArrayChar1 < tensorArrayChar3));
  assert(tensorArrayChar1 > tensorArrayChar3);
  assert(!(tensorArrayChar1 <= tensorArrayChar3));
  assert(tensorArrayChar1 >= tensorArrayChar3);
}

void test_tensorAssignmentTensorArray8Gpu()
{
  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // Create the string arrays
  Eigen::Tensor<char, 1> char1(8);
  char1.setValues({ 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar1(char1);
  Eigen::Tensor<char, 1> char2(8);
  char2.setValues({ 'a', 'b', 'a', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar2(char2);
  Eigen::Tensor<char, 1> char3(8);
  char3.setValues({ 'x', 'y', 'a', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar3(char3);

  // Create the Tensor array of strings
  size_t bytes = 3 * sizeof(TensorArrayGpu8<char>);
  TensorArrayGpu8<char>* h_in1;
  TensorArrayGpu8<char>* h_out1;
  TensorArrayGpu8<char>* d_in1;
  TensorArrayGpu8<char>* d_out1;
  assert(cudaHostAlloc((void**)(&h_in1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaHostAlloc((void**)(&h_out1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_in1), bytes) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_out1), bytes) == cudaSuccess);

  // Copy from the Cpu to the Gpu
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> in1(h_in1, 3);
  in1.setValues({ tensorArrayChar1 , tensorArrayChar2, tensorArrayChar3 });
  device.memcpyHostToDevice(d_in1, h_in1, bytes);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> gpu_in1(d_in1, 3);

  // Tensor copy
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> gpu_out1(d_out1, 3);
  gpu_out1.device(device) = gpu_in1;

  // Tensor compare
  device.memcpyDeviceToHost(h_out1, d_out1, bytes);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(cudaStreamDestroy(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> out1(h_out1, 3);
  assert(out1(0) == tensorArrayChar1);
  assert(out1(1) == tensorArrayChar2);
  assert(out1(2) == tensorArrayChar3);

  // Cleanup
  assert(cudaFree(d_in1) == cudaSuccess);
  assert(cudaFree(d_out1) == cudaSuccess);
  assert(cudaFreeHost(h_in1) == cudaSuccess);
  assert(cudaFreeHost(h_out1) == cudaSuccess);
}

void test_tensorComparisonTensorArray8Gpu()
{
  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // Create the string arrays
  Eigen::Tensor<char, 1> char1(8);
  char1.setValues({ 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar1(char1);
  Eigen::Tensor<char, 1> char2(8);
  char2.setValues({ 'a', 'b', 'a', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar2(char2);
  Eigen::Tensor<char, 1> char3(8);
  char3.setValues({ 'x', 'y', 'a', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar3(char3);

  // Create the Tensor array of strings
  size_t bytes = 3 * sizeof(TensorArrayGpu8<char>);
  TensorArrayGpu8<char>* h_in1;
  TensorArrayGpu8<char>* h_in2;
  TensorArrayGpu8<char>* d_in1;
  TensorArrayGpu8<char>* d_in2;
  assert(cudaHostAlloc((void**)(&h_in1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaHostAlloc((void**)(&h_in2), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_in1), bytes) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_in2), bytes) == cudaSuccess);

  // Create the selection indices
  int* h_index_1;
  int* d_index_1;
  assert(cudaHostAlloc((void**)(&h_index_1), 3 * sizeof(int), cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_index_1), 3 * sizeof(int)) == cudaSuccess);

  // Copy form Cpu to Gpu
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> in1(h_in1, 3);
  in1.setValues({ tensorArrayChar1 , tensorArrayChar2, tensorArrayChar3 });
  device.memcpyHostToDevice(d_in1, h_in1, bytes);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> gpu_in1(d_in1, 3);

  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> in2(h_in2, 3);
  in2.setValues({ tensorArrayChar2 , tensorArrayChar2, tensorArrayChar3 });
  device.memcpyHostToDevice(d_in2, h_in2, bytes);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> gpu_in2(d_in2, 3);

  Eigen::TensorMap<Eigen::Tensor<int, 1>> index1(h_index_1, 3);
  index1.setZero();
  device.memcpyHostToDevice(d_index_1, h_index_1, 3 * sizeof(int));
  Eigen::TensorMap<Eigen::Tensor<int, 1>> gpu_index1(d_index_1, 3);

  // Compare
  gpu_index1.device(device) = (gpu_in1 == gpu_in2).select(gpu_index1.constant(1), gpu_index1.constant(0));

  // Tensor compare
  device.memcpyDeviceToHost(h_index_1, d_index_1, 3 * sizeof(int));
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(cudaStreamDestroy(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<int, 1>> out1(h_index_1, 3);
  assert(out1(0) == 0);
  assert(out1(1) == 1);
  assert(out1(1) == 1);

  // Cleanup
  assert(cudaFree(d_in1) == cudaSuccess);
  assert(cudaFree(d_in2) == cudaSuccess);
  assert(cudaFreeHost(h_in1) == cudaSuccess);
  assert(cudaFreeHost(h_in2) == cudaSuccess);

  assert(cudaFree(d_index_1) == cudaSuccess);
  assert(cudaFreeHost(h_index_1) == cudaSuccess);
}

void test_tensorSortTensorArray8Gpu()
{
  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // Create the string arrays
  Eigen::Tensor<char, 1> char1(8);
  char1.setValues({ 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar1(char1);
  Eigen::Tensor<char, 1> char2(8);
  char2.setValues({ 'a', 'b', 'a', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar2(char2);
  Eigen::Tensor<char, 1> char3(8);
  char3.setValues({ 'x', 'y', 'a', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu8<char> tensorArrayChar3(char3);

  // Create the Tensor array of strings
  size_t bytes = 3 * sizeof(TensorArrayGpu8<char>);
  TensorArrayGpu8<char>* h_in1;
  TensorArrayGpu8<char>* h_out1;
  TensorArrayGpu8<char>* d_in1;
  assert(cudaHostAlloc((void**)(&h_in1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaHostAlloc((void**)(&h_out1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_in1), bytes) == cudaSuccess);

  // Copy from the Cpu to the Gpu
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> in1(h_in1, 3);
  in1.setValues({ tensorArrayChar1 , tensorArrayChar2, tensorArrayChar3 });
  device.memcpyHostToDevice(d_in1, h_in1, bytes);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> gpu_in1(d_in1, 3);

  // Thrust sort
  isLessThanGpu8 sortOp(8);
  thrust::cuda::par.on(device.stream());
  thrust::device_ptr<TensorArrayGpu8<char>> d_ptr(gpu_in1.data());
  thrust::sort(d_ptr, d_ptr + 3, sortOp);

  // Tensor compare
  device.memcpyDeviceToHost(h_out1, d_in1, bytes);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(cudaStreamDestroy(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 1>> out1(h_out1, 3);
  assert(out1(0) == tensorArrayChar2);
  assert(out1(1) == tensorArrayChar1);
  assert(out1(2) == tensorArrayChar3);

  // Cleanup
  assert(cudaFree(d_in1) == cudaSuccess);
  assert(cudaFreeHost(h_in1) == cudaSuccess);
  assert(cudaFreeHost(h_out1) == cudaSuccess);
}

/* TensorArrayGpu32 Tests
*/
void test_constructorTensorArray32Gpu()
{
  TensorArrayGpu32<float>* ptr = nullptr;
  TensorArrayGpu32<float>* nullPointer = nullptr;
  ptr = new TensorArrayGpu32<float>();
  assert(ptr != nullPointer);
  delete ptr;
}

void test_destructorTensorArray32Gpu()
{
  TensorArrayGpu32<float>* ptr = nullptr;
  ptr = new TensorArrayGpu32<float>();
  delete ptr;
}

void test_gettersAndSettersTensorArray32Gpu()
{
  // Check same and equal length float
  Eigen::Tensor<float, 1> same_equal_float_1(32);
  same_equal_float_1.setValues({ 1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32 });
  TensorArrayGpu32<float> tensorArrayFloat1(same_equal_float_1);
  assert(tensorArrayFloat1.getArraySize() == 32);
  assert(tensorArrayFloat1.getTensorArray()(0) == 1);
  assert(tensorArrayFloat1.getTensorArray()(1) == 2);
  assert(tensorArrayFloat1.getTensorArray()(2) == 3);
  assert(tensorArrayFloat1.getTensorArray()(3) == 4);
  assert(tensorArrayFloat1.getTensorArray()(4) == 5);
  assert(tensorArrayFloat1.getTensorArray()(5) == 6);
  assert(tensorArrayFloat1.getTensorArray()(6) == 7);
  assert(tensorArrayFloat1.getTensorArray()(7) == 8);
  assert(tensorArrayFloat1.getTensorArray()(8) == 9);
  assert(tensorArrayFloat1.getTensorArray()(9) == 10);
  assert(tensorArrayFloat1.getTensorArray()(10) == 11);
  assert(tensorArrayFloat1.getTensorArray()(11) == 12);
  assert(tensorArrayFloat1.getTensorArray()(12) == 13);
  assert(tensorArrayFloat1.getTensorArray()(13) == 14);
  assert(tensorArrayFloat1.getTensorArray()(14) == 15);
  assert(tensorArrayFloat1.getTensorArray()(15) == 16);
  assert(tensorArrayFloat1.getTensorArray()(16) == 17);
  assert(tensorArrayFloat1.getTensorArray()(17) == 18);
  assert(tensorArrayFloat1.getTensorArray()(18) == 19);
  assert(tensorArrayFloat1.getTensorArray()(19) == 20);
  assert(tensorArrayFloat1.getTensorArray()(20) == 21);
  assert(tensorArrayFloat1.getTensorArray()(21) == 22);
  assert(tensorArrayFloat1.getTensorArray()(22) == 23);
  assert(tensorArrayFloat1.getTensorArray()(23) == 24);
  assert(tensorArrayFloat1.getTensorArray()(24) == 25);
  assert(tensorArrayFloat1.getTensorArray()(25) == 26);
  assert(tensorArrayFloat1.getTensorArray()(26) == 27);
  assert(tensorArrayFloat1.getTensorArray()(27) == 28);
  assert(tensorArrayFloat1.getTensorArray()(28) == 29);
  assert(tensorArrayFloat1.getTensorArray()(29) == 30);
  assert(tensorArrayFloat1.getTensorArray()(30) == 31);
  assert(tensorArrayFloat1.getTensorArray()(31) == 32);
  assert(tensorArrayFloat1.at(0) == 1);
  assert(tensorArrayFloat1.at(1) == 2);
  assert(tensorArrayFloat1.at(2) == 3);
  assert(tensorArrayFloat1.at(3) == 4);
  assert(tensorArrayFloat1.at(4) == 5);
  assert(tensorArrayFloat1.at(5) == 6);
  assert(tensorArrayFloat1.at(6) == 7);
  assert(tensorArrayFloat1.at(7) == 8);
  assert(tensorArrayFloat1.at(8) == 9);
  assert(tensorArrayFloat1.at(9) == 10);
  assert(tensorArrayFloat1.at(10) == 11);
  assert(tensorArrayFloat1.at(11) == 12);
  assert(tensorArrayFloat1.at(12) == 13);
  assert(tensorArrayFloat1.at(13) == 14);
  assert(tensorArrayFloat1.at(14) == 15);
  assert(tensorArrayFloat1.at(15) == 16);
  assert(tensorArrayFloat1.at(16) == 17);
  assert(tensorArrayFloat1.at(17) == 18);
  assert(tensorArrayFloat1.at(18) == 19);
  assert(tensorArrayFloat1.at(19) == 20);
  assert(tensorArrayFloat1.at(20) == 21);
  assert(tensorArrayFloat1.at(21) == 22);
  assert(tensorArrayFloat1.at(22) == 23);
  assert(tensorArrayFloat1.at(23) == 24);
  assert(tensorArrayFloat1.at(24) == 25);
  assert(tensorArrayFloat1.at(25) == 26);
  assert(tensorArrayFloat1.at(26) == 27);
  assert(tensorArrayFloat1.at(27) == 28);
  assert(tensorArrayFloat1.at(28) == 29);
  assert(tensorArrayFloat1.at(29) == 30);
  assert(tensorArrayFloat1.at(30) == 31);
  assert(tensorArrayFloat1.at(31) == 32);

  TensorArrayGpu32<float> tensorArrayFloat2({ 1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32 });
  assert(tensorArrayFloat2.at(0) == 1);
  assert(tensorArrayFloat2.at(1) == 2);
  assert(tensorArrayFloat2.at(2) == 3);
  assert(tensorArrayFloat2.at(3) == 4);
  assert(tensorArrayFloat2.at(4) == 5);
  assert(tensorArrayFloat2.at(5) == 6);
  assert(tensorArrayFloat2.at(6) == 7);
  assert(tensorArrayFloat2.at(7) == 8);
  assert(tensorArrayFloat2.at(8) == 9);
  assert(tensorArrayFloat2.at(9) == 10);
  assert(tensorArrayFloat2.at(10) == 11);
  assert(tensorArrayFloat2.at(11) == 12);
  assert(tensorArrayFloat2.at(12) == 13);
  assert(tensorArrayFloat2.at(13) == 14);
  assert(tensorArrayFloat2.at(14) == 15);
  assert(tensorArrayFloat2.at(15) == 16);
  assert(tensorArrayFloat2.at(16) == 17);
  assert(tensorArrayFloat2.at(17) == 18);
  assert(tensorArrayFloat2.at(18) == 19);
  assert(tensorArrayFloat2.at(19) == 20);
  assert(tensorArrayFloat2.at(20) == 21);
  assert(tensorArrayFloat2.at(21) == 22);
  assert(tensorArrayFloat2.at(22) == 23);
  assert(tensorArrayFloat2.at(23) == 24);
  assert(tensorArrayFloat2.at(24) == 25);
  assert(tensorArrayFloat2.at(25) == 26);
  assert(tensorArrayFloat2.at(26) == 27);
  assert(tensorArrayFloat2.at(27) == 28);
  assert(tensorArrayFloat2.at(28) == 29);
  assert(tensorArrayFloat2.at(29) == 30);
  assert(tensorArrayFloat2.at(30) == 31);
  assert(tensorArrayFloat2.at(31) == 32);
}

void test_comparisonTensorArray32Gpu()
{
  // Check same and equal length float
  Eigen::Tensor<float, 1> same_equal_float_1(32);
  same_equal_float_1.setValues({ 1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32 });
  TensorArrayGpu32<float> tensorArrayFloat1(same_equal_float_1);
  Eigen::Tensor<float, 1> same_equal_float_2(32);
  same_equal_float_2.setValues({ 1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32 });
  TensorArrayGpu32<float> tensorArrayFloat2(same_equal_float_2);
  assert(tensorArrayFloat1 == tensorArrayFloat2);
  assert(!(tensorArrayFloat1 != tensorArrayFloat2));
  assert(!(tensorArrayFloat1 < tensorArrayFloat2));
  assert(!(tensorArrayFloat1 > tensorArrayFloat2));
  assert(tensorArrayFloat1 <= tensorArrayFloat2);
  assert(tensorArrayFloat1 >= tensorArrayFloat2);

  // Check different and equal length float
  Eigen::Tensor<float, 1> same_equal_float_3(32);
  same_equal_float_3.setValues({ 1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,0,28,29,30,31,32 });
  TensorArrayGpu32<float> tensorArrayFloat3(same_equal_float_3);
  assert(!(tensorArrayFloat1 == tensorArrayFloat3));
  assert(tensorArrayFloat1 != tensorArrayFloat3);
  assert(!(tensorArrayFloat1 < tensorArrayFloat3));
  assert(tensorArrayFloat1 > tensorArrayFloat3);
  assert(!(tensorArrayFloat1 <= tensorArrayFloat3));
  assert(tensorArrayFloat1 >= tensorArrayFloat3);
}

void test_getTensorArrayAsStringTensorArray32Gpu()
{
  TensorArrayGpu32<int> tensorArrayInt1({ 1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32 });
  // Check << operator
  std::ostringstream os;
  os << tensorArrayInt1;
  assert(std::string(os.str()) == "1234567891011121314151617181920212223242526272829303132");

  // Check getter
  assert(tensorArrayInt1.getTensorArrayAsString() == "1234567891011121314151617181920212223242526272829303132");
}
void test_tensorAssignmentTensorArray32Gpu()
{
  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // Create the string arrays
  Eigen::Tensor<char, 1> char1(32);
  char1.setValues({ 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu32<char> tensorArrayChar1(char1);
  Eigen::Tensor<char, 1> char2(32);
  char2.setValues({ 'a', 'b', 'a', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu32<char> tensorArrayChar2(char2);
  Eigen::Tensor<char, 1> char3(32);
  char3.setValues({ 'x', 'y', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu32<char> tensorArrayChar3(char3);

  // Create the Tensor array of strings
  size_t bytes = 3 * sizeof(TensorArrayGpu32<char>);
  TensorArrayGpu32<char>* h_in1;
  TensorArrayGpu32<char>* h_out1;
  TensorArrayGpu32<char>* d_in1;
  TensorArrayGpu32<char>* d_out1;
  assert(cudaHostAlloc((void**)(&h_in1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaHostAlloc((void**)(&h_out1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_in1), bytes) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_out1), bytes) == cudaSuccess);

  // Copy from the Cpu to the Gpu
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> in1(h_in1, 3);
  in1.setValues({ tensorArrayChar1 , tensorArrayChar2, tensorArrayChar3 });
  device.memcpyHostToDevice(d_in1, h_in1, bytes);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> gpu_in1(d_in1, 3);

  // Tensor copy
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> gpu_out1(d_out1, 3);
  gpu_out1.device(device) = gpu_in1;

  // Tensor compare
  device.memcpyDeviceToHost(h_out1, d_out1, bytes);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(cudaStreamDestroy(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> out1(h_out1, 3);
  assert(out1(0) == tensorArrayChar1);
  assert(out1(1) == tensorArrayChar2);
  assert(out1(2) == tensorArrayChar3);

  // Cleanup
  assert(cudaFree(d_in1) == cudaSuccess);
  assert(cudaFree(d_out1) == cudaSuccess);
  assert(cudaFreeHost(h_in1) == cudaSuccess);
  assert(cudaFreeHost(h_out1) == cudaSuccess);
}

void test_tensorComparisonTensorArray32Gpu()
{
  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // Create the string arrays
  Eigen::Tensor<char, 1> char1(32);
  char1.setValues({ 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu32<char> tensorArrayChar1(char1);
  Eigen::Tensor<char, 1> char2(32);
  char2.setValues({ 'a', 'b', 'a', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu32<char> tensorArrayChar2(char2);
  Eigen::Tensor<char, 1> char3(32);
  char3.setValues({ 'x', 'y', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu32<char> tensorArrayChar3(char3);

  // Create the Tensor array of strings
  size_t bytes = 3 * sizeof(TensorArrayGpu32<char>);
  TensorArrayGpu32<char>* h_in1;
  TensorArrayGpu32<char>* h_in2;
  TensorArrayGpu32<char>* d_in1;
  TensorArrayGpu32<char>* d_in2;
  assert(cudaHostAlloc((void**)(&h_in1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaHostAlloc((void**)(&h_in2), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_in1), bytes) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_in2), bytes) == cudaSuccess);

  // Create the selection indices
  int* h_index_1;
  int* d_index_1;
  assert(cudaHostAlloc((void**)(&h_index_1), 3 * sizeof(int), cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_index_1), 3 * sizeof(int)) == cudaSuccess);

  // Copy form Cpu to Gpu
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> in1(h_in1, 3);
  in1.setValues({ tensorArrayChar1 , tensorArrayChar2, tensorArrayChar3 });
  device.memcpyHostToDevice(d_in1, h_in1, bytes);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> gpu_in1(d_in1, 3);

  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> in2(h_in2, 3);
  in2.setValues({ tensorArrayChar2 , tensorArrayChar2, tensorArrayChar3 });
  device.memcpyHostToDevice(d_in2, h_in2, bytes);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> gpu_in2(d_in2, 3);

  Eigen::TensorMap<Eigen::Tensor<int, 1>> index1(h_index_1, 3);
  index1.setZero();
  device.memcpyHostToDevice(d_index_1, h_index_1, 3 * sizeof(int));
  Eigen::TensorMap<Eigen::Tensor<int, 1>> gpu_index1(d_index_1, 3);

  // Compare
  gpu_index1.device(device) = (gpu_in1 == gpu_in2).select(gpu_index1.constant(1), gpu_index1.constant(0));

  // Tensor compare
  device.memcpyDeviceToHost(h_index_1, d_index_1, 3 * sizeof(int));
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(cudaStreamDestroy(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<int, 1>> out1(h_index_1, 3);
  assert(out1(0) == 0);
  assert(out1(1) == 1);
  assert(out1(1) == 1);

  // Cleanup
  assert(cudaFree(d_in1) == cudaSuccess);
  assert(cudaFree(d_in2) == cudaSuccess);
  assert(cudaFreeHost(h_in1) == cudaSuccess);
  assert(cudaFreeHost(h_in2) == cudaSuccess);

  assert(cudaFree(d_index_1) == cudaSuccess);
  assert(cudaFreeHost(h_index_1) == cudaSuccess);
}

void test_tensorSortTensorArray32Gpu()
{
  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // Create the string arrays
  Eigen::Tensor<char, 1> char1(32);
  char1.setValues({ 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu32<char> tensorArrayChar1(char1);
  Eigen::Tensor<char, 1> char2(32);
  char2.setValues({ 'a', 'b', 'a', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu32<char> tensorArrayChar2(char2);
  Eigen::Tensor<char, 1> char3(32);
  char3.setValues({ 'x', 'y', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'e', 'a', 'b', 'c', 'd', 'e', 'f', 'g', '\0' });
  TensorArrayGpu32<char> tensorArrayChar3(char3);

  // Create the Tensor array of strings
  size_t bytes = 3 * sizeof(TensorArrayGpu32<char>);
  TensorArrayGpu32<char>* h_in1;
  TensorArrayGpu32<char>* h_out1;
  TensorArrayGpu32<char>* d_in1;
  assert(cudaHostAlloc((void**)(&h_in1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaHostAlloc((void**)(&h_out1), bytes, cudaHostAllocDefault) == cudaSuccess);
  assert(cudaMalloc((void**)(&d_in1), bytes) == cudaSuccess);

  // Copy from the Cpu to the Gpu
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> in1(h_in1, 3);
  in1.setValues({ tensorArrayChar1 , tensorArrayChar2, tensorArrayChar3 });
  device.memcpyHostToDevice(d_in1, h_in1, bytes);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> gpu_in1(d_in1, 3);

  // Thrust sort
  isLessThanGpu32 sortOp(32);
  thrust::cuda::par.on(device.stream());
  thrust::device_ptr<TensorArrayGpu32<char>> d_ptr(gpu_in1.data());
  thrust::sort(d_ptr, d_ptr + 3, sortOp);

  // Tensor compare
  device.memcpyDeviceToHost(h_out1, d_in1, bytes);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(cudaStreamDestroy(stream) == cudaSuccess);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu32<char>, 1>> out1(h_out1, 3);
  assert(out1(0) == tensorArrayChar2);
  assert(out1(1) == tensorArrayChar1);
  assert(out1(2) == tensorArrayChar3);

  // Cleanup
  assert(cudaFree(d_in1) == cudaSuccess);
  assert(cudaFreeHost(h_in1) == cudaSuccess);
  assert(cudaFreeHost(h_out1) == cudaSuccess);
}

int main(int argc, char** argv)
{
  test_constructorTensorArray8Gpu();
  test_destructorTensorArray8Gpu();
  test_gettersAndSettersTensorArray8Gpu();
  test_getTensorArrayAsStringTensorArray8Gpu();
  test_comparisonTensorArray8Gpu();
  test_tensorAssignmentTensorArray8Gpu();
  test_tensorComparisonTensorArray8Gpu();
  test_tensorSortTensorArray8Gpu();

  test_constructorTensorArray32Gpu();
  test_destructorTensorArray32Gpu();
  test_gettersAndSettersTensorArray32Gpu();
  test_comparisonTensorArray32Gpu();
  test_getTensorArrayAsStringTensorArray32Gpu();
  test_tensorAssignmentTensorArray32Gpu();
  test_tensorComparisonTensorArray32Gpu();
  test_tensorSortTensorArray32Gpu();
}
#endif