
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA
#include <TensorBase/ml/TensorDimensionGpu.h>
#include <TensorBase/ml/TensorArrayGpu.h>

using namespace TensorBase;
using namespace std;

/*TensorDimensionGpuPrimitiveT Tests*/
void test_constructorGpuPrimitiveT()
{
  TensorDimensionGpuPrimitiveT<int>* ptr = nullptr;
  TensorDimensionGpuPrimitiveT<int>* nullPointer = nullptr;
  ptr = new TensorDimensionGpuPrimitiveT<int>();
  assert(ptr != nullPointer);
  delete ptr;
}

void test_destructorGpuPrimitiveT()
{
  TensorDimensionGpuPrimitiveT<int>* ptr = nullptr;
  ptr = new TensorDimensionGpuPrimitiveT<int>();
  delete ptr;
}

void test_constructorNameGpuPrimitiveT()
{
  TensorDimensionGpuPrimitiveT<int> tensordimension("1", "dir");
  assert(tensordimension.getId() == -1);
  assert(tensordimension.getName() == "1");
  assert(tensordimension.getDir() == "dir");
  assert(tensordimension.getNLabels() == 0);
}

void test_constructorNameAndLabelsGpuPrimitiveT()
{
  Eigen::Tensor<int, 1> labels(5);
  labels.setConstant(1);
  TensorDimensionGpuPrimitiveT<int> tensordimension("1", "dir", labels);
  assert(tensordimension.getName() == "1");
  assert(tensordimension.getDir() == "dir");
  assert(tensordimension.getNLabels() == 5);
  assert(tensordimension.getLabels()(0) == 1);
  assert(tensordimension.getLabels()(4) == 1);
}

void test_gettersAndSettersGpuPrimitiveT()
{
  TensorDimensionGpuPrimitiveT<int> tensordimension;
  // Check defaults
  assert(tensordimension.getId() == -1);
  assert(tensordimension.getName() == "");
  assert(tensordimension.getDir() == "");
  assert(tensordimension.getNLabels() == 0);

  // Check getters/setters
  tensordimension.setId(1);
  tensordimension.setName("1");
  tensordimension.setDir("dir");
  Eigen::Tensor<int, 1> labels(5);
  labels.setConstant(1);
  tensordimension.setLabels(labels);

  assert(tensordimension.getId() == 1);
  assert(tensordimension.getName() == "1");
  assert(tensordimension.getDir() == "dir");
  assert(tensordimension.getNLabels() == 5);
  assert(tensordimension.getLabels()(0) == 1);
  assert(tensordimension.getLabels()(4) == 1);
}

/*TensorDimensionGpuClassT Tests*/
void test_constructorGpuClassT()
{
  TensorDimensionGpuClassT<TensorArrayGpu8, int>* ptr = nullptr;
  TensorDimensionGpuClassT<TensorArrayGpu8, int>* nullPointer = nullptr;
  ptr = new TensorDimensionGpuClassT<TensorArrayGpu8, int>();
  assert(ptr != nullPointer);
  delete ptr;
}

void test_destructorGpuClassT()
{
  TensorDimensionGpuClassT<TensorArrayGpu8, int>* ptr = nullptr;
  ptr = new TensorDimensionGpuClassT<TensorArrayGpu8, int>();
  delete ptr;
}

void test_constructorNameGpuClassT()
{
  TensorDimensionGpuClassT<TensorArrayGpu8, int> tensordimension("1", "dir");
  assert(tensordimension.getId() == -1);
  assert(tensordimension.getName() == "1");
  assert(tensordimension.getDir() == "dir");
  assert(tensordimension.getNLabels() == 0);
}

void test_constructorNameAndLabelsGpuClassT()
{
  Eigen::Tensor<TensorArrayGpu8<int>, 1> labels(5);
  labels.setConstant(TensorArrayGpu8<int>({1, 1, 1, 1, 1, 1, 1, 1}));
  TensorDimensionGpuClassT<TensorArrayGpu8, int> tensordimension("1", "dir", labels);
  assert(tensordimension.getName() == "1");
  assert(tensordimension.getDir() == "dir");
  assert(tensordimension.getNLabels() == 5);
  assert(tensordimension.getLabels()(0).getTensorArray()(0) == 1);
  assert(tensordimension.getLabels()(4).getTensorArray()(0) == 1);
}

void test_gettersAndSettersGpuClassT()
{
  TensorDimensionGpuClassT<TensorArrayGpu8, int> tensordimension;
  // Check defaults
  assert(tensordimension.getId() == -1);
  assert(tensordimension.getName() == "");
  assert(tensordimension.getDir() == "");
  assert(tensordimension.getNLabels() == 0);

  // Check getters/setters
  tensordimension.setId(1);
  tensordimension.setName("1");
  tensordimension.setDir("dir");
  Eigen::Tensor<TensorArrayGpu8<int>, 1> labels(5);
  labels.setConstant(TensorArrayGpu8<int>({ 1, 1, 1, 1, 1, 1, 1, 1 }));
  tensordimension.setLabels(labels);

  assert(tensordimension.getId() == 1);
  assert(tensordimension.getName() == "1");
  assert(tensordimension.getDir() == "dir");
  assert(tensordimension.getNLabels() == 5);
  assert(tensordimension.getLabels()(0).getTensorArray()(0) == 1);
  assert(tensordimension.getLabels()(4).getTensorArray()(0) == 1);
}

int main(int argc, char** argv)
{
  test_constructorGpuPrimitiveT();
  test_destructorGpuPrimitiveT();
  test_constructorNameGpuPrimitiveT();
  test_constructorNameAndLabelsGpuPrimitiveT();
  test_gettersAndSettersGpuPrimitiveT();

  test_constructorGpuClassT();
  test_destructorGpuClassT();
  test_constructorNameGpuClassT();
  test_constructorNameAndLabelsGpuClassT();
  test_gettersAndSettersGpuClassT();
  return 0;
}
#endif