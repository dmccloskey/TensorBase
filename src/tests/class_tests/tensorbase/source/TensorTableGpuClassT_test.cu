
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA
#include <TensorBase/ml/TensorTableGpuClassT.h>
#include <string>

using namespace TensorBase;
using namespace std;

void test_constructorGpu()
{
  TensorTableGpuClassT<TensorArrayGpu8, char, 3>* ptr = nullptr;
  TensorTableGpuClassT<TensorArrayGpu8, char, 3>* nullPointer = nullptr;
  ptr = new TensorTableGpuClassT<TensorArrayGpu8, char, 3>();
  gpuCheckNotEqual(ptr, nullPointer);
  delete ptr;
}

void test_destructorGpu()
{
  TensorTableGpuClassT<TensorArrayGpu8, char, 3>* ptr = nullptr;
  ptr = new TensorTableGpuClassT<TensorArrayGpu8, char, 3>();
  delete ptr;
}

void test_constructorNameAndAxesGpu()
{
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable("1");

  gpuCheckEqual(tensorTable.getId(), -1);
  gpuCheckEqual(tensorTable.getName(), "1");
  gpuCheckEqual(tensorTable.getDir(), "");

  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable2("1", "dir");

  gpuCheckEqual(tensorTable2.getId(), -1);
  gpuCheckEqual(tensorTable2.getName(), "1");
  gpuCheckEqual(tensorTable2.getDir(), "dir");
}

void test_gettersAndSettersGpu()
{
  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;
  // Check defaults
  gpuCheckEqual(tensorTable.getId(), -1);
  gpuCheckEqual(tensorTable.getName(), "");
  gpuCheckEqual(tensorTable.getAxes().size(), 0);
  gpuCheckEqual(tensorTable.getDir(), "");
  gpuCheckEqual(tensorTable.getTensorSize(), 0);

  // Check getters/setters
  tensorTable.setId(1);
  tensorTable.setName("1");
  tensorTable.setDir("dir");

  gpuCheckEqual(tensorTable.getId(), 1);
  gpuCheckEqual(tensorTable.getName(), "1");
  gpuCheckEqual(tensorTable.getDir(), "dir");

  // SetAxes associated getters/setters
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels1 = 2, nlabels2 = 3, nlabels3 = 5;
  Eigen::Tensor<int, 2> labels1(1, nlabels1), labels2(1, nlabels2), labels3(1, nlabels3);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  //Eigen::Tensor<std::string, 2> labels1(1, nlabels1), labels2(1, nlabels2), labels3(1, nlabels3);
  //labels1.setConstant("x-axis");
  //labels2.setConstant("y-axis");
  //labels3.setConstant("z-axis");
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // Test expected axes values
  gpuCheckEqual(tensorTable.getAxes().at("1")->getName(), "1");
  //gpuCheckEqual(tensorTable.getAxes().at("1")->getLabels()(0, 0), 1);
  ////gpuCheckEqual(tensorTable.getAxes().at("1")->getLabels()(0,0), "x-axis");
  gpuCheckEqual(tensorTable.getAxes().at("1")->getNLabels(), nlabels1);
  gpuCheckEqual(tensorTable.getAxes().at("1")->getNDimensions(), 1);
  gpuCheckEqual(tensorTable.getAxes().at("1")->getDimensions()(0), "x");
  gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(nlabels1 - 1), nlabels1);
  gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(nlabels1 - 1), nlabels1);
  gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(0), 0);
  gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(nlabels1 - 1), nlabels1);

  gpuCheckEqual(tensorTable.getAxes().at("2")->getName(), "2");
  //gpuCheckEqual(tensorTable.getAxes().at("2")->getLabels()(0, 0), 2);
  ////gpuCheckEqual(tensorTable.getAxes().at("2")->getLabels()(0, 0), "y-axis");
  gpuCheckEqual(tensorTable.getAxes().at("2")->getNLabels(), nlabels2);
  gpuCheckEqual(tensorTable.getAxes().at("2")->getNDimensions(), 1);
  gpuCheckEqual(tensorTable.getAxes().at("2")->getDimensions()(0), "y");
  gpuCheckEqual(tensorTable.getIndices().at("2")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIndices().at("2")->getData()(nlabels2 - 1), nlabels2);
  gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(nlabels2 - 1), nlabels2);
  gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(0), 0);
  gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardId().at("2")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardIndices().at("2")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardIndices().at("2")->getData()(nlabels2 - 1), nlabels2);

  gpuCheckEqual(tensorTable.getAxes().at("3")->getName(), "3");
  //gpuCheckEqual(tensorTable.getAxes().at("3")->getLabels()(0, 0), 3);
  ////gpuCheckEqual(tensorTable.getAxes().at("3")->getLabels()(0, 0), "z-axis");
  gpuCheckEqual(tensorTable.getAxes().at("3")->getNLabels(), nlabels3);
  gpuCheckEqual(tensorTable.getAxes().at("3")->getNDimensions(), 1);
  gpuCheckEqual(tensorTable.getAxes().at("3")->getDimensions()(0), "z");
  gpuCheckEqual(tensorTable.getIndices().at("3")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIndices().at("3")->getData()(nlabels3 - 1), nlabels3);
  gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(nlabels3 - 1), nlabels3);
  gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(0), 0);
  gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardId().at("3")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardIndices().at("3")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardIndices().at("3")->getData()(nlabels3 - 1), nlabels3);

  // Test expected axis to dims mapping
  gpuCheckEqual(tensorTable.getDimFromAxisName("1"), 0);
  gpuCheckEqual(tensorTable.getDimFromAxisName("2"), 1);
  gpuCheckEqual(tensorTable.getDimFromAxisName("3"), 2);

  // Test expected tensor shard spans
  gpuCheckEqual(tensorTable.getShardSpans().at("1"), 2);
  gpuCheckEqual(tensorTable.getShardSpans().at("2"), 3);
  gpuCheckEqual(tensorTable.getShardSpans().at("3"), 5);

  // Test expected tensor dimensions
  gpuCheckEqual(tensorTable.getDimensions().at(0), 2);
  gpuCheckEqual(tensorTable.getDimensions().at(1), 3);
  gpuCheckEqual(tensorTable.getDimensions().at(2), 5);
  gpuCheckEqual(tensorTable.getTensorSize(), 30);

  // Test expected maximum dimensions
  gpuCheckEqual(tensorTable.getMaximumDimensions().at(0), 36);
  gpuCheckEqual(tensorTable.getMaximumDimensions().at(1), 54);
  gpuCheckEqual(tensorTable.getMaximumDimensions().at(2), 90);

  // Test expected tensor data values
  gpuCheckEqual(tensorTable.getDataDimensions().at(0), 2);
  gpuCheckEqual(tensorTable.getDataDimensions().at(1), 3);
  gpuCheckEqual(tensorTable.getDataDimensions().at(2), 5);
  size_t test = 2 * 3 * 5 * sizeof(TensorArrayGpu8<char>);
  gpuCheckEqual(tensorTable.getDataTensorBytes(), test);

  // Test setting the data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_data(Eigen::array<Eigen::Index, 3>({ nlabels1, nlabels2, nlabels3 }));
  for (int i = 0; i < nlabels1; ++i) {
    for (int j = 0; j < nlabels2; ++j) {
      for (int k = 0; k < nlabels3; ++k) {
        tensor_data(i, j, k).setTensorArray(std::to_string(i + j + k));
      }
    }
  }
  tensorTable.setData(tensor_data);
  for (int i = 0; i < nlabels1; ++i) {
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
  }
  for (int i = 0; i < nlabels2; ++i) {
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
  }
  for (int i = 0; i < nlabels3; ++i) {
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
  }

  // Test setting the data
  tensorTable.setData();
  for (int i = 0; i < nlabels1; ++i) {
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 1);
  }
  for (int i = 0; i < nlabels2; ++i) {
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 1);
  }
  for (int i = 0; i < nlabels3; ++i) {
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 1);
  }

  // Test clear
  tensorTable.clear();
  gpuCheckEqual(tensorTable.getAxes().size(), 0);
  gpuCheckEqual(tensorTable.getIndices().size(), 0);
  gpuCheckEqual(tensorTable.getIndicesView().size(), 0);
  gpuCheckEqual(tensorTable.getIsModified().size(), 0);
  gpuCheckEqual(tensorTable.getNotInMemory().size(), 0);
  gpuCheckEqual(tensorTable.getShardId().size(), 0);
  gpuCheckEqual(tensorTable.getShardIndices().size(), 0);
  gpuCheckEqual(tensorTable.getDimensions().at(0), 0);
  gpuCheckEqual(tensorTable.getDimensions().at(1), 0);
  gpuCheckEqual(tensorTable.getDimensions().at(2), 0);
  gpuCheckEqual(tensorTable.getShardSpans().size(), 0);
}

void test_initDataGpuClassT()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // Check the dimensions and expected not_in_memory values
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 1);
  }
  gpuCheckEqual(tensorTable.getDataTensorSize(), nlabels * nlabels*nlabels);

  // Reset the not_in_memory to false
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    in_memory_map.second->getData() = in_memory_map.second->getData().constant(0);
  }
  tensorTable.syncNotInMemoryDData(device);

  // Resize the tensor data
  Eigen::array<Eigen::Index, 3> new_dimensions = { 2, 2, 2 };
  tensorTable.initData(new_dimensions, device);

  // Check the dimensions and expected not_in_memory values
  tensorTable.syncNotInMemoryHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 1);
  }
  gpuCheckEqual(tensorTable.getDataTensorSize(), 8);

  // Reset the not_in_memory to false
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    in_memory_map.second->getData() = in_memory_map.second->getData().constant(0);
  }
  tensorTable.syncNotInMemoryDData(device);

  // Resize the tensor data to 0
  tensorTable.initData(device);

  // Check the dimensions and expected not_in_memory values
  tensorTable.syncNotInMemoryHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 1);
  }
  gpuCheckEqual(tensorTable.getDataTensorSize(), 0);
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_reShardIndicesGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2, 3} });
  labels2.setValues({ {0, 1, 2, 3} });
  labels3.setValues({ {0, 1, 2, 3} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // Test the default shard span
  gpuCheckEqual(tensorTable.getShardSpans().at("1"), 4);
  gpuCheckEqual(tensorTable.getShardSpans().at("2"), 4);
  gpuCheckEqual(tensorTable.getShardSpans().at("3"), 4);

  // Reset the shard span
  int shard_span = 3;
  std::map<std::string, int> shard_span_new = { {"1", shard_span}, {"2", shard_span}, {"3", shard_span} };
  tensorTable.setShardSpans(shard_span_new);
  gpuCheckEqual(tensorTable.getShardSpans().at("1"), 3);
  gpuCheckEqual(tensorTable.getShardSpans().at("2"), 3);
  gpuCheckEqual(tensorTable.getShardSpans().at("3"), 3);
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.reShardIndices(device);
  tensorTable.syncAxesAndIndicesHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i < shard_span) {
      gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getShardId().at("2")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getShardIndices().at("2")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getShardId().at("3")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getShardIndices().at("3")->getData()(i), i + 1);
    }
    else {
      gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(i), 2);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), i - shard_span + 1);
      gpuCheckEqual(tensorTable.getShardId().at("2")->getData()(i), 2);
      gpuCheckEqual(tensorTable.getShardIndices().at("2")->getData()(i), i - shard_span + 1);
      gpuCheckEqual(tensorTable.getShardId().at("3")->getData()(i), 2);
      gpuCheckEqual(tensorTable.getShardIndices().at("3")->getData()(i), i - shard_span + 1);
    }
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_zeroIndicesViewAndResetIndicesViewGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // sync the tensorTable indices
  tensorTable.syncIndicesDData(device);
  tensorTable.syncIndicesViewDData(device);

  // test null
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
  }

  // test zero
  tensorTable.zeroIndicesView("1", device);
  tensorTable.getIndicesView().at("1")->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), 0);
  }

  // test reset
  tensorTable.getIndicesView().at("1")->setDataStatus(false, true);
  tensorTable.resetIndicesView("1", device);
  tensorTable.getIndicesView().at("1")->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
  }

  // test zero
  tensorTable.getIndicesView().at("1")->setDataStatus(false, true);
  tensorTable.zeroIndicesView("1", device);
  tensorTable.getIndicesView().at("1")->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), 0);
  }

  // test replace
  tensorTable.getIndicesView().at("1")->setDataStatus(false, true);
  tensorTable.replaceIndicesView("1", tensorTable.getIndicesView().at("2"), device);
  tensorTable.getIndicesView().at("1")->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_selectIndicesView1Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2, 3} });
  labels2.setValues({ {0, 1, 2, 3} });
  labels3.setValues({ {0, 1, 2, 3} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // set up the selection labels
  Eigen::Tensor<int, 1> select_labels_values(nlabels / 2);
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    if (i % 2 == 0) {
      select_labels_values(iter) = i;
      ++iter;
    }
  }
  TensorDataGpuPrimitiveT<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ nlabels / 2 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(select_labels);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // test the updated view
  select_labels_ptr->syncDData(device);
  tensorTable.selectIndicesView("1", 0, select_labels_ptr, device);
  tensorTable.syncIndicesViewHData(device);
  select_labels_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i % 2 == 0)
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), 0);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}
void test_selectIndicesView2Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(2), dimensions2(1), dimensions3(1);
  dimensions1(0) = "a"; dimensions1(1) = "b";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(2, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2, 3}, {4, 5, 6, 7} });
  labels2.setValues({ {0, 1, 2, 3} });
  labels3.setValues({ {0, 1, 2, 3} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // set up the selection labels
  Eigen::Tensor<int, 2> select_labels_values(2, 2);
  select_labels_values.setValues({ {0, 2}, {4, 6} });
  TensorDataGpuPrimitiveT<int, 2> select_labels(Eigen::array<Eigen::Index, 2>({ 2, 2 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 2>>(select_labels);

  // test the updated view
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  select_labels_ptr->syncDData(device);
  tensorTable.selectIndicesView("1", select_labels_ptr, device);
  tensorTable.getIndicesView().at("1")->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i % 2 == 0)
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), 0);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_broadcastSelectIndicesViewGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // sync the tensorTable indices
  tensorTable.syncAxesAndIndicesDData(device);

  // setup the indices test
  Eigen::Tensor<int, 3> indices_test(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        indices_test(i, j, k) = i + 1;
      }
    }
  }

  // test the broadcast indices values
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_view_bcast;
  tensorTable.broadcastSelectIndicesView(indices_view_bcast, "1", device);
  indices_view_bcast->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        //std::cout << "Test broadcastSelectIndicesView i,j,k :" << i << "," << j << "," << k << "; Labels: " << indices_view_bcast->getData()(i, j, k) << "; Expected: " << indices_test(i, j, k) << std::endl;
        gpuCheckEqual(indices_view_bcast->getData()(i, j, k), indices_test(i, j, k));
      }
    }
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_extractTensorDataGpuClassT()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // setup the tensor data, selection indices, and test selection data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<int, 3> indices_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_test(Eigen::array<Eigen::Index, 3>({ nlabels / 2, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        int value = i * nlabels + j * nlabels + k;
        tensor_values(i, j, k).setTensorArray(std::to_string(value));
        if (i % 2 == 0) {
          indices_values(i, j, k) = 1;
          tensor_test(i / 2, j, k).setTensorArray(std::to_string(value));
        }
        else {
          indices_values(i, j, k) = 0;
        }
      }
    }
  }
  tensorTable.setData(tensor_values);
  TensorDataGpuPrimitiveT<int, 3> indices_select(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  indices_select.setData(indices_values);
  auto indices_select_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 3>>(indices_select);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // test
  indices_select_ptr->syncDData(device);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> tensor_select;
  tensorTable.reduceTensorDataToSelectIndices(indices_select_ptr,
    tensor_select, "1", nlabels / 2, device);
  tensor_select->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels / 2; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(tensor_select->getData()(i, j, k), tensor_test(i, j, k));
      }
    }
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_selectTensorIndicesGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 2;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // setup the tensor select and values select data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_select_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<TensorArrayGpu8<char>, 1> values_select_values(Eigen::array<Eigen::Index, 1>({ nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    values_select_values(i).setTensorArray("2");
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_select_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> tensor_select(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  tensor_select.setData(tensor_select_values);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> tensor_select_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(tensor_select);
  TensorDataGpuClassT<TensorArrayGpu8, char, 1> values_select(Eigen::array<Eigen::Index, 1>({ nlabels }));
  values_select.setData(values_select_values);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 1>> values_select_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 1>>(values_select);

  // Sync the data
  tensor_select_ptr->syncDData(device);
  values_select_ptr->syncDData(device);
  tensorTable.syncAxesAndIndicesDData(device);

  // test inequality
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_select;
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::NOT_EQUAL_TO, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) == TensorArrayGpu8<char>("2"))
          gpuCheckEqual(indices_select->getData()(i, j, k), 0);
        else
          gpuCheckEqual(indices_select->getData()(i, j, k), 1);
      }
    }
  }

  // test equality
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::EQUAL_TO, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) == TensorArrayGpu8<char>("2"))
          gpuCheckEqual(indices_select->getData()(i, j, k), 1);
        else
          gpuCheckEqual(indices_select->getData()(i, j, k), 0);
      }
    }
  }

  // test less than
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::LESS_THAN, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) < TensorArrayGpu8<char>("2"))
          gpuCheckEqual(indices_select->getData()(i, j, k), 1);
        else
          gpuCheckEqual(indices_select->getData()(i, j, k), 0);
      }
    }
  }

  // test less than or equal to
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::LESS_THAN_OR_EQUAL_TO, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) <= TensorArrayGpu8<char>("2"))
          gpuCheckEqual(indices_select->getData()(i, j, k), 1);
        else
          gpuCheckEqual(indices_select->getData()(i, j, k), 0);
      }
    }
  }

  // test greater than
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::GREATER_THAN, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) > TensorArrayGpu8<char>("2"))
          gpuCheckEqual(indices_select->getData()(i, j, k), 1);
        else
          gpuCheckEqual(indices_select->getData()(i, j, k), 0);
      }
    }
  }

  // test greater than or equal to
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::GREATER_THAN_OR_EQUAL_TO, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) >= TensorArrayGpu8<char>("2"))
          gpuCheckEqual(indices_select->getData()(i, j, k), 1);
        else
          gpuCheckEqual(indices_select->getData()(i, j, k), 0);
      }
    }
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_applyIndicesSelectToIndicesViewGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // setup the indices select
  Eigen::Tensor<int, 3> indices_select_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i == j && j == k && k == i
          && i < nlabels - 1 && j < nlabels - 1 && k < nlabels - 1) // the first 2 diagonal elements
          indices_select_values(i, j, k) = 1;
        else
          indices_select_values(i, j, k) = 0;
      }
    }
  }
  TensorDataGpuPrimitiveT<int, 3> indices_select(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  indices_select.setData(indices_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_select_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 3>>(indices_select);
  indices_select_ptr->syncDData(device);

  // test using the second indices view
  tensorTable.getIndicesView().at("2")->getData()(nlabels - 1) = 0;
  tensorTable.setIndicesViewDataStatus(true, false);
  tensorTable.syncIndicesViewDData(device);

  // test for OR within continuator and OR prepend continuator
  tensorTable.applyIndicesSelectToIndicesView(indices_select_ptr, "1", "2", logicalContinuators::logicalContinuator::OR, logicalContinuators::logicalContinuator::OR, device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i == nlabels - 1)
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
  }

  // reset and modify the indices view
  tensorTable.syncIndicesViewDData(device);
  tensorTable.resetIndicesView("2", device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  tensorTable.getIndicesView().at("2")->getData()(0) = 0;
  tensorTable.syncIndicesViewDData(device);

  // test for AND within continuator and OR prepend continuator
  tensorTable.applyIndicesSelectToIndicesView(indices_select_ptr, "1", "2", logicalContinuators::logicalContinuator::AND, logicalContinuators::logicalContinuator::OR, device);  
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i == 0)
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
  }

  // Reset and modify the indices view
  tensorTable.syncIndicesViewDData(device);
  tensorTable.resetIndicesView("2", device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  tensorTable.getIndicesView().at("2")->getData()(0) = 0;
  tensorTable.syncIndicesViewDData(device);

  // test for OR within continuator and AND prepend continuator
  tensorTable.applyIndicesSelectToIndicesView(indices_select_ptr, "1", "2", logicalContinuators::logicalContinuator::OR, logicalContinuators::logicalContinuator::AND, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i != 0 && i < nlabels - 1)
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);
  }

  // Reset the indices view
  tensorTable.syncIndicesViewDData(device);
  tensorTable.resetIndicesView("2", device);

  // and update the indices_select_ptr
  indices_select_ptr->setDataStatus(true, false);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i == j && j == k && k == i
          && i < nlabels - 1 && j < nlabels - 1 && k < nlabels - 1) // the first 2 diagonal elements
          indices_select_ptr->getData()(i, j, k) = 1;
        else if (j == 0)
          indices_select_ptr->getData()(i, j, k) = 1; // all elements along the first index of the selection dim
        else
          indices_select_ptr->getData()(i, j, k) = 0;
      }
    }
  }
  indices_select_ptr->syncDData(device);

  // test for AND within continuator and AND prepend continuator
  tensorTable.applyIndicesSelectToIndicesView(indices_select_ptr, "1", "2", logicalContinuators::logicalContinuator::AND, logicalContinuators::logicalContinuator::AND, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i == 0)
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);
  }

  // TODO: lacking code coverage for the case of TDim = 2
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_whereIndicesViewData1Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2, 3} });
  labels2.setValues({ {0, 1, 2, 3} });
  labels3.setValues({ {0, 1, 2, 3} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // set up the selection labels
  Eigen::Tensor<int, 1> select_labels_values(2);
  select_labels_values(0) = 0; select_labels_values(1) = 2;
  TensorDataGpuPrimitiveT<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 2 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(select_labels);
  select_labels_ptr->syncDData(device);

  // set up the selection values
  Eigen::Tensor<TensorArrayGpu8<char>, 1> select_values_values(2);
  select_values_values(0).setTensorArray("9"); select_values_values(1).setTensorArray("9");
  TensorDataGpuClassT<TensorArrayGpu8, char, 1> select_values(Eigen::array<Eigen::Index, 1>({ 2 }));
  select_values.setData(select_values_values);
  std::shared_ptr<TensorDataGpuClassT<TensorArrayGpu8, char, 1>> select_values_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 1>>(select_values);
  select_values_ptr->syncDData(device);

  // test
  tensorTable.whereIndicesView("1", 0, select_labels_ptr, select_values_ptr,
    logicalComparitors::logicalComparitor::EQUAL_TO, logicalModifiers::logicalModifier::NONE,
    logicalContinuators::logicalContinuator::OR, logicalContinuators::logicalContinuator::AND, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    // indices view 1
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1); // Unchanged

    // indices view 2
    if (i == 2) 
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);

    // indices view 3
    if (i == 1)
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), 0);
  }

  // Write the original data to disk, clear the data, and repeat the tests
  tensorTable.clear();
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);
  tensorTable.setData(tensor_values);
  tensorTable.syncIndicesDData(device);
  tensorTable.syncIndicesViewDData(device);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncShardIdDData(device);
  tensorTable.syncShardIndicesDData(device);
  tensorTable.syncAxesDData(device);
  tensorTable.syncDData(device);
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);

  // test  // FIXME: call to whereIndicesView is causing assertion failures for TensorArrayGpu8 size
  tensorTable.whereIndicesView("1", 0, select_labels_ptr, select_values_ptr,
    logicalComparitors::logicalComparitor::EQUAL_TO, logicalModifiers::logicalModifier::NONE,
    logicalContinuators::logicalContinuator::OR, logicalContinuators::logicalContinuator::AND, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    // indices view 1
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1); // Unchanged

    // indices view 2
    if (i == 2) // FIXME: i==0?
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);

    // indices view 3
    if (i == 1) // FIXME: i==3?
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), 0);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_whereIndicesViewData2Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2, 3} });
  labels2.setValues({ {0, 1, 2, 3} });
  labels3.setValues({ {0, 1, 2, 3} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // set up the selection labels
  Eigen::Tensor<int, 2> select_labels_values(1, 2);
  select_labels_values(0, 0) = 0; select_labels_values(0, 1) = 2;
  TensorDataGpuPrimitiveT<int, 2> select_labels(Eigen::array<Eigen::Index, 2>({ 1,2 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 2>>(select_labels);
  select_labels_ptr->syncDData(device);

  // set up the selection values
  Eigen::Tensor<TensorArrayGpu8<char>, 1> select_values_values(2);
  select_values_values(0).setTensorArray("9"); select_values_values(1).setTensorArray("9");
  TensorDataGpuClassT<TensorArrayGpu8, char, 1> select_values(Eigen::array<Eigen::Index, 1>({ 2 }));
  select_values.setData(select_values_values);
  std::shared_ptr<TensorDataGpuClassT<TensorArrayGpu8, char, 1>> select_values_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 1>>(select_values);
  select_values_ptr->syncDData(device);

  // test
  tensorTable.whereIndicesView("1", select_labels_ptr, select_values_ptr,
    logicalComparitors::logicalComparitor::EQUAL_TO, logicalModifiers::logicalModifier::NONE,
    logicalContinuators::logicalContinuator::OR, logicalContinuators::logicalContinuator::AND, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    // indices view 1
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1); // Unchanged

    // indices view 2
    if (i == 2)
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);

    // indices view 3
    if (i == 1)
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), 0);
  }

  // Write the original data to disk, clear the data, and repeat the tests
  tensorTable.clear();
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);
  tensorTable.setData(tensor_values);
  tensorTable.syncIndicesDData(device);
  tensorTable.syncIndicesViewDData(device);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncShardIdDData(device);
  tensorTable.syncShardIndicesDData(device);
  tensorTable.syncAxesDData(device);
  tensorTable.syncDData(device);
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);

  // FIXME: call to whereIndicesView is causing assertion failures for TensorArrayGpu8 size
  tensorTable.whereIndicesView("1", select_labels_ptr, select_values_ptr,
    logicalComparitors::logicalComparitor::EQUAL_TO, logicalModifiers::logicalModifier::NONE,
    logicalContinuators::logicalContinuator::OR, logicalContinuators::logicalContinuator::AND, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    // indices view 1
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1); // Unchanged

    // indices view 2
    if (i == 2)
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);

    // indices view 3
    if (i == 1)
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    else
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), 0);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_sliceTensorForSortGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // test sliceTensorForSort for axis 2
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 1>> tensor_sort;
  tensorTable.sliceTensorDataForSort(tensor_sort, "1", 1, "2", device); 
  tensor_sort->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  std::vector<TensorArrayGpu8<char>> tensor_slice_2_test = { TensorArrayGpu8<char>("9"),  TensorArrayGpu8<char>("12"),  TensorArrayGpu8<char>("15") };
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensor_sort->getData()(i), tensor_slice_2_test.at(i));
  }

  // test sliceTensorForSort for axis 2
  tensor_sort.reset();
  tensorTable.sliceTensorDataForSort(tensor_sort, "1", 1, "3", device);
  tensor_sort->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  std::vector<TensorArrayGpu8<char>> tensor_slice_3_test = { TensorArrayGpu8<char>("9"),  TensorArrayGpu8<char>("10"),  TensorArrayGpu8<char>("11") };
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensor_sort->getData()(i), tensor_slice_3_test.at(i));
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_sortIndicesViewData1Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // set up the selection labels
  Eigen::Tensor<int, 1> select_labels_values(1);
  select_labels_values(0) = 1;
  TensorDataGpuPrimitiveT<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(select_labels);
  select_labels_ptr->syncDData(device);

  // test sort ASC
  tensorTable.sortIndicesView("1", 0, select_labels_ptr, sortOrder::ASC, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  std::cout << "\nFailing test_sortIndicesViewData1Gpu tests:" << std::endl;
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    std::cout << "Predicted IndicesView2: " << tensorTable.getIndicesView().at("2")->getData()(i) << " Expected: " << i + 1 << std::endl;
    std::cout << "Predicted IndicesView3: " << tensorTable.getIndicesView().at("3")->getData()(i) << " Expected: " << i + 1 << std::endl;
    //gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1); // FIXME
    //gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1); // FIXME
  }

  // test sort DESC
  tensorTable.setIndicesViewDataStatus(false, true);
  tensorTable.sortIndicesView("1", 0, select_labels_ptr, sortOrder::DESC, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    std::cout << "Predicted IndicesView2: " << tensorTable.getIndicesView().at("2")->getData()(i) << " Expected: " << nlabels - i << std::endl;
    std::cout << "Predicted IndicesView3: " << tensorTable.getIndicesView().at("3")->getData()(i) << " Expected: " << nlabels - i << std::endl;
    //gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), nlabels - i); // FIXME
    //gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), nlabels - i); // FIXME
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_sortIndicesViewData2Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // set up the selection labels
  Eigen::Tensor<int, 2> select_labels_values(1, 1);
  select_labels_values(0,0) = 1;
  TensorDataGpuPrimitiveT<int, 2> select_labels(Eigen::array<Eigen::Index, 2>({ 1, 1 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 2>>(select_labels);
  select_labels_ptr->syncDData(device);

  // test sort ASC
  tensorTable.sortIndicesView("1", select_labels_ptr, sortOrder::ASC, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    std::cout << "Predicted IndicesView2: " << tensorTable.getIndicesView().at("2")->getData()(i) << " Expected: " << i + 1 << std::endl;
    std::cout << "Predicted IndicesView3: " << tensorTable.getIndicesView().at("3")->getData()(i) << " Expected: " << i + 1 << std::endl;
    //gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1); // FIXME
    //gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1); // FIXME
  }

  // test sort DESC
  tensorTable.setIndicesViewDataStatus(false, true);
  tensorTable.sortIndicesView("1", select_labels_ptr, sortOrder::DESC, device);
  tensorTable.syncHData(device);
  tensorTable.syncIndicesViewHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    std::cout << "Predicted IndicesView2: " << tensorTable.getIndicesView().at("2")->getData()(i) << " Expected: " << nlabels - i << std::endl;
    std::cout << "Predicted IndicesView3: " << tensorTable.getIndicesView().at("3")->getData()(i) << " Expected: " << nlabels - i << std::endl;
    //gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), nlabels - i); // FIXME
    //gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), nlabels - i); // FIXME
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeSelectIndicesFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // Test null
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_select;
  tensorTable.makeSelectIndicesFromTensorIndicesComponent(tensorTable.getIndicesView(), indices_select, device);
  indices_select->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(indices_select->getData()(i, j, k), 1);
      }
    }
  }

  // make the expected indices tensor
  Eigen::Tensor<int, 3> indices_select_test(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i == 1)
          indices_select_test(i, j, k) = 1;
        else
          indices_select_test(i, j, k) = 0;
      }
    }
  }

  // select
  TensorDataGpuPrimitiveT<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  Eigen::Tensor<int, 1> select_labels_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels_values.setValues({ 1 });
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(select_labels);
  select_labels_ptr->syncDData(device);
  tensorTable.selectIndicesView("1", 0, select_labels_ptr, device);

  // Test selected
  indices_select.reset();
  tensorTable.makeSelectIndicesFromTensorIndicesComponent(tensorTable.getIndicesView(), indices_select, device);
  indices_select->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(indices_select->getData()(i, j, k), indices_select_test(i, j, k));
      }
    }
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_getSelectTensorDataFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // select label 1 from axis 1
  TensorDataGpuPrimitiveT<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  Eigen::Tensor<int, 1> select_labels_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels_values.setValues({ 1 });
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(select_labels);
  select_labels_ptr->syncDData(device);
  tensorTable.selectIndicesView("1", 0, select_labels_ptr, device);

  // make the expected dimensions
  Eigen::array<Eigen::Index, 3> select_dimensions = { 1, 3, 3 };

  // make the indices_select
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_select_test(select_dimensions);
  Eigen::Tensor<int, 3> indices_select_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i == 1) {
          indices_select_values(i, j, k) = 1;
          tensor_select_test(0, j, k).setTensorArray(std::to_string(iter));
        }
        else {
          indices_select_values(i, j, k) = 0;
        }
        ++iter;
      }
    }
  }
  TensorDataGpuPrimitiveT<int, 3> indices_select(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  indices_select.setData(indices_select_values);
  std::shared_ptr<TensorDataGpuPrimitiveT<int, 3>> indices_select_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 3>>(indices_select);
  indices_select_ptr->syncDData(device);

  // test for the selected data
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> tensor_select_ptr;
  tensorTable.getSelectTensorDataFromIndicesView(tensor_select_ptr, indices_select_ptr, device);
  tensor_select_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(tensor_select_ptr->getDimensions(), select_dimensions);
  for (int j = 0; j < nlabels; ++j) {
    for (int k = 0; k < nlabels; ++k) {
      gpuCheckEqual(tensor_select_ptr->getData()(0, j, k), tensor_select_test(0, j, k), 1e-3);
    }
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_selectTensorDataGpuClassT()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // select label 1 from axis 1
  TensorDataGpuPrimitiveT<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  Eigen::Tensor<int, 1> select_labels_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels_values.setValues({ 1 });
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(select_labels);
  select_labels_ptr->syncDData(device);
  tensorTable.selectIndicesView("1", 0, select_labels_ptr, device);

  // Test `selectTensorData`
  tensorTable.selectTensorData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));

  // Test expected axes values
  gpuCheckEqual(tensorTable.getAxes().at("1")->getName(), "1");
  gpuCheckEqual(tensorTable.getAxes().at("1")->getNLabels(), 1);
  gpuCheckEqual(tensorTable.getAxes().at("1")->getDimensions()(0), "x");
  gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(0), 0);
  gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(0), 1);

  gpuCheckEqual(tensorTable.getAxes().at("2")->getName(), "2");
  gpuCheckEqual(tensorTable.getAxes().at("2")->getNLabels(), nlabels);
  gpuCheckEqual(tensorTable.getAxes().at("2")->getNDimensions(), 1);
  gpuCheckEqual(tensorTable.getAxes().at("2")->getDimensions()(0), "y");
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndices().at("2")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getShardId().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getShardIndices().at("2")->getData()(i), i + 1);
  }

  gpuCheckEqual(tensorTable.getAxes().at("3")->getName(), "3");
  gpuCheckEqual(tensorTable.getAxes().at("3")->getNLabels(), nlabels);
  gpuCheckEqual(tensorTable.getAxes().at("3")->getNDimensions(), 1);
  gpuCheckEqual(tensorTable.getAxes().at("3")->getDimensions()(0), "z");
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndices().at("3")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getShardId().at("3")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getShardIndices().at("3")->getData()(i), i + 1);
  }

  // Test expected axis to dims mapping
  gpuCheckEqual(tensorTable.getDimFromAxisName("1"), 0);
  gpuCheckEqual(tensorTable.getDimFromAxisName("2"), 1);
  gpuCheckEqual(tensorTable.getDimFromAxisName("3"), 2);

  // Test expected tensor dimensions
  gpuCheckEqual(tensorTable.getDimensions().at(0), 1);
  gpuCheckEqual(tensorTable.getDimensions().at(1), 3);
  gpuCheckEqual(tensorTable.getDimensions().at(2), 3);

  // Test expected tensor data values
  gpuCheckEqual(tensorTable.getDataDimensions().at(0), 1);
  gpuCheckEqual(tensorTable.getDataDimensions().at(1), 3);
  gpuCheckEqual(tensorTable.getDataDimensions().at(2), 3);
  size_t test = 1 * 3 * 3 * sizeof(TensorArrayGpu8<char>);
  gpuCheckEqual(tensorTable.getDataTensorBytes(), test);

  // Write the original data to disk, clear the data, and repeat the tests
  tensorTable.clear();
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);
  tensorTable.setData(tensor_values);
  tensorTable.syncIndicesDData(device);
  tensorTable.syncIndicesViewDData(device);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncShardIdDData(device);
  tensorTable.syncShardIndicesDData(device);
  tensorTable.syncAxesDData(device);
  tensorTable.syncDData(device);
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);

  // Test selectTensorData
  tensorTable.selectIndicesView("1", 0, select_labels_ptr, device);
  tensorTable.selectTensorData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));

  // Test expected axes values
  gpuCheckEqual(tensorTable.getAxes().at("1")->getName(), "1");
  gpuCheckEqual(tensorTable.getAxes().at("1")->getNLabels(), 1);
  gpuCheckEqual(tensorTable.getAxes().at("1")->getDimensions()(0), "x");
  gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(0), 0);
  gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(0), 1);
  gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(0), 1);

  gpuCheckEqual(tensorTable.getAxes().at("2")->getName(), "2");
  gpuCheckEqual(tensorTable.getAxes().at("2")->getNLabels(), nlabels);
  gpuCheckEqual(tensorTable.getAxes().at("2")->getNDimensions(), 1);
  gpuCheckEqual(tensorTable.getAxes().at("2")->getDimensions()(0), "y");
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndices().at("2")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getShardId().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getShardIndices().at("2")->getData()(i), i + 1);
  }

  gpuCheckEqual(tensorTable.getAxes().at("3")->getName(), "3");
  gpuCheckEqual(tensorTable.getAxes().at("3")->getNLabels(), nlabels);
  gpuCheckEqual(tensorTable.getAxes().at("3")->getNDimensions(), 1);
  gpuCheckEqual(tensorTable.getAxes().at("3")->getDimensions()(0), "z");
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndices().at("3")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getShardId().at("3")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getShardIndices().at("3")->getData()(i), i + 1);
  }

  // Test expected axis to dims mapping
  gpuCheckEqual(tensorTable.getDimFromAxisName("1"), 0);
  gpuCheckEqual(tensorTable.getDimFromAxisName("2"), 1);
  gpuCheckEqual(tensorTable.getDimFromAxisName("3"), 2);

  // Test expected tensor dimensions
  gpuCheckEqual(tensorTable.getDimensions().at(0), 1);
  gpuCheckEqual(tensorTable.getDimensions().at(1), 3);
  gpuCheckEqual(tensorTable.getDimensions().at(2), 3);

  // Test expected tensor data values
  gpuCheckEqual(tensorTable.getDataDimensions().at(0), 1);
  gpuCheckEqual(tensorTable.getDataDimensions().at(1), 3);
  gpuCheckEqual(tensorTable.getDataDimensions().at(2), 3);
  gpuCheckEqual(tensorTable.getDataTensorBytes(), test);

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeSortIndicesViewFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // make the expected tensor indices
  Eigen::Tensor<int, 3> indices_test(nlabels, nlabels, nlabels);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        indices_test(i, j, k) = i + j * nlabels + k * nlabels*nlabels + 1;
      }
    }
  }

  // Test for the sort indices
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_sort_ptr;
  tensorTable.makeSortIndicesFromTensorIndicesComponent(tensorTable.getIndicesView(), indices_sort_ptr, device);
  indices_sort_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(indices_sort_ptr->getData()(i, j, k), indices_test(i, j, k));
      }
    }
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_sortTensorDataGpuClassT()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // set up the selection labels
  Eigen::Tensor<int, 1> select_labels_values(1);
  select_labels_values(0) = 0;
  TensorDataGpuPrimitiveT<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(select_labels);
  select_labels_ptr->syncDData(device);

  // make the expected sorted tensor
  TensorArrayGpu8<char> sorted_data[] = { TensorArrayGpu8<char>("24"),  TensorArrayGpu8<char>("25"),  TensorArrayGpu8<char>("26"),  TensorArrayGpu8<char>("21"),  TensorArrayGpu8<char>("22"),  TensorArrayGpu8<char>("23"),  TensorArrayGpu8<char>("18"),  TensorArrayGpu8<char>("19"),  TensorArrayGpu8<char>("20"),  TensorArrayGpu8<char>("15"),  TensorArrayGpu8<char>("16"),  TensorArrayGpu8<char>("17"),  TensorArrayGpu8<char>("12"),  TensorArrayGpu8<char>("13"),  TensorArrayGpu8<char>("14"),  TensorArrayGpu8<char>("9"),  TensorArrayGpu8<char>("10"),  TensorArrayGpu8<char>("11"),  TensorArrayGpu8<char>("6"),  TensorArrayGpu8<char>("7"),  TensorArrayGpu8<char>("8"),  TensorArrayGpu8<char>("3"),  TensorArrayGpu8<char>("4"),  TensorArrayGpu8<char>("5"),  TensorArrayGpu8<char>("0"),  TensorArrayGpu8<char>("1"),  TensorArrayGpu8<char>("2") };
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 3>> tensor_sorted_values(sorted_data, Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));

  // sort each of the axes
  tensorTable.sortIndicesView("1", 0, select_labels_ptr, sortOrder::DESC, device);

  // Test for sorted tensor data and reset indices view
  tensorTable.sortTensorData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  std::cout << "test_sortTensorDataGpuClassT\n" << std::endl;
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    gpuCheckEqual(axis_1_ptr->getLabels()(0, i), i);
    gpuCheckEqual(axis_2_ptr->getLabels()(0, i), nlabels - i - 1);
    std::cout << "axis_3_ptr->getLabels() Predicted: " << axis_3_ptr->getLabels()(0, i) << " Expected: " << nlabels - i - 1 << std::endl;
    //gpuCheckEqual(axis_3_ptr->getLabels()(0, i), nlabels - i - 1); //FIXME
  }
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        std::cout << "Predicted: " << tensorTable.getData()(i, j, k) << " Expected: " << tensor_sorted_values(i, j, k) << std::endl;
        //gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_sorted_values(i, j, k)); //FIXME
      }
    }
  }

  // Write the original data to disk, clear the data, and repeat the tests
  tensorTable.clear();
  axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);
  tensorTable.setData(tensor_values);
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);

  // sort each of the axes
  tensorTable.sortIndicesView("1", 0, select_labels_ptr, sortOrder::DESC, device);

  // Test for sorted tensor data and reset indices view
  tensorTable.sortTensorData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    gpuCheckEqual(axis_1_ptr->getLabels()(0, i), i);
    gpuCheckEqual(axis_2_ptr->getLabels()(0, i), nlabels - i - 1);
    std::cout << "axis_3_ptr->getLabels() Predicted: " << axis_3_ptr->getLabels()(0, i) << " Expected: " << nlabels - i - 1 << std::endl;
    //gpuCheckEqual(axis_3_ptr->getLabels()(0, i), nlabels - i - 1); // FIXME
  }
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        std::cout << "Predicted: " << tensorTable.getData()(i, j, k) << " Expected: " << tensor_sorted_values(i, j, k) << std::endl;
        //gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_sorted_values(i, j, k)); // FIXME
      }
    }
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_updateSelectTensorDataValues1Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data and the update values
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<TensorArrayGpu8<char>, 3> update_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        update_values(i, j, k).setTensorArray(std::to_string(100));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> values_new(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  values_new.setData(update_values);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(values_new);
  
  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  values_new_ptr->syncDData(device);

  // Test update
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> values_old_ptr;
  tensorTable.updateSelectTensorDataValues(values_new_ptr, values_old_ptr, device);
  values_old_ptr->syncHData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(values_old_ptr->getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(iter)));
        gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(100)));
        ++iter;
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
  }

  // Write the original data to disk, clear the data, and repeat the tests
  tensorTable.clear();
  axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);
  tensorTable.setData(tensor_values);
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);

  // Test update
  values_old_ptr.reset();
  tensorTable.updateSelectTensorDataValues(values_new_ptr, values_old_ptr, device);
  values_old_ptr->syncHData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(values_old_ptr->getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(iter)));
        gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(100)));
        ++iter;
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_updateSelectTensorDataValues2Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data and the update values
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<TensorArrayGpu8<char>, 3> update_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        update_values(i, j, k).setTensorArray(std::to_string(100));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> values_new(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  values_new.setData(update_values);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(values_new);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  values_new_ptr->syncDData(device);

  // Test update
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> values_old(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  values_old.setData();
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> values_old_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(values_old);
  values_old_ptr->syncDData(device);
  tensorTable.updateSelectTensorDataValues(values_new_ptr->getDataPointer(), values_old_ptr->getDataPointer(), device);
  values_old_ptr->syncHData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(values_old_ptr->getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(iter)));
        gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(100)));
        ++iter;
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
  }

  // Write the original data to disk, clear the data, and repeat the tests
  tensorTable.clear();
  axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);
  tensorTable.setData(tensor_values);
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);

  // Test update
  values_old_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(values_old);
  values_old_ptr->syncDData(device);
  tensorTable.updateSelectTensorDataValues(values_new_ptr->getDataPointer(), values_old_ptr->getDataPointer(), device);
  values_old_ptr->syncHData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(values_old_ptr->getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(iter)));
        gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(100)));
        ++iter;
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_updateTensorDataValuesGpu()
{
	// setup the table
	TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

	// Initialize the device
	cudaStream_t stream;
	gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);

	// setup the axes
	Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
	dimensions1(0) = "x";
	dimensions2(0) = "y";
	dimensions3(0) = "z";
	int nlabels = 3;
	Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
	labels1.setValues({ {0, 1, 2} });
	labels2.setValues({ {0, 1, 2} });
	labels3.setValues({ {0, 1, 2} });
	std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
	std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
	std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
	tensorTable.addTensorAxis(axis_1_ptr);
	tensorTable.addTensorAxis(axis_2_ptr);
	tensorTable.addTensorAxis(axis_3_ptr);
	tensorTable.setAxes(device);

	// setup the tensor data and the update values
	Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
	Eigen::Tensor<TensorArrayGpu8<char>, 3> update_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
	int iter = 0;
	for (int k = 0; k < nlabels; ++k) {
		for (int j = 0; j < nlabels; ++j) {
			for (int i = 0; i < nlabels; ++i) {
				tensor_values(i, j, k).setTensorArray(std::to_string(iter));
				update_values(i, j, k).setTensorArray(std::to_string(100));
				++iter;
			}
		}
	}
	tensorTable.setData(tensor_values);
	TensorDataGpuClassT<TensorArrayGpu8, char, 3> values_new(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
	values_new.setData(update_values);
	std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(values_new);

	// sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
	values_new_ptr->syncDData(device);

	// Test update
	std::shared_ptr<TensorTable<TensorArrayGpu8<char>, Eigen::GpuDevice, 2>> values_old_ptr;
	tensorTable.updateTensorDataValues(values_new_ptr->getDataPointer(), values_old_ptr, device);
	values_old_ptr->syncHData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
	gpuErrchk(cudaStreamSynchronize(stream));
	for (int k = 0; k < nlabels; ++k) {
		for (int j = 0; j < nlabels; ++j) {
			for (int i = 0; i < nlabels; ++i) {
				gpuCheckEqual(values_old_ptr->getData()(i + j * nlabels + k * nlabels * nlabels), tensor_values(i, j, k));
				gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(100)));
			}
		}
	}

	// Test for the in_memory and is_modified attributes
	for (int i = 0; i < nlabels; ++i) {
		gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
		gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
		gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
		gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
		gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
		gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
	}

	// Write the original data to disk, clear the data, and repeat the tests
	tensorTable.clear();
	axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
	axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
	axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
	tensorTable.addTensorAxis(axis_1_ptr);
	tensorTable.addTensorAxis(axis_2_ptr);
	tensorTable.addTensorAxis(axis_3_ptr);
	tensorTable.setAxes(device);
	tensorTable.setData(tensor_values);
	tensorTable.syncAxesAndIndicesDData(device);
	tensorTable.syncDData(device);
	tensorTable.storeTensorTableBinary("", device);
	tensorTable.setData();
	tensorTable.setNotInMemoryDataStatus(true, false);
	tensorTable.syncNotInMemoryDData(device);
	tensorTable.setIsModifiedDataStatus(true, false);
	tensorTable.syncIsModifiedDData(device);
	tensorTable.syncDData(device);

	// Test update
	values_old_ptr.reset();
	tensorTable.updateTensorDataValues(values_new_ptr->getDataPointer(), values_old_ptr, device);
	values_old_ptr->syncHData(device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
	gpuErrchk(cudaStreamSynchronize(stream));
	for (int k = 0; k < nlabels; ++k) {
		for (int j = 0; j < nlabels; ++j) {
			for (int i = 0; i < nlabels; ++i) {
				gpuCheckEqual(values_old_ptr->getData()(i + j * nlabels + k * nlabels * nlabels), tensor_values(i, j, k));
				gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(100)));
			}
		}
	}

	// Test for the in_memory and is_modified attributes
	for (int i = 0; i < nlabels; ++i) {
		gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
		gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
		gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
		gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
		gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
		gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
	}

	gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeAppendIndicesGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  //std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", 1, 0));
  axis_3_ptr->setDimensions(dimensions3);
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);

  // test the making the append indices
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_ptr;
  tensorTable.makeAppendIndices("1", nlabels, indices_ptr, device);
  indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(indices_ptr->getData()(i), nlabels + i + 1);
  }

  // test the making the append indices
  indices_ptr.reset();
  tensorTable.makeAppendIndices("3", nlabels, indices_ptr, device);
  indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(indices_ptr->getData()(i), i + 1);
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_appendToIndicesGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the new indices
  Eigen::Tensor<int, 1> indices_new_values(nlabels - 1);
  for (int i = 0; i < nlabels - 1; ++i) {
    indices_new_values(i) = nlabels + i + 1;
  }
  TensorDataGpuPrimitiveT<int, 1> indices_new(Eigen::array<Eigen::Index, 1>({ nlabels - 1 }));
  indices_new.setData(indices_new_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_new_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(indices_new);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  indices_new_ptr->syncDData(device);

  // test appendToIndices
  tensorTable.appendToIndices("1", indices_new_ptr, device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(tensorTable.getDimensions().at(tensorTable.getDimFromAxisName("1")), nlabels + nlabels - 1);
  for (int i = 0; i < nlabels + nlabels - 1; ++i) {
    gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(i), 1);
    if (i < nlabels) {
      gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), i + 1);
    }
    else {
      gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), 0);
    }
  }

  // check the existing indices
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndices().at("2")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getShardId().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getShardIndices().at("2")->getData()(i), i + 1);
  }
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getIndices().at("3")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getShardId().at("3")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getShardIndices().at("3")->getData()(i), i + 1);
  }

  // Check the dimensions and tensor size
  Eigen::array<Eigen::Index, 3> dimensions_test = { nlabels + nlabels - 1, nlabels, nlabels };
  gpuCheckEqual(tensorTable.getDimensions(), dimensions_test);
  gpuCheckEqual(tensorTable.getTensorSize(), (nlabels + nlabels - 1) * nlabels * nlabels);

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_appendToAxis1Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // setup the new tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> update_values(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      update_values(0, i, j).setTensorArray(std::to_string(i));
    }
  }
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> values_new(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  values_new.setData(update_values);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(values_new);

  // setup the new axis labels
  Eigen::Tensor<int, 2> labels_values(Eigen::array<Eigen::Index, 2>({ 1, 1 }));
  labels_values(0, 0) = 3;
  TensorDataGpuPrimitiveT<int, 2> labels_new(Eigen::array<Eigen::Index, 2>({ 1, 1 }));
  labels_new.setData(labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_new_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 2>>(labels_new);

  // setup the new indices
  TensorDataGpuPrimitiveT<int, 1> indices_new(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_new.setData();
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_new_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(indices_new);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  labels_new_ptr->syncDData(device);
  values_new_ptr->syncDData(device);
  indices_new_ptr->syncDData(device);

  // test appendToAxis
  tensorTable.appendToAxis("1", labels_new_ptr, values_new_ptr->getDataPointer(), indices_new_ptr, device);
  tensorTable.syncAxesHData(device);
  tensorTable.syncHData(device);
  indices_new_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(axis_1_ptr->getLabels()(0, i), labels1(i));
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }
  gpuCheck(axis_1_ptr->getLabels()(0, nlabels), 3);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      gpuCheckEqual(tensorTable.getData()(nlabels, i, j), update_values(0, i, j));
    }
  }
  gpuCheckEqual(indices_new_ptr->getData()(0), nlabels + 1);

  // test the expected dimensions
  Eigen::array<Eigen::Index, 3> dimensions_test = { nlabels + 1, nlabels, nlabels };
  gpuCheckEqual(tensorTable.getDimensions(), dimensions_test);
  gpuCheckEqual(tensorTable.getTensorSize(), (nlabels + 1) * nlabels * nlabels);

  // Write the original data to disk, clear the data, and repeat the tests
  tensorTable.clear();
  axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);
  tensorTable.setData(tensor_values);
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);
  indices_new_ptr->setDataStatus(false, true);

  // test appendToAxis
  tensorTable.appendToAxis("1", labels_new_ptr, values_new_ptr->getDataPointer(), indices_new_ptr, device);
  tensorTable.syncAxesHData(device);
  tensorTable.syncHData(device);
  indices_new_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(axis_1_ptr->getLabels()(0, i), labels1(i));
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }
  gpuCheck(axis_1_ptr->getLabels()(0, nlabels), 3);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      gpuCheckEqual(tensorTable.getData()(nlabels, i, j), update_values(0, i, j));
    }
  }
  gpuCheckEqual(indices_new_ptr->getData()(0), nlabels + 1);

  // test the expected dimensions
  dimensions_test = Eigen::array<Eigen::Index, 3>({ nlabels + 1, nlabels, nlabels });
  gpuCheckEqual(tensorTable.getDimensions(), dimensions_test);
  gpuCheckEqual(tensorTable.getTensorSize(), (nlabels + 1) * nlabels * nlabels);

  // Check that the binarized data was written correctly
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.syncDData(device);

  // Reset the in_memory values
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    in_memory_map.second->getData() = in_memory_map.second->getData().constant(1);
  }
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);

  tensorTable.loadTensorTableBinary("", device);
  // Test the new TensorTable
  tensorTable.syncHData(device);
  indices_new_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      gpuCheckEqual(tensorTable.getData()(nlabels, i, j), update_values(0, i, j));
    }
  }

  // test the expected dimensions
  dimensions_test = Eigen::array<Eigen::Index, 3>({ nlabels + 1, nlabels, nlabels });
  gpuCheckEqual(tensorTable.getDimensions(), dimensions_test);
  gpuCheckEqual(tensorTable.getTensorSize(), (nlabels + 1) * nlabels * nlabels);

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_appendToAxis2Gpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", 1, 0));
  axis_1_ptr->setDimensions(dimensions1);
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);
  tensorTable.setData();

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(iter));
        ++iter;
      }
    }
  }
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> values_new(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  values_new.setData(tensor_values);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(values_new);

  // setup the new axis labels
  TensorDataGpuPrimitiveT<int, 2> labels_new(Eigen::array<Eigen::Index, 2>({ 1, nlabels }));
  labels_new.setData(labels1);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_new_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 2>>(labels_new);

  // setup the new indices
  TensorDataGpuPrimitiveT<int, 1> indices_new(Eigen::array<Eigen::Index, 1>({ nlabels }));
  indices_new.setData();
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_new_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(indices_new);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  labels_new_ptr->syncDData(device);
  values_new_ptr->syncDData(device);
  indices_new_ptr->syncDData(device);

  // test appendToAxis
  tensorTable.appendToAxis("1", labels_new_ptr, values_new_ptr->getDataPointer(), indices_new_ptr, device);
  tensorTable.syncAxesHData(device);
  tensorTable.syncHData(device);
  indices_new_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(axis_1_ptr->getLabels()(0, i), labels1(i));
    gpuCheckEqual(indices_new_ptr->getData()(i), i + 1);
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }

  // test the expected dimensions
  Eigen::array<Eigen::Index, 3> dimensions_test = { nlabels, nlabels, nlabels };
  gpuCheckEqual(tensorTable.getDimensions(), dimensions_test);
  gpuCheckEqual(tensorTable.getTensorSize(), nlabels * nlabels * nlabels);

  // Check that the binarized data was written correctly
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.syncDData(device);

  // Reset the in_memory values
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    in_memory_map.second->getData() = in_memory_map.second->getData().constant(1);
  }
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);

  tensorTable.loadTensorTableBinary("", device);
  // Test the new TensorTable  
  tensorTable.syncAxesHData(device);
  tensorTable.syncHData(device);
  indices_new_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(axis_1_ptr->getLabels()(0, i), labels1(i));
    gpuCheckEqual(indices_new_ptr->getData()(i), i + 1);
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }

  // test the expected dimensions
  dimensions_test = Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels });
  gpuCheckEqual(tensorTable.getDimensions(), dimensions_test);
  gpuCheckEqual(tensorTable.getTensorSize(), nlabels * nlabels * nlabels);


  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeIndicesViewSelectFromIndicesGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the selection indices
  Eigen::Tensor<int, 1> indices_to_select_values(Eigen::array<Eigen::Index, 1>({ 2 }));
  indices_to_select_values.setValues({ 1, 2 });
  TensorDataGpuPrimitiveT<int, 1> indices_to_select(Eigen::array<Eigen::Index, 1>({ 2 }));
  indices_to_select.setData(indices_to_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_to_select_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(indices_to_select);
  
  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  indices_to_select_ptr->syncDData(device);

  // test makeIndicesViewSelectFromIndices
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_select_ptr;
  tensorTable.makeIndicesViewSelectFromIndices("1", indices_select_ptr, indices_to_select_ptr, true, device);
  indices_select_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i > 1)
      gpuCheckEqual(indices_select_ptr->getData()(i), 1);
    else
      gpuCheckEqual(indices_select_ptr->getData()(i), 0);
  }
  indices_select_ptr.reset();
  tensorTable.makeIndicesViewSelectFromIndices("1", indices_select_ptr, indices_to_select_ptr, false, device);
  indices_select_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i <= 1)
      gpuCheckEqual(indices_select_ptr->getData()(i), 1);
    else
      gpuCheckEqual(indices_select_ptr->getData()(i), 0);
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_deleteFromIndicesGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the selection indices
  Eigen::Tensor<int, 1> indices_to_select_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_to_select_values.setValues({ 2 });
  TensorDataGpuPrimitiveT<int, 1> indices_to_select(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_to_select.setData(indices_to_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_to_select_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(indices_to_select);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  indices_to_select_ptr->syncDData(device);

  // test deleteFromIndices
  tensorTable.deleteFromIndices("1", indices_to_select_ptr, device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(tensorTable.getDimensions().at(tensorTable.getDimFromAxisName("1")), nlabels - 1);
  for (int i = 0; i < nlabels - 1; ++i) {
    if (i == 0) {
      gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), i + 1);
    }
    else {
      gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(i), i + 2);
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 2);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), i + 2);
    }
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(i), 1);
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeSelectIndicesFromIndicesGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the selection indices
  Eigen::Tensor<int, 1> indices_to_select_values(Eigen::array<Eigen::Index, 1>({ nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    if (i % 2 == 0) indices_to_select_values(i) = i + 1;
    else indices_to_select_values(i) = 0;
  }
  TensorDataGpuPrimitiveT<int, 1> indices_to_select(Eigen::array<Eigen::Index, 1>({ nlabels }));
  indices_to_select.setData(indices_to_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_to_select_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(indices_to_select);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  indices_to_select_ptr->syncDData(device);

  // test the selection indices
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_select_ptr;
  tensorTable.makeSelectIndicesFromIndices("1", indices_to_select_ptr, indices_select_ptr, device);
  indices_select_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i % 2 == 0)
          gpuCheckEqual(indices_select_ptr->getData()(i, j, k), 1);
        else
          gpuCheckEqual(indices_select_ptr->getData()(i, j, k), 0);
      }
    }
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_deleteFromAxisGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<TensorArrayGpu8<char>, 3> new_values(Eigen::array<Eigen::Index, 3>({ nlabels - 1, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k).setTensorArray(std::to_string(i + j * nlabels + k * nlabels*nlabels));
        if (i != 1) {
          new_values(iter, j, k).setTensorArray(std::to_string(i + j * nlabels + k * nlabels*nlabels));
        }
      }
    }
    if (i != 1) ++iter;
  }
  tensorTable.setData(tensor_values);

  // setup the selection indices
  Eigen::Tensor<int, 1> indices_to_select_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_to_select_values.setValues({ 2 });
  TensorDataGpuPrimitiveT<int, 1> indices_to_select(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_to_select.setData(indices_to_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_to_select_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(indices_to_select);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  indices_to_select_ptr->syncDData(device);

  // test deleteFromAxis
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> values(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  values.setData();
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> values_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(values);
  values_ptr->syncDData(device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_ptr;
  tensorTable.deleteFromAxis("1", indices_to_select_ptr, labels_ptr, values_ptr->getDataPointer(), device);

  // test the expected indices sizes and values
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  values_ptr->syncHData(device);
  labels_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(tensorTable.getDimensions().at(tensorTable.getDimFromAxisName("1")), nlabels - 1);
  for (int i = 0; i < nlabels - 1; ++i) {
    if (i == 0) {
      gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), i + 1);
    }
    else {
      gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(i), i + 2);
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 2);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), i + 2);
    }
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(i), 1);
  }

  // Test the expected data values
  for (int i = 0; i < nlabels - 1; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), new_values(i, j, k));
      }
    }
  }

  // Test the expected axis values
  std::vector<int> expected_labels = { 0, 2 };
  for (int i = 0; i < nlabels - 1; ++i) {
    gpuCheckEqual(axis_1_ptr->getLabels()(0, i), expected_labels.at(i));
  }

  // Test the expected returned labels
  gpuCheckEqual(labels_ptr->getData()(0, 0), 1);

  // Test the expected returned data
  for (int i = 0; i < 1; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(values_ptr->getData()(i, j, k), tensor_values(1, j, k));
      }
    }
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeIndicesFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // modify the indices view for axis 1
  tensorTable.getIndicesView().at("1")->getData()(0) = 0;

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);

  // test makeIndicesFromIndicesView
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_ptr;
  tensorTable.makeIndicesFromIndicesView("1", indices_ptr, device);
  indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels - 1; ++i) {
    gpuCheckEqual(indices_ptr->getData()(i), i + 2);
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_insertIntoAxisGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(i + j * nlabels + k * nlabels*nlabels));
      }
    }
  }
  tensorTable.setData(tensor_values);

  // setup the new tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> update_values(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      update_values(0, i, j).setTensorArray(std::to_string(100));
    }
  }
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> values_new(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  values_new.setData(update_values);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(values_new);

  // setup the new axis labels
  Eigen::Tensor<int, 2> labels_values(Eigen::array<Eigen::Index, 2>({ 1, 1 }));
  labels_values(0, 0) = 100;
  TensorDataGpuPrimitiveT<int, 2> labels_new(Eigen::array<Eigen::Index, 2>({ 1, 1 }));
  labels_new.setData(labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_new_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 2>>(labels_new);

  // setup the new indices
  Eigen::Tensor<int, 1> indices_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_values(0) = 3;
  TensorDataGpuPrimitiveT<int, 1> indices_new(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_new.setData(indices_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_new_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 1>>(indices_new);

  // Change the indices and indices view to simulate a deletion
  tensorTable.getIndices().at("1")->getData()(nlabels - 1) = 4;
  tensorTable.getIndicesView().at("1")->getData()(nlabels - 1) = 4;

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  values_new_ptr->syncDData(device);
  labels_new_ptr->syncDData(device);
  indices_new_ptr->syncDData(device);

  // test insertIntoAxis
  tensorTable.insertIntoAxis("1", labels_new_ptr, values_new_ptr->getDataPointer(), indices_new_ptr, device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  int iter = 0;
  for (int i = 0; i < nlabels + 1; ++i) {
    // check the axis
    if (i == 2)
      gpuCheckEqual(axis_1_ptr->getLabels()(0, i), 100);
    else
      gpuCheckEqual(axis_1_ptr->getLabels()(0, i), labels1(iter));

    // check the indices
    gpuCheckEqual(tensorTable.getIndices().at("1")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    if (i >= nlabels) {
      gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(i), 2);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), i - nlabels + 1);
    }
    else {
      gpuCheckEqual(tensorTable.getShardId().at("1")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getShardIndices().at("1")->getData()(i), i + 1);
    }

    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        // check the tensor data
        if (i == 2)
          gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(100)));
        else
          gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(iter, j, k));
      }
    }
    if (i != 2) ++iter;
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeSparseAxisLabelsFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<int, 2> expected_values(Eigen::array<Eigen::Index, 2>({ 3, nlabels*nlabels*nlabels }));
  expected_values.setValues({
    {1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3 },
    {1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3 },
    {1,1,1,1,1,1,1,1,1,2,2,2,2,2,2,2,2,2,3,3,3,3,3,3,3,3,3 } });

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // Test
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_ptr;
  tensorTable.makeSparseAxisLabelsFromIndicesView(labels_ptr, device);
  labels_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(labels_ptr->getDimensions().at(0), 3);
  gpuCheckEqual(labels_ptr->getDimensions().at(1), nlabels*nlabels*nlabels);
  for (int i = 0; i < 3; ++i) {
    for (int j = 0; j < nlabels*nlabels*nlabels; ++j) {
      gpuCheckEqual(labels_ptr->getData()(i, j), expected_values(i, j));
    }
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeSparseTensorTableGpu()
{
  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the expected axes
  Eigen::Tensor<std::string, 1> dimensions1(3), dimensions2(1);
  dimensions1.setValues({ "0", "1", "2" });
  dimensions2(0) = "Values";

  // setup the expected labels
  int nlabels1 = 27;
  Eigen::Tensor<int, 2> labels1(3, nlabels1), labels2(1, 1);
  labels1.setValues({
    {1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3 },
    {1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3 },
    {1,1,1,1,1,1,1,1,1,2,2,2,2,2,2,2,2,2,3,3,3,3,3,3,3,3,3 } });
  TensorDataGpuPrimitiveT<int, 2> sparse_labels(Eigen::array<Eigen::Index, 2>({ 3, nlabels1 }));
  sparse_labels.setData(labels1);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> sparse_labels_ptr = std::make_shared<TensorDataGpuPrimitiveT<int, 2>>(sparse_labels);

  labels2.setConstant(0);

  // setup the expected data
  int nlabels = 3;
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(i + j * nlabels + k * nlabels*nlabels));
      }
    }
  }
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> sparse_data(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  sparse_data.setData(tensor_values);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> sparse_data_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(sparse_data);

  // Test
  std::shared_ptr<TensorTable<TensorArrayGpu8<char>, Eigen::GpuDevice, 2>> sparse_table_ptr;
  sparse_labels_ptr->syncDData(device);
  sparse_data_ptr->syncDData(device);
  sparse_labels_ptr->syncDData(device);
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;
  tensorTable.makeSparseTensorTable(dimensions1, sparse_labels_ptr, sparse_data_ptr, sparse_table_ptr, device);
  sparse_labels_ptr->syncHData(device);
  sparse_data_ptr->syncHData(device);
  sparse_table_ptr->syncAxesAndIndicesHData(device);
  sparse_table_ptr->syncHData(device);
  sparse_labels_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));

  // Check for the correct dimensions
  gpuCheckEqual(sparse_table_ptr->getDimensions().at(0), nlabels1);
  gpuCheckEqual(sparse_table_ptr->getDimensions().at(1), 1);

  // Check the data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(sparse_table_ptr->getData()(i + j * nlabels + k * nlabels*nlabels), tensor_values(i, j, k));
      }
    }
  }

  // Check the Indices axes
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getName(), "Indices");
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getNLabels(), nlabels1);
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getNDimensions(), 3);

  std::shared_ptr<int[]> labels1_ptr;
  sparse_table_ptr->getAxes().at("Indices")->getLabelsHDataPointer(labels1_ptr);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels_values(labels1_ptr.get(), 3, nlabels1);
  for (int i = 0; i < 3; ++i) {
    gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getDimensions()(i), std::to_string(i));
    for (int j = 0; j < nlabels1; ++j) {
      gpuCheckEqual(labels_values(i, j), labels1(i, j));
    }
  }

  // Check the Values axes
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getName(), "Values");
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getNLabels(), 1);
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getNDimensions(), 1);

  std::shared_ptr<int[]> labels2_ptr;
  sparse_table_ptr->getAxes().at("Values")->getLabelsHDataPointer(labels2_ptr);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels2_values(labels2_ptr.get(), 1, 1);
  gpuCheckEqual(labels2_values(0, 0), 0);
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getDimensions()(0), "Values");

  // Check the indices axis indices
  for (int i = 0; i < nlabels1; ++i) {
    gpuCheckEqual(sparse_table_ptr->getIndices().at("Indices")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIndicesView().at("Indices")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIsModified().at("Indices")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getNotInMemory().at("Indices")->getData()(i), 0);
    gpuCheckEqual(sparse_table_ptr->getShardId().at("Indices")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getShardIndices().at("Indices")->getData()(i), i + 1);
  }

  // Check the values axis indices
  for (int i = 0; i < 1; ++i) {
    gpuCheckEqual(sparse_table_ptr->getIndices().at("Values")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIndicesView().at("Values")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIsModified().at("Values")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getNotInMemory().at("Values")->getData()(i), 0);
    gpuCheckEqual(sparse_table_ptr->getShardId().at("Values")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getShardIndices().at("Values")->getData()(i), i + 1);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_getSelectTensorDataAsSparseTensorTableGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(i + j * nlabels + k * nlabels*nlabels));
      }
    }
  }
  tensorTable.setData(tensor_values);

  // setup the expected labels
  int nlabels1 = 27;
  Eigen::Tensor<int, 2> labels1_expected(3, nlabels1);
  labels1_expected.setValues({
    {1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3 },
    {1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3 },
    {1,1,1,1,1,1,1,1,1,2,2,2,2,2,2,2,2,2,3,3,3,3,3,3,3,3,3 } });

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // Test
  std::shared_ptr<TensorTable<TensorArrayGpu8<char>, Eigen::GpuDevice, 2>> sparse_table_ptr;
  tensorTable.getSelectTensorDataAsSparseTensorTable(sparse_table_ptr, device);
  sparse_table_ptr->syncAxesAndIndicesHData(device);
  sparse_table_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));

  // Check for the correct dimensions
  gpuCheckEqual(sparse_table_ptr->getDimensions().at(0), nlabels1);
  gpuCheckEqual(sparse_table_ptr->getDimensions().at(1), 1);

  // Check the data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(sparse_table_ptr->getData()(i + j * nlabels + k * nlabels*nlabels), tensor_values(i, j, k));
      }
    }
  }

  // Check the Indices axes
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getName(), "Indices");
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getNLabels(), nlabels1);
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getNDimensions(), 3);

  std::shared_ptr<int[]> labels1_ptr;
  sparse_table_ptr->getAxes().at("Indices")->getLabelsHDataPointer(labels1_ptr);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels_values(labels1_ptr.get(), 3, nlabels1);
  for (int i = 0; i < 3; ++i) {
    gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getDimensions()(i), std::to_string(i + 1));
    for (int j = 0; j < nlabels1; ++j) {
      gpuCheckEqual(labels_values(i, j), labels1_expected(i, j));
    }
  }

  // Check the Values axes
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getName(), "Values");
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getNLabels(), 1);
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getNDimensions(), 1);

  std::shared_ptr<int[]> labels2_ptr;
  sparse_table_ptr->getAxes().at("Values")->getLabelsHDataPointer(labels2_ptr);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels2_values(labels2_ptr.get(), 1, 1);
  gpuCheckEqual(labels2_values(0, 0), 0);
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getDimensions()(0), "Values");

  // Check the indices axis indices
  for (int i = 0; i < nlabels1; ++i) {
    gpuCheckEqual(sparse_table_ptr->getIndices().at("Indices")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIndicesView().at("Indices")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIsModified().at("Indices")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getNotInMemory().at("Indices")->getData()(i), 0);
    gpuCheckEqual(sparse_table_ptr->getShardId().at("Indices")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getShardIndices().at("Indices")->getData()(i), i + 1);
  }

  // Check the values axis indices
  for (int i = 0; i < 1; ++i) {
    gpuCheckEqual(sparse_table_ptr->getIndices().at("Values")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIndicesView().at("Values")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIsModified().at("Values")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getNotInMemory().at("Values")->getData()(i), 0);
    gpuCheckEqual(sparse_table_ptr->getShardId().at("Values")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getShardIndices().at("Values")->getData()(i), i + 1);
  }

  // Write the original data to disk, clear the data, and repeat the tests
  tensorTable.setData(tensor_values);
  tensorTable.syncDData(device);
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);

  // Re-test getSelectTensorDataAsSparseTensorTable
  sparse_table_ptr.reset();
  tensorTable.getSelectTensorDataAsSparseTensorTable(sparse_table_ptr, device);
  sparse_table_ptr->syncAxesAndIndicesHData(device);
  sparse_table_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));

  // Check for the correct dimensions
  gpuCheckEqual(sparse_table_ptr->getDimensions().at(0), nlabels1);
  gpuCheckEqual(sparse_table_ptr->getDimensions().at(1), 1);

  // Check the data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(sparse_table_ptr->getData()(i + j * nlabels + k * nlabels*nlabels), tensor_values(i, j, k));
      }
    }
  }

  // Check the Indices axes
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getName(), "Indices");
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getNLabels(), nlabels1);
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getNDimensions(), 3);
  labels1_ptr.reset();
  sparse_table_ptr->getAxes().at("Indices")->getLabelsHDataPointer(labels1_ptr);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels_values3(labels1_ptr.get(), 3, nlabels1);
  for (int i = 0; i < 3; ++i) {
    gpuCheckEqual(sparse_table_ptr->getAxes().at("Indices")->getDimensions()(i), std::to_string(i + 1));
    for (int j = 0; j < nlabels1; ++j) {
      gpuCheckEqual(labels_values3(i, j), labels1_expected(i, j));
    }
  }

  // Check the Values axes
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getName(), "Values");
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getNLabels(), 1);
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getNDimensions(), 1);

  labels2_ptr.reset();
  sparse_table_ptr->getAxes().at("Values")->getLabelsHDataPointer(labels2_ptr);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels2_values2(labels2_ptr.get(), 1, 1);
  gpuCheckEqual(labels2_values2(0, 0), 0);
  gpuCheckEqual(sparse_table_ptr->getAxes().at("Values")->getDimensions()(0), "Values");

  // Check the indices axis indices
  for (int i = 0; i < nlabels1; ++i) {
    gpuCheckEqual(sparse_table_ptr->getIndices().at("Indices")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIndicesView().at("Indices")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIsModified().at("Indices")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getNotInMemory().at("Indices")->getData()(i), 0);
    gpuCheckEqual(sparse_table_ptr->getShardId().at("Indices")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getShardIndices().at("Indices")->getData()(i), i + 1);
  }

  // Check the values axis indices
  for (int i = 0; i < 1; ++i) {
    gpuCheckEqual(sparse_table_ptr->getIndices().at("Values")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIndicesView().at("Values")->getData()(i), i + 1);
    gpuCheckEqual(sparse_table_ptr->getIsModified().at("Values")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getNotInMemory().at("Values")->getData()(i), 0);
    gpuCheckEqual(sparse_table_ptr->getShardId().at("Values")->getData()(i), 1);
    gpuCheckEqual(sparse_table_ptr->getShardIndices().at("Values")->getData()(i), i + 1);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_updateTensorDataConstantGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(i + j * nlabels + k * nlabels*nlabels));
      }
    }
  }
  tensorTable.setData(tensor_values);

  // setup the update values
  TensorDataGpuClassT<TensorArrayGpu8, char, 1> values_new(Eigen::array<Eigen::Index, 1>({ 1 }));
  values_new.setData();
  values_new.getData()(0).setTensorArray(std::to_string(100));
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 1>> values_new_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 1>>(values_new);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  values_new_ptr->syncDData(device);

  // Test update
  std::shared_ptr<TensorTable<TensorArrayGpu8<char>, Eigen::GpuDevice, 2>> values_old_ptr;
  tensorTable.updateTensorDataConstant(values_new_ptr, values_old_ptr, device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  values_old_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));

  // Test the data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(values_old_ptr->getData()(i + j * nlabels + k * nlabels*nlabels), tensor_values(i, j, k));
        gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(100)));
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
  }

  // reset is_modified attribute
  for (auto& is_modified_map : tensorTable.getIsModified()) {
    is_modified_map.second->getData() = is_modified_map.second->getData().constant(1);
  }

  // Revert the operation and test
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  values_old_ptr->syncDData(device);
  tensorTable.updateTensorDataFromSparseTensorTable(values_old_ptr, device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
  }

  // TODO: Test after a selection (see test for TensorOperation TensorUpdateConstant)

  // Write the original data to disk, clear the data, and repeat the tests
  tensorTable.setData(tensor_values);
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);

  // Test update
  values_old_ptr.reset();
  tensorTable.updateTensorDataConstant(values_new_ptr, values_old_ptr, device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  values_old_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));

  // Test the data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(values_old_ptr->getData()(i + j * nlabels + k * nlabels*nlabels), tensor_values(i, j, k));
        gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(100)));
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
  }

  // clear the data
  tensorTable.setData();
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncDData(device);

  // Revert the operation and test
  values_old_ptr->setDataStatus(false, true);
  tensorTable.updateTensorDataFromSparseTensorTable(values_old_ptr, device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeShardIndicesFromShardIDsGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 6;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2, 3, 4, 5} });
  labels2.setValues({ {0, 1, 2, 3, 4, 5} });
  labels3.setValues({ {0, 1, 2, 3, 4, 5} });
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3)));
  tensorTable.setAxes(device);

  // Reshard indices
  int shard_span = 2;
  std::map<std::string, int> shard_span_new = { {"1", shard_span}, {"2", shard_span}, {"3", shard_span} };
  tensorTable.setShardSpans(shard_span_new);
  tensorTable.setMaximumDimensions(Eigen::array<Eigen::Index, 3>({ nlabels , nlabels , nlabels }));

  // Test for the shard indices
  tensorTable.syncShardIdDData(device);
  tensorTable.syncShardIndicesDData(device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_shard_ptr;
  tensorTable.makeShardIndicesFromShardIDs(indices_shard_ptr, device);
  indices_shard_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(indices_shard_ptr->getData()(i, j, k), 1);
      }
    }
  }

  // make the expected tensor indices
  int shard_n_indices = 3;
  std::vector<int> shard_id_indices = { 0, 0, 1, 1, 2, 2 };
  Eigen::Tensor<int, 3> indices_test(nlabels, nlabels, nlabels);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        indices_test(i, j, k) = shard_id_indices.at(i) + shard_id_indices.at(j) * shard_n_indices + shard_id_indices.at(k) * shard_n_indices*shard_n_indices + 1;
      }
    }
  }

  // Test for the shard indices
  tensorTable.reShardIndices(device);
  indices_shard_ptr.reset();
  tensorTable.makeShardIndicesFromShardIDs(indices_shard_ptr, device);
  indices_shard_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        gpuCheckEqual(indices_shard_ptr->getData()(i, j, k), indices_test(i, j, k));
      }
    }
  }
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeModifiedShardIDTensorGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // Reshard indices
  tensorTable.syncAxesAndIndicesDData(device);
  int shard_span = 2;
  std::map<std::string, int> shard_span_new = { {"1", shard_span}, {"2", shard_span}, {"3", shard_span} };
  tensorTable.setShardSpans(shard_span_new);
  tensorTable.reShardIndices(device);
  tensorTable.setMaximumDimensions(Eigen::array<Eigen::Index, 3>({ nlabels , nlabels , nlabels }));

  // Test the unmodified case
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> shard_id_indices_ptr;
  tensorTable.makeModifiedShardIDTensor(shard_id_indices_ptr, device);
  shard_id_indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(shard_id_indices_ptr->getTensorSize(), 0);

  std::map<int, std::pair<Eigen::array<Eigen::Index, 3>, Eigen::array<Eigen::Index, 3>>> slice_indices;
  Eigen::array<Eigen::Index, 3> shard_data_dimensions;
  int shard_data_size = 0;
  shard_id_indices_ptr->syncDData(device);
  shard_data_size = tensorTable.makeSliceIndicesFromShardIndices(shard_id_indices_ptr, slice_indices, shard_data_dimensions, device);
  gpuCheckEqual(slice_indices.size(), 0);
  gpuCheckEqual(shard_data_size, 0);

  // Test the fully modified case
  for (auto& is_modified_map : tensorTable.getIsModified()) {
    is_modified_map.second->getData() = is_modified_map.second->getData().constant(1);
  }
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  shard_id_indices_ptr.reset();
  tensorTable.makeModifiedShardIDTensor(shard_id_indices_ptr, device);
  shard_id_indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(shard_id_indices_ptr->getTensorSize(), 8);
  for (int i = 0; i < shard_id_indices_ptr->getTensorSize(); ++i) {
    gpuCheckEqual(shard_id_indices_ptr->getData()(i), i + 1);
  }

  slice_indices.clear();
  shard_data_dimensions = Eigen::array<Eigen::Index, 3>();
  shard_id_indices_ptr->syncDData(device);
  shard_data_size = tensorTable.makeSliceIndicesFromShardIndices(shard_id_indices_ptr, slice_indices, shard_data_dimensions, device);
  std::map<int, std::pair<Eigen::array<Eigen::Index, 3>, Eigen::array<Eigen::Index, 3>>> slice_indices_test;
  slice_indices_test.emplace(1, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,0,0 }), Eigen::array<Eigen::Index, 3>({ 2,2,2 })));
  slice_indices_test.emplace(2, std::make_pair(Eigen::array<Eigen::Index, 3>({ 2,0,0 }), Eigen::array<Eigen::Index, 3>({ 1,2,2 })));
  slice_indices_test.emplace(3, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,2,0 }), Eigen::array<Eigen::Index, 3>({ 2,1,2 })));
  slice_indices_test.emplace(4, std::make_pair(Eigen::array<Eigen::Index, 3>({ 2,2,0 }), Eigen::array<Eigen::Index, 3>({ 1,1,2 })));
  slice_indices_test.emplace(5, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,0,2 }), Eigen::array<Eigen::Index, 3>({ 2,2,1 })));
  slice_indices_test.emplace(6, std::make_pair(Eigen::array<Eigen::Index, 3>({ 2,0,2 }), Eigen::array<Eigen::Index, 3>({ 1,2,1 })));
  slice_indices_test.emplace(7, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,2,2 }), Eigen::array<Eigen::Index, 3>({ 2,1,1 })));
  slice_indices_test.emplace(8, std::make_pair(Eigen::array<Eigen::Index, 3>({ 2,2,2 }), Eigen::array<Eigen::Index, 3>({ 1,1,1 })));
  Eigen::array<Eigen::Index, 3> shard_data_dimensions_test = { nlabels, nlabels, nlabels };
  int iter = 1;
  for (const auto& slice_indices_map : slice_indices) {
    gpuCheckEqual(slice_indices_map.first, iter);
    gpuCheckEqual(slice_indices_map.second.first, slice_indices_test.at(slice_indices_map.first).first);
    gpuCheckEqual(slice_indices_map.second.second, slice_indices_test.at(slice_indices_map.first).second);
    ++iter;
  }
  for (int i = 0; i < 3; ++i) {
    gpuCheckEqual(shard_data_dimensions.at(i), shard_data_dimensions_test.at(i));
  }
  gpuCheckEqual(shard_data_size, nlabels * nlabels * nlabels);

  // Test the partially modified case
  for (auto& is_modified_map : tensorTable.getIsModified()) {
    for (int i = 0; i < nlabels; ++i) {
      if (i < shard_span)
        is_modified_map.second->getData()(i) = 1;
      else
        is_modified_map.second->getData()(i) = 0;
    }
  }
  tensorTable.setIsModifiedDataStatus(true, false);
  tensorTable.syncIsModifiedDData(device);
  shard_id_indices_ptr.reset();
  tensorTable.makeModifiedShardIDTensor(shard_id_indices_ptr, device);
  shard_id_indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(shard_id_indices_ptr->getTensorSize(), 1);
  for (int i = 0; i < shard_id_indices_ptr->getTensorSize(); ++i) {
    gpuCheckEqual(shard_id_indices_ptr->getData()(i), i + 1);
  }

  slice_indices.clear();
  shard_data_dimensions = Eigen::array<Eigen::Index, 3>();
  shard_id_indices_ptr->syncDData(device);
  shard_data_size = tensorTable.makeSliceIndicesFromShardIndices(shard_id_indices_ptr, slice_indices, shard_data_dimensions, device);
  slice_indices_test.clear();
  slice_indices_test.emplace(1, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,0,0 }), Eigen::array<Eigen::Index, 3>({ 2,2,2 })));
  shard_data_dimensions_test = Eigen::array<Eigen::Index, 3>({ 2, 2, 2 });
  iter = 1;
  for (const auto& slice_indices_map : slice_indices) {
    gpuCheckEqual(slice_indices_map.first, iter);
    gpuCheckEqual(slice_indices_map.second.first, slice_indices_test.at(slice_indices_map.first).first);
    gpuCheckEqual(slice_indices_map.second.second, slice_indices_test.at(slice_indices_map.first).second);
    ++iter;
  }
  for (int i = 0; i < 3; ++i) {
    gpuCheckEqual(shard_data_dimensions.at(i), shard_data_dimensions_test.at(i));
  }
  gpuCheckEqual(shard_data_size, 8);
  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeNotInMemoryShardIDTensorGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // Reshard indices
  tensorTable.syncAxesAndIndicesDData(device);
  int shard_span = 2;
  std::map<std::string, int> shard_span_new = { {"1", shard_span}, {"2", shard_span}, {"3", shard_span} };
  tensorTable.setShardSpans(shard_span_new);
  tensorTable.reShardIndices(device);
  tensorTable.setMaximumDimensions(Eigen::array<Eigen::Index, 3>({ nlabels , nlabels , nlabels }));

  // Test all in memory case and all selected case
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    in_memory_map.second->getData() = in_memory_map.second->getData().constant(0);
  }
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> shard_id_indices_ptr;
  tensorTable.makeNotInMemoryShardIDTensor(shard_id_indices_ptr, device);
  shard_id_indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(shard_id_indices_ptr->getTensorSize(), 0);

  std::map<int, std::pair<Eigen::array<Eigen::Index, 3>, Eigen::array<Eigen::Index, 3>>> slice_indices;
  Eigen::array<Eigen::Index, 3> shard_data_dimensions;
  int shard_data_size = 0;
  shard_id_indices_ptr->syncDData(device);
  shard_data_size = tensorTable.makeSliceIndicesFromShardIndices(shard_id_indices_ptr, slice_indices, shard_data_dimensions, device);
  gpuCheckEqual(slice_indices.size(), 0);
  gpuCheckEqual(shard_data_size, 0);

  // Test not all in memory case and none selected case
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    in_memory_map.second->getData() = in_memory_map.second->getData().constant(1);
  }
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  for (auto& indices_view_map : tensorTable.getIndicesView()) {
    indices_view_map.second->getData() = indices_view_map.second->getData().constant(0);
  }
  tensorTable.setIndicesViewDataStatus(true, false);
  tensorTable.syncIndicesViewDData(device);
  shard_id_indices_ptr.reset();
  tensorTable.makeNotInMemoryShardIDTensor(shard_id_indices_ptr, device);
  shard_id_indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(shard_id_indices_ptr->getTensorSize(), 0);

  slice_indices.clear();
  shard_data_dimensions = Eigen::array<Eigen::Index, 3>();
  shard_id_indices_ptr->syncDData(device);
  shard_data_size = tensorTable.makeSliceIndicesFromShardIndices(shard_id_indices_ptr, slice_indices, shard_data_dimensions, device);
  gpuCheckEqual(slice_indices.size(), 0);
  gpuCheckEqual(shard_data_size, 0);

  // Test all not in memory case and all selected case
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    in_memory_map.second->getData() = in_memory_map.second->getData().constant(1);
  }
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.resetIndicesView("1", device);
  tensorTable.resetIndicesView("2", device);
  tensorTable.resetIndicesView("3", device);
  //tensorTable.setIndicesViewDataStatus(true, false);
  //tensorTable.syncIndicesViewDData(device);
  shard_id_indices_ptr.reset();
  tensorTable.makeNotInMemoryShardIDTensor(shard_id_indices_ptr, device);
  shard_id_indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(shard_id_indices_ptr->getTensorSize(), 8);
  for (int i = 0; i < shard_id_indices_ptr->getTensorSize(); ++i) {
    gpuCheckEqual(shard_id_indices_ptr->getData()(i), i + 1);
  }

  slice_indices.clear();
  shard_data_dimensions = Eigen::array<Eigen::Index, 3>();
  shard_id_indices_ptr->syncDData(device);
  shard_data_size = tensorTable.makeSliceIndicesFromShardIndices(shard_id_indices_ptr, slice_indices, shard_data_dimensions, device);
  std::map<int, std::pair<Eigen::array<Eigen::Index, 3>, Eigen::array<Eigen::Index, 3>>> slice_indices_test;
  slice_indices_test.emplace(1, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,0,0 }), Eigen::array<Eigen::Index, 3>({ 2,2,2 })));
  slice_indices_test.emplace(2, std::make_pair(Eigen::array<Eigen::Index, 3>({ 2,0,0 }), Eigen::array<Eigen::Index, 3>({ 1,2,2 })));
  slice_indices_test.emplace(3, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,2,0 }), Eigen::array<Eigen::Index, 3>({ 2,1,2 })));
  slice_indices_test.emplace(4, std::make_pair(Eigen::array<Eigen::Index, 3>({ 2,2,0 }), Eigen::array<Eigen::Index, 3>({ 1,1,2 })));
  slice_indices_test.emplace(5, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,0,2 }), Eigen::array<Eigen::Index, 3>({ 2,2,1 })));
  slice_indices_test.emplace(6, std::make_pair(Eigen::array<Eigen::Index, 3>({ 2,0,2 }), Eigen::array<Eigen::Index, 3>({ 1,2,1 })));
  slice_indices_test.emplace(7, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,2,2 }), Eigen::array<Eigen::Index, 3>({ 2,1,1 })));
  slice_indices_test.emplace(8, std::make_pair(Eigen::array<Eigen::Index, 3>({ 2,2,2 }), Eigen::array<Eigen::Index, 3>({ 1,1,1 })));
  Eigen::array<Eigen::Index, 3> shard_data_dimensions_test = { nlabels, nlabels, nlabels };
  int iter = 1;
  for (const auto& slice_indices_map : slice_indices) {
    gpuCheckEqual(slice_indices_map.first, iter);
    gpuCheckEqual(slice_indices_map.second.first, slice_indices_test.at(slice_indices_map.first).first);
    gpuCheckEqual(slice_indices_map.second.second, slice_indices_test.at(slice_indices_map.first).second);
    ++iter;
  }
  for (int i = 0; i < 3; ++i) {
    gpuCheckEqual(shard_data_dimensions.at(i), shard_data_dimensions_test.at(i));
  }
  gpuCheckEqual(shard_data_size, nlabels * nlabels * nlabels);

  // Test the partially in memory case and all selected case
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    for (int i = 0; i < nlabels; ++i) {
      if (i < shard_span)
        in_memory_map.second->getData()(i) = 1;
      else
        in_memory_map.second->getData()(i) = 0;
    }
  }
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  shard_id_indices_ptr.reset();
  tensorTable.makeNotInMemoryShardIDTensor(shard_id_indices_ptr, device);
  shard_id_indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(shard_id_indices_ptr->getTensorSize(), 1);
  for (int i = 0; i < shard_id_indices_ptr->getTensorSize(); ++i) {
    gpuCheckEqual(shard_id_indices_ptr->getData()(i), i + 1);
  }

  slice_indices.clear();
  shard_data_dimensions = Eigen::array<Eigen::Index, 3>();
  shard_id_indices_ptr->syncDData(device);
  shard_data_size = tensorTable.makeSliceIndicesFromShardIndices(shard_id_indices_ptr, slice_indices, shard_data_dimensions, device);
  slice_indices_test.clear();
  slice_indices_test.emplace(1, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,0,0 }), Eigen::array<Eigen::Index, 3>({ 2,2,2 })));
  shard_data_dimensions_test = Eigen::array<Eigen::Index, 3>({ 2, 2, 2 });
  iter = 1;
  for (const auto& slice_indices_map : slice_indices) {
    gpuCheckEqual(slice_indices_map.first, iter);
    gpuCheckEqual(slice_indices_map.second.first, slice_indices_test.at(slice_indices_map.first).first);
    gpuCheckEqual(slice_indices_map.second.second, slice_indices_test.at(slice_indices_map.first).second);
    ++iter;
  }
  for (int i = 0; i < 3; ++i) {
    gpuCheckEqual(shard_data_dimensions.at(i), shard_data_dimensions_test.at(i));
  }
  gpuCheckEqual(shard_data_size, 8);
  
  // Test the partially in memory case and partially selected case
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    for (int i = 0; i < nlabels; ++i) {
      if (i < shard_span)
        in_memory_map.second->getData()(i) = 1;
      else
        in_memory_map.second->getData()(i) = 0;
    }
  }
  tensorTable.setNotInMemoryDataStatus(true, false);
  tensorTable.syncNotInMemoryDData(device);
  for (auto& indices_view_map : tensorTable.getIndicesView()) {
    for (int i = 0; i < nlabels; ++i) {
      if (i < 1)
        indices_view_map.second->getData()(i) = i + 1;
      else
        indices_view_map.second->getData()(i) = 0;
    }
  }
  tensorTable.setIndicesViewDataStatus(true, false);
  tensorTable.syncIndicesViewDData(device);
  shard_id_indices_ptr.reset();
  tensorTable.makeNotInMemoryShardIDTensor(shard_id_indices_ptr, device);
  shard_id_indices_ptr->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(shard_id_indices_ptr->getTensorSize(), 1);
  for (int i = 0; i < shard_id_indices_ptr->getTensorSize(); ++i) {
    gpuCheckEqual(shard_id_indices_ptr->getData()(i), i + 1);
  }

  slice_indices.clear();
  shard_data_dimensions = Eigen::array<Eigen::Index, 3>();
  shard_id_indices_ptr->syncDData(device);
  shard_data_size = tensorTable.makeSliceIndicesFromShardIndices(shard_id_indices_ptr, slice_indices, shard_data_dimensions, device);
  slice_indices_test.clear();
  slice_indices_test.emplace(1, std::make_pair(Eigen::array<Eigen::Index, 3>({ 0,0,0 }), Eigen::array<Eigen::Index, 3>({ 2,2,2 })));
  shard_data_dimensions_test = Eigen::array<Eigen::Index, 3>({ 2, 2, 2 });
  iter = 1;
  for (const auto& slice_indices_map : slice_indices) {
    gpuCheckEqual(slice_indices_map.first, iter);
    gpuCheckEqual(slice_indices_map.second.first, slice_indices_test.at(slice_indices_map.first).first);
    gpuCheckEqual(slice_indices_map.second.second, slice_indices_test.at(slice_indices_map.first).second);
    ++iter;
  }
  for (int i = 0; i < 3; ++i) {
    gpuCheckEqual(shard_data_dimensions.at(i), shard_data_dimensions_test.at(i));
  }
  gpuCheckEqual(shard_data_size, 8);

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_makeTensorTableShardFilenameGpu()
{
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;
  gpuCheckEqual(tensorTable.makeTensorTableShardFilename("dir/", "table1", 1), "dir/table1_1.tts");
}

void test_storeAndLoadBinaryGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k).setTensorArray(std::to_string(i + j * nlabels + k * nlabels * nlabels));
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // Reshard indices
  int shard_span = 2;
  std::map<std::string, int> shard_span_new = { {"1", shard_span}, {"2", shard_span}, {"3", shard_span} };
  tensorTable.setShardSpans(shard_span_new);
  tensorTable.reShardIndices(device);

  // Test store/load for the case of all `is_modified`, all not `not_in_memory`, and selected `indices_view`
  tensorTable.storeTensorTableBinary("", device);

  // Test for the in_memory and is_modified attributes
  tensorTable.syncIsModifiedHData(device);
  tensorTable.syncNotInMemoryHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 0);
  }

  // Reset the in_memory values
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    in_memory_map.second->getData() = in_memory_map.second->getData().constant(1);
  }
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncNotInMemoryDData(device);

  // Load the data
  tensorTable.loadTensorTableBinary("", device);

  // Test for the in_memory and is_modified attributes
  tensorTable.syncIsModifiedHData(device);
  tensorTable.syncNotInMemoryHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 0);
  }

  // Test for the original data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }

  // Test store/load for the case of partially `is_modified`, all not `not_in_memory`, and partially selected `indices_view`
  for (auto& is_modified_map : tensorTable.getIsModified()) { // Shards 1-7
    for (int i = 0; i < nlabels; ++i) {
      if (i < 2)
        is_modified_map.second->getData()(i) = 1;
      else
        is_modified_map.second->getData()(i) = 0;
    }
  }
  for (auto& indices_view_map : tensorTable.getIndicesView()) { // Shard 1
    for (int i = 0; i < nlabels; ++i) {
      if (i < 1)
        indices_view_map.second->getData()(i) = i + 1;
      else
        indices_view_map.second->getData()(i) = 0;
    }
  }
  tensorTable.syncIsModifiedDData(device);
  tensorTable.setIndicesViewDataStatus(true, false);
  tensorTable.syncIndicesViewDData(device);
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.syncDData(device);

  // Test for the in_memory, is_modified, and indices_view attributes before store
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    if (i < 2) {
      gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
    }
    else {
      gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 0);
    }
    if (i < 1) {
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    }
    else {
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), 0);
    }
  }

  // Test for the in_memory, is_modified, and indices_view attributes after store
  tensorTable.storeTensorTableBinary("", device);
  tensorTable.syncIsModifiedHData(device);
  tensorTable.syncNotInMemoryHData(device);
  tensorTable.syncIndicesViewHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 0);
    if (i < 1) {
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), i + 1);
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), i + 1);
    }
    else {
      gpuCheckEqual(tensorTable.getIndicesView().at("1")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getIndicesView().at("2")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getIndicesView().at("3")->getData()(i), 0);
    }
  }

  // Reset the in_memory values and Zero the TensorData
  for (auto& in_memory_map : tensorTable.getNotInMemory()) {
    in_memory_map.second->getData() = in_memory_map.second->getData().constant(1);
  }
  tensorTable.syncNotInMemoryDData(device);
  tensorTable.syncIsModifiedDData(device);
  tensorTable.syncIndicesViewDData(device);
  tensorTable.getData() = tensorTable.getData().constant(TensorArrayGpu8<char>("0"));
  tensorTable.syncDData(device);

  // Load the data
  tensorTable.loadTensorTableBinary("", device);

  // Test for the in_memory and is_modified attributes
  tensorTable.syncIsModifiedHData(device);
  tensorTable.syncNotInMemoryHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int i = 0; i < nlabels; ++i) {
    if (i < shard_span) {
      gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
      gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    }
    else {
      gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 1);
      gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 1);
    }
  }

  // Test for the original data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        if (i < shard_span && j < shard_span && k < shard_span)
          gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k));
        else
          gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>("0"));
      }
    }
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_storeAndLoadTensorTableAxesGpu()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable1;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels1 = 2, nlabels2 = 3, nlabels3 = 5;
  Eigen::Tensor<int, 2> labels1(1, nlabels1), labels2(1, nlabels2), labels3(1, nlabels3);
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable1.addTensorAxis(axis_1_ptr);
  tensorTable1.addTensorAxis(axis_2_ptr);
  tensorTable1.addTensorAxis(axis_3_ptr);
  tensorTable1.setAxes(device);

  // sync the tensorTable
  tensorTable1.syncAxesAndIndicesDData(device);

  // Store the axes
  tensorTable1.storeTensorTableAxesBinary("", device);

  // Remake empty axes
  tensorTable1.clear();
  tensorTable1.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", 1, nlabels1)));
  tensorTable1.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", 1, nlabels2)));
  tensorTable1.addTensorAxis((std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>>)std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", 1, nlabels3)));
  tensorTable1.setAxes(device);

  // sync the tensorTable
  tensorTable1.syncAxesAndIndicesDData(device);

  // Load the axes
  tensorTable1.loadTensorTableAxesBinary("", device);

  // Test for the correct axes data
  std::shared_ptr<int[]> labels1_ptr;
  tensorTable1.getAxes().at("1")->getLabelsHDataPointer(labels1_ptr);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels1_values(labels1_ptr.get(), 1, nlabels1);
  for (int j = 0; j < nlabels1; ++j) {
    gpuCheckEqual(labels1_values(0, j), labels1(0, j));
  }

  std::shared_ptr<int[]> labels2_ptr;
  tensorTable1.getAxes().at("2")->getLabelsHDataPointer(labels2_ptr);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels2_values(labels2_ptr.get(), 1, nlabels2);
  for (int j = 0; j < nlabels2; ++j) {
    gpuCheckEqual(labels2_values(0, j), labels2(0, j));
  }

  std::shared_ptr<int[]> labels3_ptr;
  tensorTable1.getAxes().at("3")->getLabelsHDataPointer(labels3_ptr);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels3_values(labels3_ptr.get(), 1, nlabels3);
  for (int j = 0; j < nlabels3; ++j) {
    gpuCheckEqual(labels3_values(0, j), labels3(0, j));
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_getCsvDataRowGpuClassT()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  std::vector<std::string> row_0_test, row_1_test, row_4_test;
  int row_iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        int value = i + j * nlabels + k * nlabels*nlabels;
        tensor_values(i, j, k) = TensorArrayGpu8<char>(std::to_string(value));
        if (row_iter == 0) {
          row_0_test.push_back(std::to_string(value));
        }
        else if (row_iter == 1) {
          row_1_test.push_back(std::to_string(value));
        }
        else if (row_iter == 4) {
          row_4_test.push_back(std::to_string(value));
        }
      }
      ++row_iter;
    }
  }
  tensorTable.setData(tensor_values);

  // Test getCsvDataRow
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  // TODO: also test char and tensorArray types
  std::vector<std::string> row_0 = tensorTable.getCsvDataRow(0);
  std::vector<std::string> row_1 = tensorTable.getCsvDataRow(1);
  std::vector<std::string> row_4 = tensorTable.getCsvDataRow(4);
  gpuCheckEqual(row_0.size(), nlabels);
  gpuCheckEqual(row_1.size(), nlabels);
  gpuCheckEqual(row_4.size(), nlabels);
  std::cout << "Passing/Failing test_getCsvDataRowGpuClassT\n" << std::endl;
  for (int i = 0; i < nlabels; ++i) {
    std::cout << "row_0.at(" << i << "): [Test] " << row_0.at(i) << " [Expected] " << row_0_test.at(i) << std::endl;
    std::cout << "row_1.at(" << i << "): [Test] " << row_1.at(i) << " [Expected] " << row_1_test.at(i) << std::endl;
    std::cout << "row_4.at(" << i << "): [Test] " << row_4.at(i) << " [Expected] " << row_4_test.at(i) << std::endl;
    //gpuCheckEqual(row_0.at(i), row_0_test.at(i)); // FIXME: issue with \0?
    //gpuCheckEqual(row_1.at(i), row_1_test.at(i));
    //gpuCheckEqual(row_4.at(i), row_4_test.at(i));
  }

  // Make the expected labels row values
  std::map<std::string, std::vector<std::string>> labels_row_0_test = { {"2", {"0"}}, {"3", {"0"}} };
  std::map<std::string, std::vector<std::string>> labels_row_1_test = { {"2", {"1"}}, {"3", {"0"}} };
  std::map<std::string, std::vector<std::string>> labels_row_4_test = { {"2", {"1"}}, {"3", {"1"}} };

  // Test getCsvAxesLabelsRow
  std::map<std::string, std::vector<std::string>> labels_row_0 = tensorTable.getCsvAxesLabelsRow(0);
  std::map<std::string, std::vector<std::string>> labels_row_1 = tensorTable.getCsvAxesLabelsRow(1);
  std::map<std::string, std::vector<std::string>> labels_row_4 = tensorTable.getCsvAxesLabelsRow(4);
  gpuCheck(labels_row_0.size(), 2);
  gpuCheck(labels_row_1.size(), 2);
  gpuCheck(labels_row_4.size(), 2);
  for (int i = 2; i < 4; ++i) {
    std::string axis_name = std::to_string(i);
    for (int j = 0; j < 1; ++j) {
      gpuCheckEqual(labels_row_0.at(axis_name).at(j), labels_row_0_test.at(axis_name).at(j));
      gpuCheckEqual(labels_row_1.at(axis_name).at(j), labels_row_1_test.at(axis_name).at(j));
      gpuCheckEqual(labels_row_4.at(axis_name).at(j), labels_row_4_test.at(axis_name).at(j));
    }
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_insertIntoTableFromCsvGpuClassT()
{
  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, 1), labels3(1, 1);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0} });
  labels3.setValues({ {0} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data, the new tensor data from csv, and the new axes labels from csv
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, 1, 1 }));
  Eigen::Tensor<std::string, 2> new_values_str(Eigen::array<Eigen::Index, 2>({ nlabels, 8 }));
  Eigen::Tensor<std::string, 2> labels_2_str(1, 8);
  Eigen::Tensor<std::string, 2> labels_3_str(1, 8);
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        if (j == 0 && k == 0) {
          tensor_values(i, j, k) = TensorArrayGpu8<char>(std::to_string(i + j * nlabels + k * nlabels*nlabels));
        }
        else {
          int index = j + k * nlabels - 1;
          new_values_str(i, index) = std::to_string(i + j * nlabels + k * nlabels*nlabels);
          labels_2_str(0, index) = std::to_string(j);
          labels_3_str(0, index) = std::to_string(k);
        }
      }
    }
  }
  tensorTable.setData(tensor_values);

  // setup the new axis labels from csv
  std::map<std::string, Eigen::Tensor<std::string, 2>> labels_new_str;
  labels_new_str.emplace("2", labels_2_str);
  labels_new_str.emplace("3", labels_3_str);

  // Test
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);
  tensorTable.insertIntoTableFromCsv(labels_new_str, new_values_str, device);
  tensorTable.syncAxesAndIndicesHData(device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));

  // Test for the tensor data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), TensorArrayGpu8<char>(std::to_string(i + j * nlabels + k * nlabels*nlabels)));
      }
    }
  }

  // Test for the axis labels
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(axis_1_ptr->getLabels()(0, i), i);
    gpuCheckEqual(axis_2_ptr->getLabels()(0, i), i);
    gpuCheckEqual(axis_3_ptr->getLabels()(0, i), i);
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    gpuCheckEqual(tensorTable.getNotInMemory().at("1")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("2")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getNotInMemory().at("3")->getData()(i), 0);
    gpuCheckEqual(tensorTable.getIsModified().at("1")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("2")->getData()(i), 1);
    gpuCheckEqual(tensorTable.getIsModified().at("3")->getData()(i), 1);
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_applyFunctorGpu()
{
  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = TensorArrayGpu8<char>(std::to_string(i + j * nlabels + k * nlabels * nlabels));
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // test using the copy functor
  TensorDataGpuClassT<TensorArrayGpu8, char, 3> results(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  results.setData();
  results.syncDData(device);
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 3>> results_ptr = std::make_shared<TensorDataGpuClassT<TensorArrayGpu8, char, 3>>(results);
  TensorDataCopy<TensorArrayGpu8<char>, Eigen::GpuDevice, 3> sumReduction(results_ptr);
  tensorTable.applyFunctor(sumReduction, device);
  results.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(results.getData()(i, j, k), tensor_values(i, j, k)); // no change
      }
    }
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_reduceTensorDataGpu()
{
  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = TensorArrayGpu8<char>(std::to_string(i + j * nlabels + k * nlabels * nlabels));
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // test using the different reduction functions
  tensorTable.reduceTensorData(reductionFunctions::COUNT, device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(tensorTable.getData()(0, 0, 0), tensor_values(0,0,0)); // no change
  gpuCheckEqual(tensorTable.getData()(nlabels - 1, nlabels - 1, nlabels - 1), tensor_values(nlabels - 1, nlabels - 1, nlabels - 1)); // no change

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_scanTensorDataGpu()
{
  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = TensorArrayGpu8<char>(std::to_string(i + j * nlabels + k * nlabels * nlabels));
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncAxesAndIndicesDData(device);
  tensorTable.syncDData(device);

  // test using the different reduction functions
  tensorTable.scanTensorData({ "1", "2" }, scanFunctions::CUMSUM, device);
  tensorTable.syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(tensorTable.getData()(i, j, k), tensor_values(i, j, k)); // no change
      }
    }
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

void test_copyGpu()
{
  // Initialize the device
  cudaStream_t stream;
  gpuErrchk(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking));
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the table
  TensorTableGpuClassT<TensorArrayGpu8, char, 3> tensorTable;

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_1_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("1", dimensions1, labels1));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_2_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("2", dimensions2, labels2));
  std::shared_ptr<TensorAxis<int, Eigen::GpuDevice>> axis_3_ptr = std::make_shared<TensorAxisGpuPrimitiveT<int>>(TensorAxisGpuPrimitiveT<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes(device);

  // setup the tensor data
  Eigen::Tensor<TensorArrayGpu8<char>, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = TensorArrayGpu8<char>(std::to_string(i + j * nlabels + k * nlabels * nlabels));
      }
    }
  }
  tensorTable.setData(tensor_values);

  // test copy
  auto tensorTableCopy = tensorTable.copyToHost(device);
  tensorTableCopy->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(*(tensorTableCopy.get()), tensorTable);
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(tensorTableCopy->getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }
  auto tensorTableCopy2 = tensorTable.copyToDevice(device);
  tensorTableCopy2->syncHData(device);
  gpuErrchk(cudaStreamSynchronize(stream));
  gpuCheckEqual(*(tensorTableCopy2.get()), tensorTable);
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        gpuCheckEqual(tensorTableCopy2->getData()(i, j, k), tensor_values(i, j, k));
      }
    }
  }

  gpuErrchk(cudaStreamDestroy(stream));
}

int main(int argc, char** argv)
{	
  test_constructorGpu();
  test_destructorGpu(); 
  test_constructorNameAndAxesGpu();
  test_gettersAndSettersGpu();
  test_initDataGpuClassT();
  test_reShardIndicesGpu();
  test_zeroIndicesViewAndResetIndicesViewGpu();
  test_selectIndicesView1Gpu();
  test_selectIndicesView2Gpu();
  test_broadcastSelectIndicesViewGpu();
  test_extractTensorDataGpuClassT();
  test_selectTensorIndicesGpu();
  test_applyIndicesSelectToIndicesViewGpu();
  test_whereIndicesViewData1Gpu();
  test_whereIndicesViewData2Gpu();
  test_sliceTensorForSortGpu();
  test_sortIndicesViewData1Gpu();
  test_sortIndicesViewData2Gpu();
  test_makeSelectIndicesFromIndicesViewGpu();
  test_getSelectTensorDataFromIndicesViewGpu();
  test_selectTensorDataGpuClassT();
  test_makeSortIndicesViewFromIndicesViewGpu();
  test_sortTensorDataGpuClassT();
  test_updateSelectTensorDataValues1Gpu();
  test_updateSelectTensorDataValues2Gpu();
	test_updateTensorDataValuesGpu();
  test_makeAppendIndicesGpu();
  test_appendToIndicesGpu();
  test_appendToAxis1Gpu();
  test_appendToAxis2Gpu();
  test_makeIndicesViewSelectFromIndicesGpu();
  test_deleteFromIndicesGpu();
  test_makeSelectIndicesFromIndicesGpu();
  test_deleteFromAxisGpu();
  test_makeIndicesFromIndicesViewGpu();
  test_insertIntoAxisGpu();
  test_makeSparseAxisLabelsFromIndicesViewGpu();
  test_makeSparseTensorTableGpu();
  test_getSelectTensorDataAsSparseTensorTableGpu();
  test_updateTensorDataConstantGpu();
  test_makeShardIndicesFromShardIDsGpu();
  test_makeModifiedShardIDTensorGpu();
  test_makeNotInMemoryShardIDTensorGpu();
  test_makeTensorTableShardFilenameGpu();
  test_storeAndLoadBinaryGpu();
  test_storeAndLoadTensorTableAxesGpu();
  test_getCsvDataRowGpuClassT();
  test_insertIntoTableFromCsvGpuClassT();
  test_applyFunctorGpu();
  test_reduceTensorDataGpu();
  test_scanTensorDataGpu();
  test_copyGpu();
  return 0;
}

#endif