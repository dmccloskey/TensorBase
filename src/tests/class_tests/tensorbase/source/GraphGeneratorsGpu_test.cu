
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA
#include <TensorBase/core/GraphGeneratorsGpu.h>

using namespace TensorBase;
using namespace std;

void test_kroneckerGraphGeneratorMakeKroneckerGraphGpu()
{
  // init the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // test making the kronecker graph
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> indices;
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 2>> weights;
  KroneckerGraphGeneratorGpu<int, float> graph_generator;
  graph_generator.makeKroneckerGraph(4, 8, indices, weights, device);
  Eigen::array<Eigen::Index, 2> indices_dims = { int(std::pow(2, 4) * 8), 2 };
  assert(indices->getDimensions() == indices_dims);
  Eigen::array<Eigen::Index, 2> weights_dims = { int(std::pow(2, 4) * 8), 1 };
  assert(weights->getDimensions() == weights_dims);

  // test getting the node/link ids for the entire graph
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> node_ids;
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> link_ids;
  graph_generator.getNodeAndLinkIds(0, std::pow(2, 4) * 8, indices, node_ids, link_ids, device);
  link_ids->syncHAndDData(device);
  node_ids->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  Eigen::array<Eigen::Index, 1> link_ids_dims = { int(std::pow(2, 4) * 8) };
  assert(link_ids->getDimensions() == link_ids_dims);
  for (int i = 0; i < link_ids_dims.at(0); ++i) {
    assert(link_ids->getData()(i) == i);
  }
  Eigen::array<Eigen::Index, 1> node_ids_dims = { int(std::pow(2, 4) * 8) };
  assert(node_ids->getDimensions().at(0) <= std::pow(2, 4));

  // test getting the node/link ids
  node_ids.reset();
  link_ids.reset();
  graph_generator.getNodeAndLinkIds(8, 16, indices, node_ids, link_ids, device);
  link_ids->syncHAndDData(device);
  node_ids->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  link_ids_dims = Eigen::array<Eigen::Index, 1>({ 16 });
  assert(link_ids->getDimensions() == link_ids_dims);
  for (int i = 0; i < link_ids_dims.at(0); ++i) {
    assert(link_ids->getData()(i) == 8 + i);
  }
  assert(node_ids->getDimensions().at(0) <= 16);

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_BinaryTreeGraphGeneratorMakeBinaryTree()
{
  // init the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);
  const int depth = 3;
  const int n_nodes = std::pow(2, depth) - 1;
  const int n_links = std::pow(2, depth) - 2;

  // test making the Binary Tree graph
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> indices;
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 2>> weights;
  BinaryTreeGraphGeneratorGpu<int, float> graph_generator;
  graph_generator.makeBinaryTree(depth, indices, weights, device);
  indices->syncHAndDData(device);
  weights->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  Eigen::array<Eigen::Index, 2> indices_dims = { n_links, 2 };
  assert(indices->getDimensions() == indices_dims);
  std::vector<int> expected_in_nodes = { 0,0,1,1,2,2,3,3,4,4,5,5,6,6,7,7 };
  for (int i = 0; i < n_links; ++i) {
    assert(indices->getData()(i, 0) == expected_in_nodes.at(i));
    if (i % 2 == 0) assert(indices->getData()(i, 1) == expected_in_nodes.at(i) * 2 + 1);
    else assert(indices->getData()(i, 1) == expected_in_nodes.at(i) * 2 + 2);
  }
  Eigen::array<Eigen::Index, 2> weights_dims = { n_links, 1 };
  assert(weights->getDimensions() == weights_dims);

  // test getting the node/link ids for the entire graph
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> node_ids;
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> link_ids;
  graph_generator.getNodeAndLinkIds(0, n_links, indices, node_ids, link_ids, device);
  node_ids->syncHAndDData(device);
  link_ids->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(link_ids->getDimensions().at(0) == n_links);
  for (int i = 0; i < n_links; ++i) {
    assert(link_ids->getData()(i) == i);
  }
  assert(node_ids->getDimensions().at(0) == n_nodes);
  for (int i = 0; i < n_nodes; ++i) {
    assert(node_ids->getData()(i) == i);
  }

  // test getting the node/link ids for a subset
  node_ids.reset();
  link_ids.reset();
  graph_generator.getNodeAndLinkIds(2, 4, indices, node_ids, link_ids, device);
  node_ids->syncHAndDData(device);
  link_ids->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(link_ids->getDimensions().at(0) == 4);
  for (int i = 0; i < 4; ++i) {
    assert(link_ids->getData()(i) == 2 + i);
  }
  assert(node_ids->getDimensions().at(0) == 6);
  for (int i = 0; i < 6; ++i) {
    assert(node_ids->getData()(i) == i + 1);
  }

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

int main(int argc, char** argv)
{
  assert(cudaDeviceReset() == cudaSuccess);
  test_kroneckerGraphGeneratorMakeKroneckerGraphGpu();
  test_BinaryTreeGraphGeneratorMakeBinaryTree();
  return 0;
}
#endif