
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA
#include <TensorBase/benchmarks/BenchmarkGraphGpu.h>

using namespace TensorBase;
using namespace TensorBaseBenchmarks;
using namespace std;

void test_InsertUpdateDeleteGpu()
{
  // Parameters for the test
  std::string data_dir = "";
  const int scale = 8;
  const int edge_factor = 16;
  const bool in_memory = true;
  const int data_size = std::pow(2, scale) * edge_factor;
  const double shard_span_perc = 1;
  const int n_engines = 1;
  const int dim_span = std::pow(data_size, 0.25);

  // Setup the Benchmarking suite
  BenchmarkGraph1LinkGpu benchmark_1_link;

  // Setup the GraphTensorCollectionGenerator
  GraphTensorCollectionGeneratorGpu tensor_collection_generator;

  // Setup the device
  cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);

  // Make the nD TensorTables
  std::shared_ptr<TensorCollection<Eigen::GpuDevice>> n_dim_tensor_collection = tensor_collection_generator.makeTensorCollection(scale, edge_factor, shard_span_perc, device);

  // Setup the transaction manager
  TransactionManager<Eigen::GpuDevice> transaction_manager;
  transaction_manager.setMaxOperations(data_size + 1);
  transaction_manager.setTensorCollection(n_dim_tensor_collection);

  // Test the initial tensor collection
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("2_columns")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("2_columns")->getNLabels() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardSpans().at("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardSpans().at("1_links") == TensorCollectionShardHelper::round_1(data_size, shard_span_perc));
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getMaxDimSizeFromAxisName("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getMaxDimSizeFromAxisName("1_links") == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("2_columns")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("2_columns")->getNLabels() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getShardSpans().at("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getShardSpans().at("1_links") == TensorCollectionShardHelper::round_1(data_size, shard_span_perc));
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getMaxDimSizeFromAxisName("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getMaxDimSizeFromAxisName("1_links") == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("2_columns")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("2_columns")->getNLabels() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("1_nodes")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("1_nodes")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getShardSpans().at("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getShardSpans().at("1_links") == TensorCollectionShardHelper::round_1(data_size, shard_span_perc));
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getMaxDimSizeFromAxisName("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getMaxDimSizeFromAxisName("1_links") == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("2_columns")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("2_columns")->getNLabels() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getShardSpans().at("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getShardSpans().at("1_links") == TensorCollectionShardHelper::round_1(data_size, shard_span_perc));
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getMaxDimSizeFromAxisName("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getMaxDimSizeFromAxisName("1_links") == data_size);

  // Make the expected tensor axes labels and tensor data after insert
  benchmark_1_link.makeKroneckerGraph(scale, edge_factor, device);
  GraphManagerSparseIndicesGpu<int, float, int, int> graph_manager_sparse_indices(false);
  GraphManagerWeightsGpu<int, float, int, float> graph_manager_weights(false);
  GraphManagerNodePropertyGpu<int, float, int, TensorArrayGpu8<char>> graph_manager_node_property(false);
  GraphManagerLinkPropertyGpu<int, float, int, TensorArrayGpu8<char>> graph_manager_link_property(false);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_sparse_indices_ptr;
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> values_sparse_indices_ptr;
  graph_manager_sparse_indices.getInsertData(0, data_size, labels_sparse_indices_ptr, values_sparse_indices_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_labels_ptr;
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 2>> values_labels_ptr;
  graph_manager_weights.getInsertData(0, data_size, labels_labels_ptr, values_labels_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_node_property_ptr;
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 2>> values_node_property_ptr;
  graph_manager_node_property.getInsertData(0, data_size, labels_node_property_ptr, values_node_property_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_link_property_ptr;
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 2>> values_link_property_ptr;
  graph_manager_link_property.getInsertData(0, data_size, labels_link_property_ptr, values_link_property_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);

  // Test the expected tensor collection after insert
  benchmark_1_link.insert1Link(transaction_manager, scale, edge_factor, in_memory, device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (auto& table_map : n_dim_tensor_collection->tables_) {
    table_map.second->syncAxesAndIndicesHData(device);
    table_map.second->syncHData(device);
  }

  // Test the expected tensor axes after insert
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNLabels() == data_size);
  std::shared_ptr<int[]> labels_indices_insert_data;
  n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getLabelsDataPointer(labels_indices_insert_data);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels_indices_insert_values(labels_indices_insert_data.get(), 1, data_size);
  for (int i = 0; i < 1; ++i) {
    for (int j = 0; j < data_size; ++j) {
      assert(labels_indices_insert_values(i, j) == labels_sparse_indices_ptr->getData()(i, j));
    }
  }

  // Test the expected axis 1_links after insert
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndices().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndicesView().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIsModified().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getNotInMemory().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardId().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardIndices().at("1_links")->getTensorSize() == data_size);
  for (int i = 0; i < data_size; ++i) {
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndices().at("1_links")->getData()(i) == i + 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndicesView().at("1_links")->getData()(i) == i + 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIsModified().at("1_links")->getData()(i) == 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getNotInMemory().at("1_links")->getData()(i) == 0);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardId().at("1_links")->getData()(i) == TensorCollectionShardHelper::calc_shard_id(TensorCollectionShardHelper::round_1(data_size, shard_span_perc), i));
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardIndices().at("1_links")->getData()(i) == TensorCollectionShardHelper::calc_shard_index(TensorCollectionShardHelper::round_1(data_size, shard_span_perc), i));
  }

  // Test the expected data after insert
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataTensorSize() == 2 * data_size);
  std::shared_ptr<int[]> data_insert_data_sparse_indices;
  n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataPointer(data_insert_data_sparse_indices);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> data_insert_values_sparse_indices(data_insert_data_sparse_indices.get(), data_size, 2);
  for (int i = 0; i < data_size; ++i) {
    for (int j = 0; j < 2; ++j) {
      assert(data_insert_values_sparse_indices(i, j) == values_sparse_indices_ptr->getData()(i, j));
    }
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getDataTensorSize() == 1 * data_size);
  std::shared_ptr<float[]> data_insert_data_weights;
  n_dim_tensor_collection->tables_.at("Graph_weights")->getDataPointer(data_insert_data_weights);
  Eigen::TensorMap<Eigen::Tensor<float, 2>> data_insert_values_weights(data_insert_data_weights.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_insert_values_weights(i, 0) == values_labels_ptr->getData()(i, 0));
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataTensorSize() <= 1 * data_size);
  std::shared_ptr<TensorArrayGpu8<char>[]> data_insert_data_node_property;
  n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataPointer(data_insert_data_node_property);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 2>> data_insert_values_node_property(data_insert_data_node_property.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_insert_values_node_property(i, 0) == values_node_property_ptr->getData()(i, 0));
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataTensorSize() == 1 * data_size);
  std::shared_ptr<TensorArrayGpu8<char>[]> data_insert_data_link_property;
  n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataPointer(data_insert_data_link_property);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 2>> data_insert_values_link_property(data_insert_data_link_property.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_insert_values_link_property(i, 0) == values_link_property_ptr->getData()(i, 0));
  }

  // Query for the number of black nodes

  // Query for the number of dashed links

  // Query for the adjacency matrix

  // Query for the BFS

  // Query for the SSSP

  // Make the expected tensor axes labels and tensor data after update
  graph_manager_sparse_indices.setUseRandomValues(true);
  graph_manager_weights.setUseRandomValues(true);
  graph_manager_node_property.setUseRandomValues(true);
  graph_manager_link_property.setUseRandomValues(true);
  labels_sparse_indices_ptr.reset();
  values_sparse_indices_ptr.reset();
  graph_manager_sparse_indices.getInsertData(0, data_size, labels_sparse_indices_ptr, values_sparse_indices_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  labels_labels_ptr.reset();
  values_labels_ptr.reset();
  graph_manager_weights.getInsertData(0, data_size, labels_labels_ptr, values_labels_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  labels_node_property_ptr.reset();
  values_node_property_ptr.reset();
  graph_manager_node_property.getInsertData(0, data_size, labels_node_property_ptr, values_node_property_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  labels_link_property_ptr.reset();
  values_link_property_ptr.reset();
  graph_manager_link_property.getInsertData(0, data_size, labels_link_property_ptr, values_link_property_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);

  // Test the expected tensor collection after update
  benchmark_1_link.update1Link(transaction_manager, scale, edge_factor, in_memory, device);
  for (auto& table_map : n_dim_tensor_collection->tables_) {
    table_map.second->syncAxesAndIndicesHData(device);
    table_map.second->syncHData(device);
  }

  // Test the expected tensor axes after update
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNLabels() == data_size);
  std::shared_ptr<int[]> labels_indices_update_data;
  n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getLabelsDataPointer(labels_indices_update_data);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels_indices_update_values(labels_indices_update_data.get(), 1, data_size);
  for (int i = 0; i < 1; ++i) {
    for (int j = 0; j < data_size; ++j) {
      assert(labels_indices_update_values(i, j) == labels_sparse_indices_ptr->getData()(i, j));
    }
  }

  // Test the expected axis 1_links after update
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndices().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndicesView().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIsModified().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getNotInMemory().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardId().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardIndices().at("1_links")->getTensorSize() == data_size);
  for (int i = 0; i < data_size; ++i) {
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndices().at("1_links")->getData()(i) == i + 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndicesView().at("1_links")->getData()(i) == i + 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIsModified().at("1_links")->getData()(i) == 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getNotInMemory().at("1_links")->getData()(i) == 0);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardId().at("1_links")->getData()(i) == TensorCollectionShardHelper::calc_shard_id(TensorCollectionShardHelper::round_1(data_size, shard_span_perc), i));
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardIndices().at("1_links")->getData()(i) == TensorCollectionShardHelper::calc_shard_index(TensorCollectionShardHelper::round_1(data_size, shard_span_perc), i));
  }

  // Test the expected data after update
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataTensorSize() == 2 * data_size);
  std::shared_ptr<int[]> data_update_data_sparse_indices;
  n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataPointer(data_update_data_sparse_indices);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> data_update_values(data_update_data_sparse_indices.get(), data_size, 2);
  for (int i = 0; i < data_size; ++i) {
    for (int j = 0; j < 2; ++j) {
      assert(data_update_values(i, j) == values_sparse_indices_ptr->getData()(i, j));
    }
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getDataTensorSize() == 1 * data_size);
  std::shared_ptr<float[]> data_update_data_weights;
  n_dim_tensor_collection->tables_.at("Graph_weights")->getDataPointer(data_update_data_weights);
  Eigen::TensorMap<Eigen::Tensor<float, 2>> data_update_values_weights(data_update_data_weights.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_update_values_weights(i, 0) == values_labels_ptr->getData()(i, 0));
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataTensorSize() <= 1 * data_size);
  std::shared_ptr<TensorArrayGpu8<char>[]> data_update_data_node_property;
  n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataPointer(data_update_data_node_property);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 2>> data_update_values_node_property(data_update_data_node_property.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_update_values_node_property(i, 0) == values_node_property_ptr->getData()(i, 0));
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataTensorSize() == 1 * data_size);
  std::shared_ptr<TensorArrayGpu8<char>[]> data_update_data_link_property;
  n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataPointer(data_update_data_link_property);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 2>> data_update_values_link_property(data_update_data_link_property.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_update_values_link_property(i, 0) == values_link_property_ptr->getData()(i, 0));
  }

  // Test the expected tensor collection after deletion
  benchmark_1_link.delete1Link(transaction_manager, scale, edge_factor, in_memory, device);
  for (auto& table_map : n_dim_tensor_collection->tables_) {
    table_map.second->syncAxesAndIndicesHData(device);
    table_map.second->syncHData(device);
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("1_nodes")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("1_nodes")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataTensorSize() == 0);

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_InsertUpdateDeleteShardingGpu()
{
  // Parameters for the test
  std::string data_dir = "";
  const int scale = 8; 
  const int edge_factor = 16;
  const bool in_memory = true;
  const int data_size = std::pow(2, scale) * edge_factor;
  const double shard_span_perc = 0.05;
  const int n_engines = 1;
  const int dim_span = std::pow(data_size, 0.25);

  // Setup the Benchmarking suite
  BenchmarkGraph1LinkGpu benchmark_1_link;

  // Setup the GraphTensorCollectionGenerator
  GraphTensorCollectionGeneratorGpu tensor_collection_generator;

  // Setup the device
  cudaStream_t stream; assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess); Eigen::GpuStreamDevice stream_device(&stream, 0); Eigen::GpuDevice device(&stream_device);

  // Make the nD TensorTables
  std::shared_ptr<TensorCollection<Eigen::GpuDevice>> n_dim_tensor_collection = tensor_collection_generator.makeTensorCollection(scale, edge_factor, shard_span_perc, device);

  // Setup the transaction manager
  TransactionManager<Eigen::GpuDevice> transaction_manager;
  transaction_manager.setMaxOperations(data_size + 1);
  transaction_manager.setTensorCollection(n_dim_tensor_collection);

  // Test the initial tensor collection
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("2_columns")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("2_columns")->getNLabels() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardSpans().at("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardSpans().at("1_links") == TensorCollectionShardHelper::round_1(data_size, shard_span_perc));
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getMaxDimSizeFromAxisName("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getMaxDimSizeFromAxisName("1_links") == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("2_columns")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("2_columns")->getNLabels() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getShardSpans().at("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getShardSpans().at("1_links") == TensorCollectionShardHelper::round_1(data_size, shard_span_perc));
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getMaxDimSizeFromAxisName("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getMaxDimSizeFromAxisName("1_links") == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("2_columns")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("2_columns")->getNLabels() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("1_nodes")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("1_nodes")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getShardSpans().at("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getShardSpans().at("1_links") == TensorCollectionShardHelper::round_1(data_size, shard_span_perc));
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getMaxDimSizeFromAxisName("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getMaxDimSizeFromAxisName("1_links") == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("2_columns")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("2_columns")->getNLabels() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getShardSpans().at("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getShardSpans().at("1_links") == TensorCollectionShardHelper::round_1(data_size, shard_span_perc));
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getMaxDimSizeFromAxisName("2_columns") == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getMaxDimSizeFromAxisName("1_links") == data_size);

  // Make the expected tensor axes labels and tensor data after insert
  benchmark_1_link.makeKroneckerGraph(scale, edge_factor, device);
  GraphManagerSparseIndicesGpu<int, float, int, int> graph_manager_sparse_indices(false);
  GraphManagerWeightsGpu<int, float, int, float> graph_manager_weights(false);
  GraphManagerNodePropertyGpu<int, float, int, TensorArrayGpu8<char>> graph_manager_node_property(false);
  GraphManagerLinkPropertyGpu<int, float, int, TensorArrayGpu8<char>> graph_manager_link_property(false);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_sparse_indices_ptr;
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> values_sparse_indices_ptr;
  graph_manager_sparse_indices.getInsertData(0, data_size, labels_sparse_indices_ptr, values_sparse_indices_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_labels_ptr;
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 2>> values_labels_ptr;
  graph_manager_weights.getInsertData(0, data_size, labels_labels_ptr, values_labels_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_node_property_ptr;
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 2>> values_node_property_ptr;
  graph_manager_node_property.getInsertData(0, data_size, labels_node_property_ptr, values_node_property_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_link_property_ptr;
  std::shared_ptr<TensorData<TensorArrayGpu8<char>, Eigen::GpuDevice, 2>> values_link_property_ptr;
  graph_manager_link_property.getInsertData(0, data_size, labels_link_property_ptr, values_link_property_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);

  // Test the expected tensor collection after insert
  benchmark_1_link.insert1Link(transaction_manager, scale, edge_factor, in_memory, device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (auto& table_map : n_dim_tensor_collection->tables_) {
    table_map.second->syncAxesAndIndicesHData(device);
    table_map.second->syncHData(device);
  }

  // Test the expected tensor axes after insert
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNLabels() == data_size);
  std::shared_ptr<int[]> labels_indices_insert_data;
  n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getLabelsDataPointer(labels_indices_insert_data);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels_indices_insert_values(labels_indices_insert_data.get(), 1, data_size);
  for (int i = 0; i < 1; ++i) {
    for (int j = 0; j < data_size; ++j) {
      assert(labels_indices_insert_values(i, j) == labels_sparse_indices_ptr->getData()(i, j));
    }
  }

  // Test the expected axis 1_links after insert
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndices().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndicesView().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIsModified().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getNotInMemory().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardId().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardIndices().at("1_links")->getTensorSize() == data_size);
  for (int i = 0; i < data_size; ++i) {
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndices().at("1_links")->getData()(i) == i + 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndicesView().at("1_links")->getData()(i) == i + 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIsModified().at("1_links")->getData()(i) == 0);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getNotInMemory().at("1_links")->getData()(i) == 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardId().at("1_links")->getData()(i) == TensorCollectionShardHelper::calc_shard_id(TensorCollectionShardHelper::round_1(data_size, shard_span_perc), i));
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardIndices().at("1_links")->getData()(i) == TensorCollectionShardHelper::calc_shard_index(TensorCollectionShardHelper::round_1(data_size, shard_span_perc), i));
  }

  // Test the expected data after insert
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataTensorSize() == 2 * data_size);
  std::shared_ptr<int[]> data_insert_data_sparse_indices;
  n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataPointer(data_insert_data_sparse_indices);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> data_insert_values_sparse_indices(data_insert_data_sparse_indices.get(), data_size, 2);
  for (int i = 0; i < data_size; ++i) {
    for (int j = 0; j < 2; ++j) {
      assert(data_insert_values_sparse_indices(i, j) == values_sparse_indices_ptr->getData()(i, j));
    }
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getDataTensorSize() == 1 * data_size);
  std::shared_ptr<float[]> data_insert_data_weights;
  n_dim_tensor_collection->tables_.at("Graph_weights")->getDataPointer(data_insert_data_weights);
  Eigen::TensorMap<Eigen::Tensor<float, 2>> data_insert_values_weights(data_insert_data_weights.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_insert_values_weights(i, 0) == values_labels_ptr->getData()(i, 0));
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataTensorSize() <= 1 * data_size);
  std::shared_ptr<TensorArrayGpu8<char>[]> data_insert_data_node_property;
  n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataPointer(data_insert_data_node_property);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 2>> data_insert_values_node_property(data_insert_data_node_property.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_insert_values_node_property(i, 0) == values_node_property_ptr->getData()(i, 0));
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataTensorSize() == 1 * data_size);
  std::shared_ptr<TensorArrayGpu8<char>[]> data_insert_data_link_property;
  n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataPointer(data_insert_data_link_property);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 2>> data_insert_values_link_property(data_insert_data_link_property.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_insert_values_link_property(i, 0) == values_link_property_ptr->getData()(i, 0));
  }

  // Query for the number of black nodes

  // Query for the number of dashed links

  // Query for the adjacency matrix

  // Query for the BFS

  // Query for the SSSP

  // Make the expected tensor axes labels and tensor data after update
  graph_manager_sparse_indices.setUseRandomValues(true);
  graph_manager_weights.setUseRandomValues(true);
  graph_manager_node_property.setUseRandomValues(true);
  graph_manager_link_property.setUseRandomValues(true);
  labels_sparse_indices_ptr.reset();
  values_sparse_indices_ptr.reset();
  graph_manager_sparse_indices.getInsertData(0, data_size, labels_sparse_indices_ptr, values_sparse_indices_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  labels_labels_ptr.reset();
  values_labels_ptr.reset();
  graph_manager_weights.getInsertData(0, data_size, labels_labels_ptr, values_labels_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  labels_node_property_ptr.reset();
  values_node_property_ptr.reset();
  graph_manager_node_property.getInsertData(0, data_size, labels_node_property_ptr, values_node_property_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);
  labels_link_property_ptr.reset();
  values_link_property_ptr.reset();
  graph_manager_link_property.getInsertData(0, data_size, labels_link_property_ptr, values_link_property_ptr, benchmark_1_link.graph_manager_helper_.kronecker_graph_indices_, benchmark_1_link.graph_manager_helper_.kronecker_graph_weights_, benchmark_1_link.graph_manager_helper_.kronecker_graph_node_ids_, benchmark_1_link.graph_manager_helper_.kronecker_graph_link_ids_, device);

  // Test the expected tensor collection after update
  benchmark_1_link.update1Link(transaction_manager, scale, edge_factor, in_memory, device);
  for (auto& table_map : n_dim_tensor_collection->tables_) {
    table_map.second->syncAxesAndIndicesHData(device);
    table_map.second->syncHData(device);
  }

  // Test the expected tensor axes after update
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNLabels() == data_size);
  std::shared_ptr<int[]> labels_indices_update_data;
  n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getLabelsDataPointer(labels_indices_update_data);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> labels_indices_update_values(labels_indices_update_data.get(), 1, data_size);
  for (int i = 0; i < 1; ++i) {
    for (int j = 0; j < data_size; ++j) {
      assert(labels_indices_update_values(i, j) == labels_sparse_indices_ptr->getData()(i, j));
    }
  }

  // Test the expected axis 1_links after update
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndices().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndicesView().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIsModified().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getNotInMemory().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardId().at("1_links")->getTensorSize() == data_size);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardIndices().at("1_links")->getTensorSize() == data_size);
  for (int i = 0; i < data_size; ++i) {
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndices().at("1_links")->getData()(i) == i + 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIndicesView().at("1_links")->getData()(i) == i + 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getIsModified().at("1_links")->getData()(i) == 0);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getNotInMemory().at("1_links")->getData()(i) == 1);
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardId().at("1_links")->getData()(i) == TensorCollectionShardHelper::calc_shard_id(TensorCollectionShardHelper::round_1(data_size, shard_span_perc), i));
    assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getShardIndices().at("1_links")->getData()(i) == TensorCollectionShardHelper::calc_shard_index(TensorCollectionShardHelper::round_1(data_size, shard_span_perc), i));
  }

  // Test the expected data after update
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataTensorSize() == 2 * data_size);
  std::shared_ptr<int[]> data_update_data_sparse_indices;
  n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataPointer(data_update_data_sparse_indices);
  Eigen::TensorMap<Eigen::Tensor<int, 2>> data_update_values(data_update_data_sparse_indices.get(), data_size, 2);
  for (int i = 0; i < data_size; ++i) {
    for (int j = 0; j < 2; ++j) {
      assert(data_update_values(i, j) == values_sparse_indices_ptr->getData()(i, j));
    }
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getDataTensorSize() == 1 * data_size);
  std::shared_ptr<float[]> data_update_data_weights;
  n_dim_tensor_collection->tables_.at("Graph_weights")->getDataPointer(data_update_data_weights);
  Eigen::TensorMap<Eigen::Tensor<float, 2>> data_update_values_weights(data_update_data_weights.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_update_values_weights(i, 0) == values_labels_ptr->getData()(i, 0));
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataTensorSize() <= 1 * data_size);
  std::shared_ptr<TensorArrayGpu8<char>[]> data_update_data_node_property;
  n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataPointer(data_update_data_node_property);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 2>> data_update_values_node_property(data_update_data_node_property.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_update_values_node_property(i, 0) == values_node_property_ptr->getData()(i, 0));
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataTensorSize() == 1 * data_size);
  std::shared_ptr<TensorArrayGpu8<char>[]> data_update_data_link_property;
  n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataPointer(data_update_data_link_property);
  Eigen::TensorMap<Eigen::Tensor<TensorArrayGpu8<char>, 2>> data_update_values_link_property(data_update_data_link_property.get(), data_size, 1);
  for (int i = 0; i < data_size; ++i) {
    assert(data_update_values_link_property(i, 0) == values_link_property_ptr->getData()(i, 0));
  }

  // Test the expected tensor collection after deletion
  benchmark_1_link.delete1Link(transaction_manager, scale, edge_factor, in_memory, device);
  for (auto& table_map : n_dim_tensor_collection->tables_) {
    table_map.second->syncAxesAndIndicesHData(device);
    table_map.second->syncHData(device);
  }
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_sparse_indices")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_weights")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("1_nodes")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getAxes().at("1_nodes")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_node_property")->getDataTensorSize() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("1_links")->getNDimensions() == 1);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getAxes().at("1_links")->getNLabels() == 0);
  assert(n_dim_tensor_collection->tables_.at("Graph_link_property")->getDataTensorSize() == 0);

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

int main(int argc, char** argv)
{
  test_InsertUpdateDeleteGpu();
  test_InsertUpdateDeleteShardingGpu();
  return 0;
}
#endif