
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA
#include <TensorBase/ml/TensorTableGpu.h>
#include <string>

using namespace TensorBase;
using namespace std;

void test_constructorGpu()
{
  TensorTableGpu<float, 3>* ptr = nullptr;
  TensorTableGpu<float, 3>* nullPointer = nullptr;
  ptr = new TensorTableGpu<float, 3>();
  assert(ptr != nullPointer);
  delete ptr;
}

void test_destructorGpu()
{
  TensorTableGpu<float, 3>* ptr = nullptr;
  ptr = new TensorTableGpu<float, 3>();
  delete ptr;
}

void test_constructorNameAndAxesGpu()
{
  TensorTableGpu<float, 3> tensorTable("1");

  assert(tensorTable.getId() == -1);
  assert(tensorTable.getName() == "1");
}

void test_gettersAndSettersGpu()
{
  TensorTableGpu<float, 3> tensorTable;
  // Check defaults
  assert(tensorTable.getId() == -1);
  assert(tensorTable.getName() == "");
  assert(tensorTable.getAxes().size() == 0);

  // Check getters/setters
  tensorTable.setId(1);
  tensorTable.setName("1");
  std::map<std::string, int> shard_span = {
    {"1", 2}, {"2", 2}, {"3", 3} };
  tensorTable.setShardSpans(shard_span);

  assert(tensorTable.getId() == 1);
  assert(tensorTable.getName() == "1");
  assert(tensorTable.getShardSpans() == shard_span);

  // SetAxes associated getters/setters
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels1 = 2, nlabels2 = 3, nlabels3 = 5;
  Eigen::Tensor<int, 2> labels1(1, nlabels1), labels2(1, nlabels2), labels3(1, nlabels3);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  //Eigen::Tensor<std::string, 2> labels1(1, nlabels1), labels2(1, nlabels2), labels3(1, nlabels3);
  //labels1.setConstant("x-axis");
  //labels2.setConstant("y-axis");
  //labels3.setConstant("z-axis");
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // Test expected axes values
  assert(tensorTable.getAxes().at("1")->getName() == "1");
  //assert(tensorTable.getAxes().at("1")->getLabels()(0, 0) == 1);
  ////assert(tensorTable.getAxes().at("1")->getLabels()(0,0) == "x-axis");
  assert(tensorTable.getAxes().at("1")->getNLabels() == nlabels1);
  assert(tensorTable.getAxes().at("1")->getNDimensions() == 1);
  assert(tensorTable.getAxes().at("1")->getDimensions()(0) == "x");
  assert(tensorTable.getIndices().at("1")->getData()(0) == 1);
  assert(tensorTable.getIndices().at("1")->getData()(nlabels1 - 1) == nlabels1);
  assert(tensorTable.getIndicesView().at("1")->getData()(0) == 1);
  assert(tensorTable.getIndicesView().at("1")->getData()(nlabels1 - 1) == nlabels1);
  assert(tensorTable.getIsModified().at("1")->getData()(0) == 0);
  assert(tensorTable.getInMemory().at("1")->getData()(0) == 0);
  assert(tensorTable.getShardId().at("1")->getData()(0) == 1);
  assert(tensorTable.getShardIndices().at("1")->getData()(0) == 1);
  assert(tensorTable.getShardIndices().at("1")->getData()(nlabels1 - 1) == nlabels1);

  assert(tensorTable.getAxes().at("2")->getName() == "2");
  //assert(tensorTable.getAxes().at("2")->getLabels()(0, 0) == 2);
  ////assert(tensorTable.getAxes().at("2")->getLabels()(0, 0) == "y-axis");
  assert(tensorTable.getAxes().at("2")->getNLabels() == nlabels2);
  assert(tensorTable.getAxes().at("2")->getNDimensions() == 1);
  assert(tensorTable.getAxes().at("2")->getDimensions()(0) == "y");
  assert(tensorTable.getIndices().at("2")->getData()(0) == 1);
  assert(tensorTable.getIndices().at("2")->getData()(nlabels2 - 1) == nlabels2);
  assert(tensorTable.getIndicesView().at("2")->getData()(0) == 1);
  assert(tensorTable.getIndicesView().at("2")->getData()(nlabels2 - 1) == nlabels2);
  assert(tensorTable.getIsModified().at("2")->getData()(0) == 0);
  assert(tensorTable.getInMemory().at("2")->getData()(0) == 0);
  assert(tensorTable.getShardId().at("2")->getData()(0) == 1);
  assert(tensorTable.getShardIndices().at("2")->getData()(0) == 1);
  assert(tensorTable.getShardIndices().at("2")->getData()(nlabels2 - 1) == nlabels);

  assert(tensorTable.getAxes().at("3")->getName() == "3");
  //assert(tensorTable.getAxes().at("3")->getLabels()(0, 0) == 3);
  ////assert(tensorTable.getAxes().at("3")->getLabels()(0, 0) == "z-axis");
  assert(tensorTable.getAxes().at("3")->getNLabels() == nlabels3);
  assert(tensorTable.getAxes().at("3")->getNDimensions() == 1);
  assert(tensorTable.getAxes().at("3")->getDimensions()(0) == "z");
  assert(tensorTable.getIndices().at("3")->getData()(0) == 1);
  assert(tensorTable.getIndices().at("3")->getData()(nlabels3 - 1) == nlabels3);
  assert(tensorTable.getIndicesView().at("3")->getData()(0) == 1);
  assert(tensorTable.getIndicesView().at("3")->getData()(nlabels3 - 1) == nlabels3);
  assert(tensorTable.getIsModified().at("3")->getData()(0) == 0);
  assert(tensorTable.getInMemory().at("3")->getData()(0) == 0);
  assert(tensorTable.getShardId().at("3")->getData()(0) == 1);
  assert(tensorTable.getShardIndices().at("3")->getData()(0) == 1);
  assert(tensorTable.getShardIndices().at("3")->getData()(nlabels3 - 1) == nlabels3);

  // Test expected axis to dims mapping
  assert(tensorTable.getDimFromAxisName("1") == 0);
  assert(tensorTable.getDimFromAxisName("2") == 1);
  assert(tensorTable.getDimFromAxisName("3") == 2);

  // Test expected tensor shard spans
  assert(tensorTable.getShardSpans().at("1") == 2);
  assert(tensorTable.getShardSpans().at("2") == 3);
  assert(tensorTable.getShardSpans().at("3") == 5);

  // Test expected tensor dimensions
  assert(tensorTable.getDimensions().at(0) == 2);
  assert(tensorTable.getDimensions().at(1) == 3);
  assert(tensorTable.getDimensions().at(2) == 5);

  // Test expected tensor data values
  assert(tensorTable.getDataDimensions().at(0) == 2);
  assert(tensorTable.getDataDimensions().at(1) == 3);
  assert(tensorTable.getDataDimensions().at(2) == 5);
  size_t test = 2 * 3 * 5 * sizeof(float);
  assert(tensorTable.getDataTensorBytes() == test);

  // Test clear
  tensorTable.clear();
  assert(tensorTable.getAxes().size() == 0);
  assert(tensorTable.getIndices().size() == 0);
  assert(tensorTable.getIndicesView().size() == 0);
  assert(tensorTable.getIsModified().size() == 0);
  assert(tensorTable.getInMemory().size() == 0);
  assert(tensorTable.getShardId().size() == 0);
  assert(tensorTable.getShardIndices().size() == 0);
  assert(tensorTable.getDimensions().at(0) == 0);
  assert(tensorTable.getDimensions().at(1) == 0);
  assert(tensorTable.getDimensions().at(2) == 0);
  assert(tensorTable.getShardSpans().size() == 0);
}

void test_zeroIndicesViewAndResetIndicesViewGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // sync the tensorTable indices
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);

  // test null
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);
  }

  // test zero
  tensorTable.zeroIndicesView("1", device);
  tensorTable.getIndicesView().at("1")->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getIndicesView().at("1")->getData()(i) == 0);
  }

  // test reset
  tensorTable.getIndicesView().at("1")->setDataStatus(false, true);
  tensorTable.resetIndicesView("1", device);
  tensorTable.getIndicesView().at("1")->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);
  }

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_selectIndicesViewGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2, 3} });
  labels2.setValues({ {0, 1, 2, 3} });
  labels3.setValues({ {0, 1, 2, 3} });
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // set up the selection labels
  Eigen::Tensor<int, 1> select_labels_values(nlabels / 2);
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    if (i % 2 == 0) {
      select_labels_values(iter) = i;
      ++iter;
    }
  }
  TensorDataGpu<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ nlabels / 2 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpu<int, 1>>(select_labels);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);

  // test the updated view
  select_labels_ptr->syncHAndDData(device);
  tensorTable.selectIndicesView("1", 0, select_labels_ptr, device);
  tensorTable.syncIndicesViewHAndDData(device);
  select_labels_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    if (i % 2 == 0)
      assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);
    else
      assert(tensorTable.getIndicesView().at("1")->getData()(i) == 0);
  }

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_broadcastSelectIndicesViewGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // sync the tensorTable indices
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);

  // setup the indices test
  Eigen::Tensor<int, 3> indices_test(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        indices_test(i, j, k) = i + 1;
      }
    }
  }

  // test the broadcast indices values
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_view_bcast;
  tensorTable.broadcastSelectIndicesView(indices_view_bcast, "1", device);
  indices_view_bcast->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        //std::cout << "Test broadcastSelectIndicesView i,j,k :" << i << "," << j << "," << k << "; Labels: " << indices_view_bcast->getData()(i, j, k) << "; Expected: " << indices_test(i, j, k) << std::endl;
        assert(indices_view_bcast->getData()(i, j, k) == indices_test(i, j, k));
      }
    }
  }

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_extractTensorDataGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // setup the tensor data, selection indices, and test selection data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<int, 3> indices_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<float, 3> tensor_test(Eigen::array<Eigen::Index, 3>({ nlabels / 2, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        float value = i * nlabels + j * nlabels + k;
        tensor_values(i, j, k) = value;
        if (i % 2 == 0) {
          indices_values(i, j, k) = 1;
          tensor_test(i / 2, j, k) = value;
        }
        else {
          indices_values(i, j, k) = 0;
        }
      }
    }
  }
  tensorTable.setData(tensor_values);
  TensorDataGpu<int, 3> indices_select(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  indices_select.setData(indices_values);
  auto indices_select_ptr = std::make_shared<TensorDataGpu<int, 3>>(indices_select);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);

  // test
  indices_select_ptr->syncHAndDData(device);
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> tensor_select;
  tensorTable.reduceTensorDataToSelectIndices(indices_select_ptr,
    tensor_select, "1", nlabels / 2, device);
  tensor_select->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels / 2; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        assert(tensor_select->getData()(i, j, k) == tensor_test(i, j, k));
      }
    }
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_selectTensorIndicesGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 2;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // setup the tensor select and values select data
  Eigen::Tensor<float, 3> tensor_select_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<float, 1> values_select_values(Eigen::array<Eigen::Index, 1>({ nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    values_select_values(i) = 2.0;
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_select_values(i, j, k) = float(iter);
        ++iter;
      }
    }
  }
  TensorDataGpu<float, 3> tensor_select(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  tensor_select.setData(tensor_select_values);
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> tensor_select_ptr = std::make_shared<TensorDataGpu<float, 3>>(tensor_select);
  TensorDataGpu<float, 1> values_select(Eigen::array<Eigen::Index, 1>({ nlabels }));
  values_select.setData(values_select_values);
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 1>> values_select_ptr = std::make_shared<TensorDataGpu<float, 1>>(values_select);

  // Sync the data
  tensor_select_ptr->syncHAndDData(device);
  values_select_ptr->syncHAndDData(device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);

  // test inequality
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_select;
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::NOT_EQUAL_TO, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) == 2.0)
          assert(indices_select->getData()(i, j, k) == 0);
        else
          assert(indices_select->getData()(i, j, k) == 1);
      }
    }
  }

  // test equality
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::EQUAL_TO, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) == 2.0)
          assert(indices_select->getData()(i, j, k) == 1);
        else
          assert(indices_select->getData()(i, j, k) == 0);
      }
    }
  }

  // test less than
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::LESS_THAN, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) < 2.0)
          assert(indices_select->getData()(i, j, k) == 1);
        else
          assert(indices_select->getData()(i, j, k) == 0);
      }
    }
  }

  // test less than or equal to
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::LESS_THAN_OR_EQUAL_TO, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) <= 2.0)
          assert(indices_select->getData()(i, j, k) == 1);
        else
          assert(indices_select->getData()(i, j, k) == 0);
      }
    }
  }

  // test greater than
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::GREATER_THAN, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) > 2.0)
          assert(indices_select->getData()(i, j, k) == 1);
        else
          assert(indices_select->getData()(i, j, k) == 0);
      }
    }
  }

  // test greater than or equal to
  indices_select.reset();
  tensorTable.selectTensorIndicesOnReducedTensorData(indices_select, values_select_ptr, tensor_select_ptr,
    "1", nlabels, logicalComparitors::logicalComparitor::GREATER_THAN_OR_EQUAL_TO, logicalModifiers::logicalModifier::NONE, device);
  indices_select->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (tensor_select_values(i, j, k) >= 2.0)
          assert(indices_select->getData()(i, j, k) == 1);
        else
          assert(indices_select->getData()(i, j, k) == 0);
      }
    }
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_applyIndicesSelectToIndicesViewGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setConstant(1);
  labels2.setConstant(2);
  labels3.setConstant(3);
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);

  // setup the indices select
  Eigen::Tensor<int, 3> indices_select_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i == j && j == k && k == i
          && i < nlabels - 1 && j < nlabels - 1 && k < nlabels - 1) // the first 2 diagonal elements
          indices_select_values(i, j, k) = 1;
        else
          indices_select_values(i, j, k) = 0;
      }
    }
  }
  TensorDataGpu<int, 3> indices_select(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  indices_select.setData(indices_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_select_ptr = std::make_shared<TensorDataGpu<int, 3>>(indices_select);
  indices_select_ptr->syncHAndDData(device);

  // test using the second indices view  
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.getIndicesView().at("2")->getData()(nlabels - 1) = 0;
  tensorTable.syncIndicesViewHAndDData(device);

  // test for OR within continuator and OR prepend continuator
  tensorTable.applyIndicesSelectToIndicesView(indices_select_ptr, "1", "2", logicalContinuators::logicalContinuator::OR, logicalContinuators::logicalContinuator::OR, device);
  tensorTable.syncIndicesViewHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    if (i == nlabels - 1)
      assert(tensorTable.getIndicesView().at("2")->getData()(i) == 0);
    else
      assert(tensorTable.getIndicesView().at("2")->getData()(i) == i + 1);
  }

  // reset and modify the indices view
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.resetIndicesView("2", device);
  tensorTable.syncIndicesViewHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  tensorTable.getIndicesView().at("2")->getData()(0) = 0;
  tensorTable.syncIndicesViewHAndDData(device);

  // test for AND within continuator and OR prepend continuator
  tensorTable.applyIndicesSelectToIndicesView(indices_select_ptr, "1", "2", logicalContinuators::logicalContinuator::AND, logicalContinuators::logicalContinuator::OR, device);  
  tensorTable.syncIndicesViewHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    //std::cout << "Test applyIndicesSelectToIndicesView i " << i << "; Indices View: " << tensorTable.getIndicesView().at("2")->getData()(i) << std::endl;
    if (i == 0)
      assert(tensorTable.getIndicesView().at("2")->getData()(i) == 0);
    else
      assert(tensorTable.getIndicesView().at("2")->getData()(i) == i + 1);
  }

  // Reset and modify the indices view
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.resetIndicesView("2", device);  
  tensorTable.syncIndicesViewHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  tensorTable.getIndicesView().at("2")->getData()(0) = 0;
  tensorTable.syncIndicesViewHAndDData(device);

  // test for OR within continuator and AND prepend continuator
  tensorTable.applyIndicesSelectToIndicesView(indices_select_ptr, "1", "2", logicalContinuators::logicalContinuator::OR, logicalContinuators::logicalContinuator::AND, device);
  tensorTable.syncIndicesViewHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    if (i != 0 && i < nlabels - 1)
      assert(tensorTable.getIndicesView().at("2")->getData()(i) == i + 1);
    else
      assert(tensorTable.getIndicesView().at("2")->getData()(i) == 0);
  }

  // Reset the indices view
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.resetIndicesView("2", device);

  // and update the indices_select_ptr
  indices_select_ptr->setDataStatus(true, false);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i == j && j == k && k == i
          && i < nlabels - 1 && j < nlabels - 1 && k < nlabels - 1) // the first 2 diagonal elements
          indices_select_ptr->getData()(i, j, k) = 1;
        else if (j == 0)
          indices_select_ptr->getData()(i, j, k) = 1; // all elements along the first index of the selection dim
        else
          indices_select_ptr->getData()(i, j, k) = 0;
      }
    }
  }
  indices_select_ptr->syncHAndDData(device);

  // test for AND within continuator and AND prepend continuator
  tensorTable.applyIndicesSelectToIndicesView(indices_select_ptr, "1", "2", logicalContinuators::logicalContinuator::AND, logicalContinuators::logicalContinuator::AND, device);
  tensorTable.syncIndicesViewHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    if (i == 0)
      assert(tensorTable.getIndicesView().at("2")->getData()(i) == i + 1);
    else
      assert(tensorTable.getIndicesView().at("2")->getData()(i) == 0);
  }

  // TODO: lacking code coverage for the case of TDim = 2
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_whereIndicesViewDataGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 4;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2, 3} });
  labels2.setValues({ {0, 1, 2, 3} });
  labels3.setValues({ {0, 1, 2, 3} });
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k) = float(iter);
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncHAndDData(device);
  tensorTable.syncAxesHAndDData(device);

  // set up the selection labels
  Eigen::Tensor<int, 1> select_labels_values(2);
  select_labels_values(0) = 0; select_labels_values(1) = 2;
  TensorDataGpu<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 2 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpu<int, 1>>(select_labels);
  select_labels_ptr->syncHAndDData(device);

  // set up the selection values
  Eigen::Tensor<float, 1> select_values_values(2);
  select_values_values(0) = 9; select_values_values(1) = 9;
  TensorDataGpu<float, 1> select_values(Eigen::array<Eigen::Index, 1>({ 2 }));
  select_values.setData(select_values_values);
  std::shared_ptr<TensorDataGpu<float, 1>> select_values_ptr = std::make_shared<TensorDataGpu<float, 1>>(select_values);
  select_values_ptr->syncHAndDData(device);

  // test
  tensorTable.whereIndicesView("1", 0, select_labels_ptr, select_values_ptr,
    logicalComparitors::logicalComparitor::EQUAL_TO, logicalModifiers::logicalModifier::NONE,
    logicalContinuators::logicalContinuator::OR, logicalContinuators::logicalContinuator::AND, device);
  tensorTable.syncIndicesViewHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    // indices view 1
    assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1); // Unchanged

    //// indices view 2
    //if (i == 2) // FIXME: i==0?
    //  assert(tensorTable.getIndicesView().at("2")->getData()(i) == i + 1);
    //else
    //  assert(tensorTable.getIndicesView().at("2")->getData()(i) == 0);

    //// indices view 3
    //if (i == 1) // FIXME: i==3?
    //  assert(tensorTable.getIndicesView().at("3")->getData()(i) == i + 1);
    //else
    //  assert(tensorTable.getIndicesView().at("3")->getData()(i) == 0);
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_sliceTensorForSortGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k) = float(iter);
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);

  // test sliceTensorForSort for axis 2
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 1>> tensor_sort;
  tensorTable.sliceTensorDataForSort(tensor_sort, "1", 1, "2", device); 
  tensor_sort->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  std::vector<float> tensor_slice_2_test = { 9, 12, 15 };
  for (int i = 0; i < nlabels; ++i) {
    assert(tensor_sort->getData()(i) == tensor_slice_2_test.at(i), 1e-3);
  }

  // test sliceTensorForSort for axis 2
  tensor_sort.reset();
  tensorTable.sliceTensorDataForSort(tensor_sort, "1", 1, "3", device);
  tensor_sort->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  std::vector<float> tensor_slice_3_test = { 9, 10, 11 };
  for (int i = 0; i < nlabels; ++i) {
    assert(tensor_sort->getData()(i) == tensor_slice_3_test.at(i), 1e-3);
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_sortIndicesViewDataGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k) = float(iter);
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);

  // set up the selection labels
  Eigen::Tensor<int, 1> select_labels_values(1);
  select_labels_values(0) = 1;
  TensorDataGpu<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpu<int, 1>>(select_labels);
  select_labels_ptr->syncHAndDData(device);

  // test sort ASC
  tensorTable.sortIndicesView("1", 0, select_labels_ptr, sortOrder::ASC, device);
  tensorTable.syncIndicesViewHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);
    assert(tensorTable.getIndicesView().at("2")->getData()(i) == i + 1);
    assert(tensorTable.getIndicesView().at("3")->getData()(i) == i + 1);
  }

  // test sort DESC
  tensorTable.setIndicesViewDataStatus(false, true);
  tensorTable.sortIndicesView("1", 0, select_labels_ptr, sortOrder::DESC, device);
  tensorTable.syncIndicesViewHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);
    assert(tensorTable.getIndicesView().at("2")->getData()(i) == nlabels - i);
    assert(tensorTable.getIndicesView().at("3")->getData()(i) == nlabels - i);
  }

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_makeSelectIndicesFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);

  // Test null
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_select;
  tensorTable.makeSelectIndicesFromIndicesView(indices_select, device);
  indices_select->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        assert(indices_select->getData()(i, j, k) == 1);
      }
    }
  }

  // make the expected indices tensor
  Eigen::Tensor<int, 3> indices_select_test(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i == 1)
          indices_select_test(i, j, k) = 1;
        else
          indices_select_test(i, j, k) = 0;
      }
    }
  }

  // select
  TensorDataGpu<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  Eigen::Tensor<int, 1> select_labels_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels_values.setValues({ 1 });
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpu<int, 1>>(select_labels);
  select_labels_ptr->syncHAndDData(device);
  tensorTable.selectIndicesView("1", 0, select_labels_ptr, device);

  // Test selected
  indices_select.reset();
  tensorTable.makeSelectIndicesFromIndicesView(indices_select, device);
  indices_select->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        assert(indices_select->getData()(i, j, k) == indices_select_test(i, j, k));
      }
    }
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_getSelectTensorDataFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k) = float(iter);
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);

  // select label 1 from axis 1
  TensorDataGpu<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  Eigen::Tensor<int, 1> select_labels_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels_values.setValues({ 1 });
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpu<int, 1>>(select_labels);
  select_labels_ptr->syncHAndDData(device);
  tensorTable.selectIndicesView("1", 0, select_labels_ptr, device);

  // make the expected dimensions
  Eigen::array<Eigen::Index, 3> select_dimensions = { 1, 3, 3 };

  // make the indices_select
  Eigen::Tensor<float, 3> tensor_select_test(select_dimensions);
  Eigen::Tensor<int, 3> indices_select_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i == 1) {
          indices_select_values(i, j, k) = 1;
          tensor_select_test(0, j, k) = float(iter);
        }
        else {
          indices_select_values(i, j, k) = 0;
        }
        ++iter;
      }
    }
  }
  TensorDataGpu<int, 3> indices_select(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  indices_select.setData(indices_select_values);
  std::shared_ptr<TensorDataGpu<int, 3>> indices_select_ptr = std::make_shared<TensorDataGpu<int, 3>>(indices_select);
  indices_select_ptr->syncHAndDData(device);

  // test for the selected data
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> tensor_select_ptr;
  tensorTable.getSelectTensorDataFromIndicesView(tensor_select_ptr, indices_select_ptr, device);
  tensor_select_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(tensor_select_ptr->getDimensions() == select_dimensions);
  for (int j = 0; j < nlabels; ++j) {
    for (int k = 0; k < nlabels; ++k) {
      assert(tensor_select_ptr->getData()(0, j, k) == tensor_select_test(0, j, k), 1e-3);
    }
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_selectTensorDataGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k) = float(iter);
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);

  // select label 1 from axis 1
  TensorDataGpu<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  Eigen::Tensor<int, 1> select_labels_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels_values.setValues({ 1 });
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpu<int, 1>>(select_labels);
  select_labels_ptr->syncHAndDData(device);
  tensorTable.selectIndicesView("1", 0, select_labels_ptr, device);

  // Test `selectTensorData`
  tensorTable.selectTensorData(device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);

  // Test expected axes values
  assert(tensorTable.getAxes().at("1")->getName() == "1");
  assert(tensorTable.getAxes().at("1")->getNLabels() == 1);
  assert(tensorTable.getAxes().at("1")->getDimensions()(0) == "x");
  assert(tensorTable.getIndices().at("1")->getData()(0) == 1);
  assert(tensorTable.getIndicesView().at("1")->getData()(0) == 1);
  assert(tensorTable.getIsModified().at("1")->getData()(0) == 0);
  assert(tensorTable.getInMemory().at("1")->getData()(0) == 0);
  assert(tensorTable.getShardId().at("1")->getData()(0) == 1);
  assert(tensorTable.getShardIndices().at("1")->getData()(0) == 1);

  assert(tensorTable.getAxes().at("2")->getName() == "2");
  assert(tensorTable.getAxes().at("2")->getNLabels() == nlabels);
  assert(tensorTable.getAxes().at("2")->getNDimensions() == 1);
  assert(tensorTable.getAxes().at("2")->getDimensions()(0) == "y");
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getIndices().at("2")->getData()(i) == i + 1);
    assert(tensorTable.getIndicesView().at("2")->getData()(i) == i + 1);
    assert(tensorTable.getIsModified().at("2")->getData()(i) == 0);
    assert(tensorTable.getInMemory().at("2")->getData()(i) == 0);
    assert(tensorTable.getShardId().at("2")->getData()(i) == 1);
    assert(tensorTable.getShardIndices().at("2")->getData()(i) == i + 1);
  }

  assert(tensorTable.getAxes().at("3")->getName() == "3");
  assert(tensorTable.getAxes().at("3")->getNLabels() == nlabels);
  assert(tensorTable.getAxes().at("3")->getNDimensions() == 1);
  assert(tensorTable.getAxes().at("3")->getDimensions()(0) == "z");
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getIndices().at("3")->getData()(i) == i + 1);
    assert(tensorTable.getIndicesView().at("3")->getData()(i) == i + 1);
    assert(tensorTable.getIsModified().at("3")->getData()(i) == 0);
    assert(tensorTable.getInMemory().at("3")->getData()(i) == 0);
    assert(tensorTable.getShardId().at("3")->getData()(i) == 1);
    assert(tensorTable.getShardIndices().at("3")->getData()(i) == i + 1);
  }

  // Test expected axis to dims mapping
  assert(tensorTable.getDimFromAxisName("1") == 0);
  assert(tensorTable.getDimFromAxisName("2") == 1);
  assert(tensorTable.getDimFromAxisName("3") == 2);

  // Test expected tensor dimensions
  assert(tensorTable.getDimensions().at(0) == 1);
  assert(tensorTable.getDimensions().at(1) == 3);
  assert(tensorTable.getDimensions().at(2) == 3);

  // Test expected tensor data values
  assert(tensorTable.getDataDimensions().at(0) == 1);
  assert(tensorTable.getDataDimensions().at(1) == 3);
  assert(tensorTable.getDataDimensions().at(2) == 3);
  size_t test = 1 * 3 * 3 * sizeof(float);
  assert(tensorTable.getDataTensorBytes() == test);
}

void test_makeSortIndicesViewFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2)));
  tensorTable.addTensorAxis(std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3)));
  tensorTable.setAxes();

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);

  // make the expected tensor indices
  Eigen::Tensor<int, 3> indices_test(nlabels, nlabels, nlabels);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        indices_test(i, j, k) = i + j * nlabels + k * nlabels*nlabels + 1;
      }
    }
  }

  // Test for the sort indices
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_sort_ptr;
  tensorTable.makeSortIndicesFromIndicesView(indices_sort_ptr, device);
  indices_sort_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        assert(indices_sort_ptr->getData()(i, j, k) == indices_test(i, j, k));
      }
    }
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_sortTensorDataGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = float(iter);
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);

  // set up the selection labels
  Eigen::Tensor<int, 1> select_labels_values(1);
  select_labels_values(0) = 0;
  TensorDataGpu<int, 1> select_labels(Eigen::array<Eigen::Index, 1>({ 1 }));
  select_labels.setData(select_labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> select_labels_ptr = std::make_shared<TensorDataGpu<int, 1>>(select_labels);
  select_labels_ptr->syncHAndDData(device);

  // sort each of the axes
  tensorTable.sortIndicesView("1", 0, select_labels_ptr, sortOrder::DESC, device);

  // make the expected sorted tensor
  float sorted_data[] = { 24, 25, 26, 21, 22, 23, 18, 19, 20, 15, 16, 17, 12, 13, 14, 9, 10, 11, 6, 7, 8, 3, 4, 5, 0, 1, 2 };
  Eigen::TensorMap<Eigen::Tensor<float, 3>> tensor_sorted_values(sorted_data, Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));

  // Test for sorted tensor data and reset indices view
  tensorTable.sortTensorData(device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);
    assert(tensorTable.getIndicesView().at("2")->getData()(i) == i + 1);
    assert(tensorTable.getIndicesView().at("3")->getData()(i) == i + 1);
    assert(axis_1_ptr->getLabels()(0, i) == i);
    assert(axis_2_ptr->getLabels()(0, i) == nlabels - i - 1);
    assert(axis_3_ptr->getLabels()(0, i) == nlabels - i - 1);
  }
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        assert(tensorTable.getData()(i, j, k) == tensor_sorted_values(i, j, k));
      }
    }
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_updateTensorDataValues1Gpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the tensor data and the update values
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<float, 3> update_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = float(iter);
        update_values(i, j, k) = 100;
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);
  TensorDataGpu<float, 3> values_new(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  values_new.setData(update_values);
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpu<float, 3>>(values_new);

  // reset is_modified attribute
  for (auto& is_modified_map : tensorTable.getIsModified()) {
    is_modified_map.second->getData() = is_modified_map.second->getData().constant(1);
  }
  
  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  values_new_ptr->syncHAndDData(device);

  // Test update
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> values_old_ptr;
  tensorTable.updateTensorDataValues(values_new_ptr, values_old_ptr, device);
  values_old_ptr->syncHAndDData(device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        assert(values_old_ptr->getData()(i, j, k) == float(iter));
        assert(tensorTable.getData()(i, j, k) == 100);
        ++iter;
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getInMemory().at("1")->getData()(i) == 1);
    assert(tensorTable.getInMemory().at("2")->getData()(i) == 1);
    assert(tensorTable.getInMemory().at("3")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("1")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("2")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("3")->getData()(i) == 1);
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_updateTensorDataValues2Gpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the tensor data and the update values
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<float, 3> update_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = float(iter);
        update_values(i, j, k) = 100;
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);
  TensorDataGpu<float, 3> values_new(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  values_new.setData(update_values);
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpu<float, 3>>(values_new);

  // reset is_modified attribute
  for (auto& is_modified_map : tensorTable.getIsModified()) {
    is_modified_map.second->getData() = is_modified_map.second->getData().constant(1);
  }

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  values_new_ptr->syncHAndDData(device);

  // Test update
  TensorDataGpu<float, 3> values_old(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  values_old.setData();
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> values_old_ptr = std::make_shared<TensorDataGpu<float, 3>>(values_old);
  values_old_ptr->syncHAndDData(device);
  tensorTable.updateTensorDataValues(values_new_ptr->getDataPointer(), values_old_ptr->getDataPointer(), device);
  values_old_ptr->syncHAndDData(device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        assert(values_old_ptr->getData()(i, j, k) == float(iter));
        assert(tensorTable.getData()(i, j, k) == 100);
        ++iter;
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getInMemory().at("1")->getData()(i) == 1);
    assert(tensorTable.getInMemory().at("2")->getData()(i) == 1);
    assert(tensorTable.getInMemory().at("3")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("1")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("2")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("3")->getData()(i) == 1);
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_makeAppendIndicesGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);

  // test the making the append indices
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_ptr;
  tensorTable.makeAppendIndices("1", nlabels, indices_ptr, device);
  indices_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    assert(indices_ptr->getData()(i) == nlabels + i + 1);
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_appendToIndicesGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the new indices
  Eigen::Tensor<int, 1> indices_new_values(nlabels - 1);
  for (int i = 0; i < nlabels - 1; ++i) {
    indices_new_values(i) = nlabels + i + 1;
  }
  TensorDataGpu<int, 1> indices_new(Eigen::array<Eigen::Index, 1>({ nlabels - 1 }));
  indices_new.setData(indices_new_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_new_ptr = std::make_shared<TensorDataGpu<int, 1>>(indices_new);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  indices_new_ptr->syncHAndDData(device);

  // test appendToIndices
  tensorTable.appendToIndices("1", indices_new_ptr, device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(tensorTable.getDimensions().at(tensorTable.getDimFromAxisName("1")) == nlabels + nlabels - 1);
  for (int i = 0; i < nlabels + nlabels - 1; ++i) {
    assert(tensorTable.getIndices().at("1")->getData()(i) == i + 1);
    assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);
    assert(tensorTable.getShardId().at("1")->getData()(i) == 1);
    if (i < nlabels) {
      assert(tensorTable.getIsModified().at("1")->getData()(i) == 0);
      assert(tensorTable.getInMemory().at("1")->getData()(i) == 0);
      assert(tensorTable.getShardIndices().at("1")->getData()(i) == i + 1);
    }
    else {
      assert(tensorTable.getIsModified().at("1")->getData()(i) == 1);
      assert(tensorTable.getInMemory().at("1")->getData()(i) == 1);
      assert(tensorTable.getShardIndices().at("1")->getData()(i) == 0);
    }
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_appendToAxisGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  int iter = 0;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = float(iter);
        ++iter;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // setup the new tensor data
  Eigen::Tensor<float, 3> update_values(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      update_values(0, i, j) = i;
    }
  }
  TensorDataGpu<float, 3> values_new(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  values_new.setData(update_values);
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpu<float, 3>>(values_new);

  // setup the new axis labels
  Eigen::Tensor<int, 2> labels_values(Eigen::array<Eigen::Index, 2>({ 1, 1 }));
  labels_values(0, 0) = 3;
  TensorDataGpu<int, 2> labels_new(Eigen::array<Eigen::Index, 2>({ 1, 1 }));
  labels_new.setData(labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_new_ptr = std::make_shared<TensorDataGpu<int, 2>>(labels_new);

  // setup the new indices
  TensorDataGpu<int, 1> indices_new(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_new.setData();
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_new_ptr = std::make_shared<TensorDataGpu<int, 1>>(indices_new);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  labels_new_ptr->syncHAndDData(device);
  values_new_ptr->syncHAndDData(device);
  indices_new_ptr->syncHAndDData(device);

  // test appendToAxis
  tensorTable.appendToAxis("1", labels_new_ptr, values_new_ptr->getDataPointer(), indices_new_ptr, device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  indices_new_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    assert(axis_1_ptr->getLabels()(0, i) == labels1(i));
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        assert(tensorTable.getData()(i, j, k) == tensor_values(i, j, k));
      }
    }
  }
  assert(axis_1_ptr->getLabels()(0, nlabels), 3);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      assert(tensorTable.getData()(nlabels, i, j) == update_values(0, i, j));
    }
  }
  assert(indices_new_ptr->getData()(0) == nlabels + 1);
}

void test_makeIndicesViewSelectFromIndicesGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the selection indices
  Eigen::Tensor<int, 1> indices_to_select_values(Eigen::array<Eigen::Index, 1>({ 2 }));
  indices_to_select_values.setValues({ 1, 2 });
  TensorDataGpu<int, 1> indices_to_select(Eigen::array<Eigen::Index, 1>({ 2 }));
  indices_to_select.setData(indices_to_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_to_select_ptr = std::make_shared<TensorDataGpu<int, 1>>(indices_to_select);
  
  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  indices_to_select_ptr->syncHAndDData(device);

  // test makeIndicesViewSelectFromIndices
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_select_ptr;
  tensorTable.makeIndicesViewSelectFromIndices("1", indices_select_ptr, indices_to_select_ptr, true, device);
  indices_select_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    if (i > 1)
      assert(indices_select_ptr->getData()(i) == 1);
    else
      assert(indices_select_ptr->getData()(i) == 0);
  }
  indices_select_ptr.reset();
  tensorTable.makeIndicesViewSelectFromIndices("1", indices_select_ptr, indices_to_select_ptr, false, device);
  indices_select_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    if (i <= 1)
      assert(indices_select_ptr->getData()(i) == 1);
    else
      assert(indices_select_ptr->getData()(i) == 0);
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_deleteFromIndicesGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the selection indices
  Eigen::Tensor<int, 1> indices_to_select_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_to_select_values.setValues({ 2 });
  TensorDataGpu<int, 1> indices_to_select(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_to_select.setData(indices_to_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_to_select_ptr = std::make_shared<TensorDataGpu<int, 1>>(indices_to_select);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  indices_to_select_ptr->syncHAndDData(device);

  // test deleteFromIndices
  tensorTable.deleteFromIndices("1", indices_to_select_ptr, device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(tensorTable.getDimensions().at(tensorTable.getDimFromAxisName("1")) == nlabels - 1);
  for (int i = 0; i < nlabels - 1; ++i) {
    if (i == 0) {
      assert(tensorTable.getIndices().at("1")->getData()(i) == i + 1);
      assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);
      assert(tensorTable.getShardIndices().at("1")->getData()(i) == i + 1);
    }
    else {
      assert(tensorTable.getIndices().at("1")->getData()(i) == i + 2);
      assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 2);
      assert(tensorTable.getShardIndices().at("1")->getData()(i) == i + 2);
    }
    assert(tensorTable.getIsModified().at("1")->getData()(i) == 0);
    assert(tensorTable.getInMemory().at("1")->getData()(i) == 0);
    assert(tensorTable.getShardId().at("1")->getData()(i) == 1);
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_makeSelectIndicesFromIndicesGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the selection indices
  Eigen::Tensor<int, 1> indices_to_select_values(Eigen::array<Eigen::Index, 1>({ nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    if (i % 2 == 0) indices_to_select_values(i) = i + 1;
    else indices_to_select_values(i) = 0;
  }
  TensorDataGpu<int, 1> indices_to_select(Eigen::array<Eigen::Index, 1>({ nlabels }));
  indices_to_select.setData(indices_to_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_to_select_ptr = std::make_shared<TensorDataGpu<int, 1>>(indices_to_select);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  indices_to_select_ptr->syncHAndDData(device);

  // test the selection indices
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 3>> indices_select_ptr;
  tensorTable.makeSelectIndicesFromIndices("1", indices_to_select_ptr, indices_select_ptr, device);
  indices_select_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        if (i % 2 == 0)
          assert(indices_select_ptr->getData()(i, j, k) == 1);
        else
          assert(indices_select_ptr->getData()(i, j, k) == 0);
      }
    }
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_deleteFromAxisGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  Eigen::Tensor<float, 3> new_values(Eigen::array<Eigen::Index, 3>({ nlabels - 1, nlabels, nlabels }));
  int iter = 0;
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        tensor_values(i, j, k) = i + j * nlabels + k * nlabels*nlabels;
        if (i != 1) {
          new_values(iter, j, k) = i + j * nlabels + k * nlabels*nlabels;
        }
      }
    }
    if (i != 1) ++iter;
  }
  tensorTable.setData(tensor_values);

  // setup the selection indices
  Eigen::Tensor<int, 1> indices_to_select_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_to_select_values.setValues({ 2 });
  TensorDataGpu<int, 1> indices_to_select(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_to_select.setData(indices_to_select_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_to_select_ptr = std::make_shared<TensorDataGpu<int, 1>>(indices_to_select);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  indices_to_select_ptr->syncHAndDData(device);

  // test deleteFromAxis
  TensorDataGpu<float, 3> values(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  values.setData();
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> values_ptr = std::make_shared<TensorDataGpu<float, 3>>(values);
  values_ptr->syncHAndDData(device);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_ptr;
  tensorTable.deleteFromAxis("1", indices_to_select_ptr, labels_ptr, values_ptr->getDataPointer(), device);

  // test the expected indices sizes and values
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  values_ptr->syncHAndDData(device);
  labels_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(tensorTable.getDimensions().at(tensorTable.getDimFromAxisName("1")) == nlabels - 1);
  for (int i = 0; i < nlabels - 1; ++i) {
    if (i == 0) {
      assert(tensorTable.getIndices().at("1")->getData()(i) == i + 1);
      assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);
      assert(tensorTable.getShardIndices().at("1")->getData()(i) == i + 1);
    }
    else {
      assert(tensorTable.getIndices().at("1")->getData()(i) == i + 2);
      assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 2);
      assert(tensorTable.getShardIndices().at("1")->getData()(i) == i + 2);
    }
    assert(tensorTable.getIsModified().at("1")->getData()(i) == 1);
    assert(tensorTable.getInMemory().at("1")->getData()(i) == 1);
    assert(tensorTable.getShardId().at("1")->getData()(i) == 1);
  }

  // Test the expected data values
  for (int i = 0; i < nlabels - 1; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        assert(tensorTable.getData()(i, j, k) == new_values(i, j, k));
      }
    }
  }

  // Test the expected axis values
  std::vector<int> expected_labels = { 0, 2 };
  for (int i = 0; i < nlabels - 1; ++i) {
    assert(axis_1_ptr->getLabels()(0, i) == expected_labels.at(i));
  }

  // Test the expected returned labels
  assert(labels_ptr->getData()(0, 0) == 1);

  // Test the expected returned data
  for (int i = 0; i < 1; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        assert(values_ptr->getData()(i, j, k) == tensor_values(1, j, k));
      }
    }
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_makeIndicesFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // modify the indices view for axis 1
  tensorTable.getIndicesView().at("1")->getData()(0) = 0;

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);

  // test makeIndicesFromIndicesView
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_ptr;
  tensorTable.makeIndicesFromIndicesView("1", indices_ptr, device);
  indices_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  for (int i = 0; i < nlabels - 1; ++i) {
    assert(indices_ptr->getData()(i) == i + 2);
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_insertIntoAxisGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = i + j * nlabels + k * nlabels*nlabels;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // setup the new tensor data
  Eigen::Tensor<float, 3> update_values(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  for (int i = 0; i < nlabels; ++i) {
    for (int j = 0; j < nlabels; ++j) {
      update_values(0, i, j) = 100;
    }
  }
  TensorDataGpu<float, 3> values_new(Eigen::array<Eigen::Index, 3>({ 1, nlabels, nlabels }));
  values_new.setData(update_values);
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> values_new_ptr = std::make_shared<TensorDataGpu<float, 3>>(values_new);

  // setup the new axis labels
  Eigen::Tensor<int, 2> labels_values(Eigen::array<Eigen::Index, 2>({ 1, 1 }));
  labels_values(0, 0) = 100;
  TensorDataGpu<int, 2> labels_new(Eigen::array<Eigen::Index, 2>({ 1, 1 }));
  labels_new.setData(labels_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_new_ptr = std::make_shared<TensorDataGpu<int, 2>>(labels_new);

  // setup the new indices
  Eigen::Tensor<int, 1> indices_values(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_values(0) = 3;
  TensorDataGpu<int, 1> indices_new(Eigen::array<Eigen::Index, 1>({ 1 }));
  indices_new.setData(indices_values);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 1>> indices_new_ptr = std::make_shared<TensorDataGpu<int, 1>>(indices_new);

  // Change the indices and indices view to simulate a deletion
  tensorTable.getIndices().at("1")->getData()(nlabels - 1) = 4;
  tensorTable.getIndicesView().at("1")->getData()(nlabels - 1) = 4;

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  values_new_ptr->syncHAndDData(device);
  labels_new_ptr->syncHAndDData(device);
  indices_new_ptr->syncHAndDData(device);

  // test appendToAxis
  tensorTable.insertIntoAxis("1", labels_new_ptr, values_new_ptr->getDataPointer(), indices_new_ptr, device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  std::cout << "IndicesView:\n" << tensorTable.getIndicesView().at("1")->getData() << std::endl;
  std::cout << "Labels:\n" << axis_1_ptr->getLabels() << std::endl;
  std::cout << "TensorTable:\n" << tensorTable.getData() << std::endl;
  int iter = 0;
  for (int i = 0; i < nlabels + 1; ++i) {
    // check the axis
    if (i == 2)
      assert(axis_1_ptr->getLabels()(0, i) == 100);
    else
      assert(axis_1_ptr->getLabels()(0, i) == labels1(iter));

    // check the indices
    assert(tensorTable.getIndicesView().at("1")->getData()(i) == i + 1);

    for (int j = 0; j < nlabels; ++j) {
      for (int k = 0; k < nlabels; ++k) {
        // check the tensor data
        if (i == 2)
          assert(tensorTable.getData()(i, j, k) == 100);
        else
          assert(tensorTable.getData()(i, j, k) == tensor_values(iter, j, k));
      }
    }
    if (i != 2) ++iter;
  }
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_makeSparseAxisLabelsFromIndicesViewGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<int, 2> expected_values(Eigen::array<Eigen::Index, 2>({ 3, nlabels*nlabels*nlabels }));
  expected_values.setValues({
    {1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3 },
    {1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3 },
    {1,1,1,1,1,1,1,1,1,2,2,2,2,2,2,2,2,2,3,3,3,3,3,3,3,3,3 } });

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);

  // Test
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> labels_ptr;
  tensorTable.makeSparseAxisLabelsFromIndicesView(labels_ptr, device);
  labels_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  assert(labels_ptr->getDimensions().at(0) == 3);
  assert(labels_ptr->getDimensions().at(1) == nlabels*nlabels*nlabels);
  for (int i = 0; i < 3; ++i) {
    for (int j = 0; j < nlabels*nlabels*nlabels; ++j) {
      assert(labels_ptr->getData()(i, j) == expected_values(i, j));
    }
  }

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_makeSparseTensorTableGpu()
{
  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the expected axes
  Eigen::Tensor<std::string, 1> dimensions1(3), dimensions2(1);
  dimensions1.setValues({ "0", "1", "2" });
  dimensions2(0) = "Values";

  // setup the expected labels
  int nlabels1 = 27;
  Eigen::Tensor<int, 2> labels1(3, nlabels1), labels2(1, 1);
  labels1.setValues({
    {1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3 },
    {1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3 },
    {1,1,1,1,1,1,1,1,1,2,2,2,2,2,2,2,2,2,3,3,3,3,3,3,3,3,3 } });
  TensorDataGpu<int, 2> sparse_labels(Eigen::array<Eigen::Index, 2>({ 3, nlabels1 }));
  sparse_labels.setData(labels1);
  std::shared_ptr<TensorData<int, Eigen::GpuDevice, 2>> sparse_labels_ptr = std::make_shared<TensorDataGpu<int, 2>>(sparse_labels);

  labels2.setConstant(0);

  // setup the expected data
  int nlabels = 3;
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = i + j * nlabels + k * nlabels*nlabels;
      }
    }
  }
  TensorDataGpu<float, 3> sparse_data(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  sparse_data.setData(tensor_values);
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 3>> sparse_data_ptr = std::make_shared<TensorDataGpu<float, 3>>(sparse_data);

  // Test
  std::shared_ptr<TensorTable<float, Eigen::GpuDevice, 2>> sparse_table_ptr;
  sparse_labels_ptr->syncHAndDData(device);
  sparse_data_ptr->syncHAndDData(device);
  sparse_labels_ptr->syncHAndDData(device);
  TensorTableGpu<float, 3> tensorTable;
  tensorTable.makeSparseTensorTable(dimensions1, sparse_labels_ptr, sparse_data_ptr, sparse_table_ptr, device);
  sparse_labels_ptr->syncHAndDData(device);
  sparse_data_ptr->syncHAndDData(device);
  sparse_table_ptr->syncIndicesHAndDData(device);
  sparse_table_ptr->syncIndicesViewHAndDData(device);
  sparse_table_ptr->syncInMemoryHAndDData(device);
  sparse_table_ptr->syncIsModifiedHAndDData(device);
  sparse_table_ptr->syncShardIdHAndDData(device);
  sparse_table_ptr->syncShardIndicesHAndDData(device);
  sparse_table_ptr->syncAxesHAndDData(device);
  sparse_table_ptr->syncHAndDData(device);
  sparse_labels_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);

  // Check for the correct dimensions
  assert(sparse_table_ptr->getDimensions().at(0) == nlabels1);
  assert(sparse_table_ptr->getDimensions().at(1) == 1);

  // Check the data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        assert(sparse_table_ptr->getData()(i + j * nlabels + k * nlabels*nlabels) == tensor_values(i, j, k));
      }
    }
  }

  // Check the Indices axes
  assert(sparse_table_ptr->getAxes().at("Indices")->getName() == "Indices");
  assert(sparse_table_ptr->getAxes().at("Indices")->getNLabels() == nlabels1);
  assert(sparse_table_ptr->getAxes().at("Indices")->getNDimensions() == 3);
  // TODO: transfer to host
  //std::shared_ptr<int> labels1_ptr;
  //sparse_table_ptr->getAxes().at("Indices")->getLabelsDataPointer(labels1_ptr);
  //Eigen::TensorMap<Eigen::Tensor<int, 2>> labels_values(labels1_ptr.get(), 3, nlabels1);
  for (int i = 0; i < 3; ++i) {
    assert(sparse_table_ptr->getAxes().at("Indices")->getDimensions()(i) == std::to_string(i));
    //for (int j = 0; j < nlabels1; ++j) {
    //  assert(labels_values(i, j) == labels1(i, j));
    //}
  }

  // Check the Values axes
  assert(sparse_table_ptr->getAxes().at("Values")->getName() == "Values");
  assert(sparse_table_ptr->getAxes().at("Values")->getNLabels() == 1);
  assert(sparse_table_ptr->getAxes().at("Values")->getNDimensions() == 1);
  // TODO: transfer to host
  //std::shared_ptr<int> labels2_ptr;
  //sparse_table_ptr->getAxes().at("Values")->getLabelsDataPointer(labels2_ptr);
  //Eigen::TensorMap<Eigen::Tensor<int, 2>> labels2_values(labels2_ptr.get(), 1, 1);
  //assert(labels2_values(0, 0), 0);
  assert(sparse_table_ptr->getAxes().at("Values")->getDimensions()(0) == "Values");

  // Check the indices axis indices
  for (int i = 0; i < nlabels1; ++i) {
    assert(sparse_table_ptr->getIndices().at("Indices")->getData()(i) == i + 1);
    assert(sparse_table_ptr->getIndicesView().at("Indices")->getData()(i) == i + 1);
    assert(sparse_table_ptr->getIsModified().at("Indices")->getData()(i) == 1);
    assert(sparse_table_ptr->getInMemory().at("Indices")->getData()(i) == 1);
    assert(sparse_table_ptr->getShardId().at("Indices")->getData()(i) == 1);
    assert(sparse_table_ptr->getShardIndices().at("Indices")->getData()(i) == i + 1);
  }

  // Check the values axis indices
  for (int i = 0; i < 1; ++i) {
    assert(sparse_table_ptr->getIndices().at("Values")->getData()(i) == i + 1);
    assert(sparse_table_ptr->getIndicesView().at("Values")->getData()(i) == i + 1);
    assert(sparse_table_ptr->getIsModified().at("Values")->getData()(i) == 1);
    assert(sparse_table_ptr->getInMemory().at("Values")->getData()(i) == 1);
    assert(sparse_table_ptr->getShardId().at("Values")->getData()(i) == 1);
    assert(sparse_table_ptr->getShardIndices().at("Values")->getData()(i) == i + 1);
  }

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_getSelectTensorDataAsSparseTensorTableGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = i + j * nlabels + k * nlabels*nlabels;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);

  // Test
  std::shared_ptr<TensorTable<float, Eigen::GpuDevice, 2>> sparse_table_ptr;
  tensorTable.getSelectTensorDataAsSparseTensorTable(sparse_table_ptr, device);
  sparse_table_ptr->syncIndicesHAndDData(device);
  sparse_table_ptr->syncIndicesViewHAndDData(device);
  sparse_table_ptr->syncInMemoryHAndDData(device);
  sparse_table_ptr->syncIsModifiedHAndDData(device);
  sparse_table_ptr->syncShardIdHAndDData(device);
  sparse_table_ptr->syncShardIndicesHAndDData(device);
  sparse_table_ptr->syncAxesHAndDData(device);
  sparse_table_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);

  // setup the expected labels
  int nlabels1 = 27;
  Eigen::Tensor<int, 2> labels1_expected(3, nlabels1);
  labels1_expected.setValues({
    {1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3,1,2,3 },
    {1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3,1,1,1,2,2,2,3,3,3 },
    {1,1,1,1,1,1,1,1,1,2,2,2,2,2,2,2,2,2,3,3,3,3,3,3,3,3,3 } });

  // Check for the correct dimensions
  assert(sparse_table_ptr->getDimensions().at(0) == nlabels1);
  assert(sparse_table_ptr->getDimensions().at(1) == 1);

  // Check the data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        assert(sparse_table_ptr->getData()(i + j * nlabels + k * nlabels*nlabels) == tensor_values(i, j, k));
      }
    }
  }

  // Check the Indices axes
  assert(sparse_table_ptr->getAxes().at("Indices")->getName() == "Indices");
  assert(sparse_table_ptr->getAxes().at("Indices")->getNLabels() == nlabels1);
  assert(sparse_table_ptr->getAxes().at("Indices")->getNDimensions() == 3);
  // TODO: transfer to host
  //std::shared_ptr<int> labels1_ptr;
  //sparse_table_ptr->getAxes().at("Indices")->getLabelsDataPointer(labels1_ptr);
  //Eigen::TensorMap<Eigen::Tensor<int, 2>> labels_values(labels1_ptr.get(), 3, nlabels1);
  for (int i = 0; i < 3; ++i) {
    assert(sparse_table_ptr->getAxes().at("Indices")->getDimensions()(i) == std::to_string(i + 1));
    //for (int j = 0; j < nlabels1; ++j) {
    //  assert(labels_values(i, j) == labels1_expected(i, j));
    //}
  }

  // Check the Values axes
  assert(sparse_table_ptr->getAxes().at("Values")->getName() == "Values");
  assert(sparse_table_ptr->getAxes().at("Values")->getNLabels() == 1);
  assert(sparse_table_ptr->getAxes().at("Values")->getNDimensions() == 1);
  // TODO: transfer to host
  //std::shared_ptr<int> labels2_ptr;
  //sparse_table_ptr->getAxes().at("Values")->getLabelsDataPointer(labels2_ptr);
  //Eigen::TensorMap<Eigen::Tensor<int, 2>> labels2_values(labels2_ptr.get(), 1, 1);
  //assert(labels2_values(0, 0) == 0);
  assert(sparse_table_ptr->getAxes().at("Values")->getDimensions()(0) == "Values");

  // Check the indices axis indices
  for (int i = 0; i < nlabels1; ++i) {
    assert(sparse_table_ptr->getIndices().at("Indices")->getData()(i) == i + 1);
    assert(sparse_table_ptr->getIndicesView().at("Indices")->getData()(i) == i + 1);
    assert(sparse_table_ptr->getIsModified().at("Indices")->getData()(i) == 1);
    assert(sparse_table_ptr->getInMemory().at("Indices")->getData()(i) == 1);
    assert(sparse_table_ptr->getShardId().at("Indices")->getData()(i) == 1);
    assert(sparse_table_ptr->getShardIndices().at("Indices")->getData()(i) == i + 1);
  }

  // Check the values axis indices
  for (int i = 0; i < 1; ++i) {
    assert(sparse_table_ptr->getIndices().at("Values")->getData()(i) == i + 1);
    assert(sparse_table_ptr->getIndicesView().at("Values")->getData()(i) == i + 1);
    assert(sparse_table_ptr->getIsModified().at("Values")->getData()(i) == 1);
    assert(sparse_table_ptr->getInMemory().at("Values")->getData()(i) == 1);
    assert(sparse_table_ptr->getShardId().at("Values")->getData()(i) == 1);
    assert(sparse_table_ptr->getShardIndices().at("Values")->getData()(i) == i + 1);
  }

  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

void test_updateTensorDataConstantGpu()
{
  // setup the table
  TensorTableGpu<float, 3> tensorTable;

  // Initialize the device
  cudaStream_t stream;
  assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
  Eigen::GpuStreamDevice stream_device(&stream, 0);
  Eigen::GpuDevice device(&stream_device);

  // setup the axes
  Eigen::Tensor<std::string, 1> dimensions1(1), dimensions2(1), dimensions3(1);
  dimensions1(0) = "x";
  dimensions2(0) = "y";
  dimensions3(0) = "z";
  int nlabels = 3;
  Eigen::Tensor<int, 2> labels1(1, nlabels), labels2(1, nlabels), labels3(1, nlabels);
  labels1.setValues({ {0, 1, 2} });
  labels2.setValues({ {0, 1, 2} });
  labels3.setValues({ {0, 1, 2} });
  auto axis_1_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("1", dimensions1, labels1));
  auto axis_2_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("2", dimensions2, labels2));
  auto axis_3_ptr = std::make_shared<TensorAxisGpu<int>>(TensorAxisGpu<int>("3", dimensions3, labels3));
  tensorTable.addTensorAxis(axis_1_ptr);
  tensorTable.addTensorAxis(axis_2_ptr);
  tensorTable.addTensorAxis(axis_3_ptr);
  tensorTable.setAxes();

  // setup the tensor data
  Eigen::Tensor<float, 3> tensor_values(Eigen::array<Eigen::Index, 3>({ nlabels, nlabels, nlabels }));
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        tensor_values(i, j, k) = i + j * nlabels + k * nlabels*nlabels;
      }
    }
  }
  tensorTable.setData(tensor_values);

  // reset is_modified attribute
  for (auto& is_modified_map : tensorTable.getIsModified()) {
    is_modified_map.second->getData() = is_modified_map.second->getData().constant(1);
  }

  // setup the update values
  TensorDataGpu<float, 1> values_new(Eigen::array<Eigen::Index, 1>({ 1 }));
  values_new.setData();
  values_new.getData()(0) = 100;
  std::shared_ptr<TensorData<float, Eigen::GpuDevice, 1>> values_new_ptr = std::make_shared<TensorDataGpu<float, 1>>(values_new);

  // sync the tensorTable
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  values_new_ptr->syncHAndDData(device);

  // Test update
  std::shared_ptr<TensorTable<float, Eigen::GpuDevice, 2>> values_old_ptr;
  tensorTable.updateTensorDataConstant(values_new_ptr, values_old_ptr, device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  values_old_ptr->syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);

  // Test the data
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        assert(values_old_ptr->getData()(i + j * nlabels + k * nlabels*nlabels) == tensor_values(i, j, k));
        assert(tensorTable.getData()(i, j, k) == 100);
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getInMemory().at("1")->getData()(i) == 1);
    assert(tensorTable.getInMemory().at("2")->getData()(i) == 1);
    assert(tensorTable.getInMemory().at("3")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("1")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("2")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("3")->getData()(i) == 1);
  }

  // reset is_modified attribute
  for (auto& is_modified_map : tensorTable.getIsModified()) {
    is_modified_map.second->getData() = is_modified_map.second->getData().constant(1);
  }

  // Revert the operation and test
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  values_old_ptr->syncHAndDData(device);
  tensorTable.updateTensorDataFromSparseTensorTable(values_old_ptr, device);
  tensorTable.syncIndicesHAndDData(device);
  tensorTable.syncIndicesViewHAndDData(device);
  tensorTable.syncInMemoryHAndDData(device);
  tensorTable.syncIsModifiedHAndDData(device);
  tensorTable.syncShardIdHAndDData(device);
  tensorTable.syncShardIndicesHAndDData(device);
  tensorTable.syncAxesHAndDData(device);
  tensorTable.syncHAndDData(device);
  assert(cudaStreamSynchronize(stream) == cudaSuccess);
  std::cout << "tensorTable.getData()\n" << tensorTable.getData() << std::endl;
  std::cout << "tensor_values\n" << tensor_values << std::endl;
  for (int k = 0; k < nlabels; ++k) {
    for (int j = 0; j < nlabels; ++j) {
      for (int i = 0; i < nlabels; ++i) {
        assert(tensorTable.getData()(i, j, k) == tensor_values(i, j, k));
      }
    }
  }

  // Test for the in_memory and is_modified attributes
  for (int i = 0; i < nlabels; ++i) {
    assert(tensorTable.getInMemory().at("1")->getData()(i) == 1);
    assert(tensorTable.getInMemory().at("2")->getData()(i) == 1);
    assert(tensorTable.getInMemory().at("3")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("1")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("2")->getData()(i) == 1);
    assert(tensorTable.getIsModified().at("3")->getData()(i) == 1);
  }

  // TODO: Test after a selection (see test for TensorOperation TensorUpdateConstant)
  assert(cudaStreamDestroy(stream) == cudaSuccess);
}

int main(int argc, char** argv)
{	
  test_constructorGpu();
  test_destructorGpu(); 
  test_constructorNameAndAxesGpu();
  test_zeroIndicesViewAndResetIndicesViewGpu();
  test_selectIndicesViewGpu();
  test_broadcastSelectIndicesViewGpu();
  test_extractTensorDataGpu();
  test_selectTensorIndicesGpu();
  test_applyIndicesSelectToIndicesViewGpu();
  test_whereIndicesViewDataGpu();
  test_sliceTensorForSortGpu();
  test_sortIndicesViewDataGpu();
  test_makeSelectIndicesFromIndicesViewGpu();
  test_getSelectTensorDataFromIndicesViewGpu();
  test_selectTensorDataGpu();
  test_makeSortIndicesViewFromIndicesViewGpu();
  test_sortTensorDataGpu();
  test_updateTensorDataValues1Gpu();
  test_updateTensorDataValues2Gpu();
  test_makeAppendIndicesGpu();
  test_appendToIndicesGpu();
  test_appendToAxisGpu();
  test_makeIndicesViewSelectFromIndicesGpu();
  test_deleteFromIndicesGpu();
  test_makeSelectIndicesFromIndicesGpu();
  test_deleteFromAxisGpu();
  test_makeIndicesFromIndicesViewGpu();
  //test_insertIntoAxisGpu(); ?
  test_makeSparseAxisLabelsFromIndicesViewGpu();
  test_makeSparseTensorTableGpu();
  test_getSelectTensorDataAsSparseTensorTableGpu();
  test_updateTensorDataConstantGpu();

  return 0;
}

#endif